#include "hip/hip_runtime.h"
/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : u256_kernel.cu
//
// Date       : 05/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementation of uint256 arithmetic
// ------------------------------------------------------------------

*/

#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "log.h"
#include "utils_device.h"
#include "u256_device.h"
#include "asm_device.h"


/*
    Modular addition kernel

*/
__global__ void addmu256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t i;

    uint32_t __restrict__ *x;
    uint32_t __restrict__ *y;
    uint32_t __restrict__ *z;
  
    if(tid >= params->in_length/params->stride) {
      return;
    }

    x = (uint32_t *) &in_vector[tid * params->stride * U256K_OFFSET + U256_XOFFSET * params->stride/2];
    y = (uint32_t *) &in_vector[tid * params->stride * U256K_OFFSET + U256_YOFFSET * params->stride/2];
    z = (uint32_t *) &out_vector[tid * params->stride/2 * U256K_OFFSET];
    
    if (params->premod){
      #pragma unroll
      for (i=0; i< params->stride/2; i++){
        modu255(&x[i*NWORDS_256BIT],&x[i*NWORDS_256BIT], params->midx);
        modu255(&y[i*NWORDS_256BIT],&y[i*NWORDS_256BIT], params->midx);
      }
    }

   #pragma unroll
   for (i=0; i< params->stride/2; i++){
      addmu256(&z[i*NWORDS_256BIT],(const uint32_t *)&x[i*NWORDS_256BIT], (const uint32_t *)&y[i*NWORDS_256BIT], params->midx);
   }   
}

/*
    Modular addition + reduction kernel 
      In : x[N]   
      Out :z[N/(blockDim * stride)] 
      Ex:
         N        = 512
         stride   = 4
         BlockDim = 128
         Grid     = 1
         
         in sample    : x[0] x[1] x[2] x[3] ......................................................................................................... x[511]     
         thread       |    0           |   1            |...|      8         |...|     16         |...|     32         |...|     64         |...|       127       |
                      --------------------------------------------------------------------------------------------------------------------------------------------
         1)           |    Z[0]        |   Z[1]         |   |     Z[8]       |   |    Z[16]       |   |    Z[32]       |   |    Z[64]       |   |      Z[127]     |
                      -----------------------------------------------------------------------------------------------------------------------------------------
                      | x[0]+x[1]      | x[4]+x[5]+     |...| x[32]+x[33]+   |...| x[64]+x[65]+   |...| x[128]+x[129]+ |...| x[256]+x[257]+ |...|  x[508]+x[509]+ |
                      | x[2]+x[3]      | x[6]+x[7]      |...| x[34]+x[35]    |...| x[66]+x[67]    |...| x[130]+x[131]  |...| x[258]+x[259]  |...|  x[510]+x[511]  |  
                      |                |                |...|                |...|                |...|                |...|                |...|                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         2)           | Z[64k]         |  Z[64k+1]      |...| Z[64k+8]       |...| Z[64k+16]      |...| Z[64k+32]      |...|  ->Z[0]        |   |   -> Z[63]      |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
                      | x[0]+x[1]+     | x[4]+x[5]+     |...| x[32]+x[33]+   |...| x[64]+x[65]+   |...| x[128]+x[129]+ |...|                |...|                 |
                      | x[2]+x[3]+     | x[6]+x[7]+     |...| x[34]+x[35]+   |...| x[66]+x[67]+   |...| x[130]+x[131]+ |...|  ..........    |...| ..........      |
                      | x[256]+x[257]+ | x[260]+x[261]+ |...| x[288]+x[289]+ |...| x[320]+x[321]+ |...| x[384]+x[385]+ |...|                |...|                 |
                      | x[258]+x[259]  | x[262]+x[263]  |...| x[290]+x[291]  |...| x[322]+x[323]  |...| x[386]+x[387]+ |...|                |...|                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         3)           | Z[32k]         |  Z[32k+1]      |...| Z[32k+8]       |...| Z[32k+16]      |...|  ->Z[0]        |...|                |   |                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         4)           | Z[16k]         |  Z[16k+1]      |...| Z[16k+8]       |...|    -> Z[0]     |...|                |...|                |   |                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         5)           | Z[8k]          |  Z[8k+1]       |...|  ->Z[0]        |...| .........      |...|                |...|                |   |                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         6)           | Z[4k]          |  Z[4k+1]       |...| ..........     |...| .........      |...|                |...|                |   |                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         7)           | Z[2k]          |  Z[2k+1]       |...| ..........     |...| .........      |...|                |...|                |   |                 |
                      ---------------------------------------------------------------------------------------------------------------------------------------------
                      ---------------------------------------------------------------------------------------------------------------------------------------------
         8)           | Z[k]           |  -> Z[0]       |...|                |...|                |...|                |...|                |...|                 |


*/
__global__ void addmu256_reduce_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t i;

    extern __shared__ uint32_t smem[];
    uint32_t *smem_ptr = &smem[tid*NWORDS_256BIT];  // 0 .. blockDim

    uint32_t __restrict__ *x;
    uint32_t __restrict__ *z;
  
    if(idx >= params->in_length/params->stride) {
      return;
    }

    x = (uint32_t *) &in_vector[idx  * params->stride * U256K_OFFSET]; // 0 .. N-1

    if (gridDim.x == 1){
       z = (uint32_t *) out_vector;
    } else {
       z = (uint32_t *) &in_vector[blockIdx.x * U256K_OFFSET];  // 
    }

    if (params->premod){
      #pragma unroll
      for (i =0; i < params->stride; i++){
        modu255(&x[i*U256K_OFFSET],&x[i*U256K_OFFSET], params->midx);
      }
    }

    logDebugBigNumberTid(1,"smem[0]\n",smem_ptr);
    logDebugBigNumberTid(params->stride,"X[0]\n",&x[i * U256K_OFFSET]);

    addmu256(smem_ptr, (const uint32_t *)x, (const uint32_t *)&x[U256K_OFFSET], params->midx);

    logDebugBigNumberTid(1,"smem[i]\n",smem_ptr);

    #pragma unroll
    for (i =0; i < params->stride-2; i++){
      addmu256(smem_ptr, (const uint32_t *)smem_ptr, (const uint32_t *)&x[(i+2)*U256K_OFFSET], params->midx);

      logDebugTid("idx:%d\n",i);
      logDebugBigNumberTid(1,"smem[i]\n",smem_ptr);
    }
    __syncthreads();

    logDebugBigNumberTid(1,"smem[0]\n",smem_ptr);

    // reduction global mem
    if (blockDim.x >= 1024 && tid < 512){
      logDebugBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logDebugBigNumberTid(1,"+smem[512]\n",&smem[(tid+512)*NWORDS_256BIT]);

      addmu256(smem_ptr,
               (const uint32_t *)smem_ptr,
               (const uint32_t *)&smem[(tid+512)*NWORDS_256BIT], params->midx);

      logDebugBigNumberTid(1,"smem[0]\n",smem_ptr);
    }
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256){
      logDebugBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logDebugBigNumberTid(1,"+smem[256]\n",&smem[(tid+256)*NWORDS_256BIT]);

      addmu256(smem_ptr,
               (const uint32_t *)smem_ptr,
               (const uint32_t *)&smem[(tid+256)*NWORDS_256BIT], params->midx);

      logDebugBigNumberTid(1,"smem[=256]\n",smem_ptr);
    }
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128){
      logDebugBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logDebugBigNumberTid(1,"+smem[128]\n",&smem[(tid+128)*NWORDS_256BIT]);

      addmu256(smem_ptr,
               (const uint32_t *)smem_ptr,
               (const uint32_t *)&smem[(tid+128)*NWORDS_256BIT], params->midx);

      logDebugBigNumberTid(1,"smem[=128+0]\n",smem_ptr);
    }
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64){
      logDebugBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logDebugBigNumberTid(1,"+smem[64]\n",&smem[(tid+64)*NWORDS_256BIT]);

      addmu256(smem_ptr,
               (const uint32_t *)smem_ptr,
               (const uint32_t *)&smem[(tid+64)*NWORDS_256BIT], params->midx);

      logDebugBigNumberTid(1,"smem[=64+0]\n",smem_ptr);
    }
    __syncthreads();
    
    logDebugBigNumberTid(1,"smem[0]\n",smem_ptr);

    // unrolling warp
    if (tid < 32)
    {
        volatile uint32_t *vsmem = smem;
        logDebugBigNumberTid(1,"+smem[0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"+smem[32]\n",&smem[(tid+32)*NWORDS_256BIT]);

        addmu256(&vsmem[tid * NWORDS_256BIT],
                 &vsmem[tid * NWORDS_256BIT],
                 &vsmem[(tid+32)*NWORDS_256BIT], params->midx);

        logDebugBigNumberTid(1,"smem[=32+0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"+smem[0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"+smem[16]\n",&smem[(tid+16)*NWORDS_256BIT]);

        addmu256(&vsmem[tid*NWORDS_256BIT],
                 &vsmem[tid*NWORDS_256BIT],
                 &vsmem[(tid+16)*NWORDS_256BIT], params->midx);

        logDebugBigNumberTid(1,"smem[=16+0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"+smem[0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"+smem[8]\n",&smem[(tid+8)*NWORDS_256BIT]);

        addmu256(&vsmem[tid*NWORDS_256BIT],
                 &vsmem[tid*NWORDS_256BIT],
                 &vsmem[(tid+8)*NWORDS_256BIT], params->midx);

        logDebugBigNumberTid(1,"smem[=8+0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"smem[0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"smem[4]\n",&smem[(tid+4)*NWORDS_256BIT]);

        addmu256(&vsmem[tid*NWORDS_256BIT],
                 &vsmem[tid*NWORDS_256BIT],
                 &vsmem[(tid+4)*NWORDS_256BIT], params->midx);

        logDebugBigNumberTid(1,"smem[=4+0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"smem[0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"smem[2]\n",&smem[(tid+2)*NWORDS_256BIT]);

        addmu256(&vsmem[tid*NWORDS_256BIT],
                 &vsmem[tid*NWORDS_256BIT],
                 &vsmem[(tid+2)*NWORDS_256BIT], params->midx);

        logDebugBigNumberTid(1,"smem[=2+0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"smem[0]\n",(uint32_t *)vsmem);
        logDebugBigNumberTid(1,"smem[1]\n",&smem[(tid+1)*NWORDS_256BIT]);

        addmu256(&vsmem[tid*NWORDS_256BIT],
                 &vsmem[tid*NWORDS_256BIT],
                 &vsmem[(tid+1)*NWORDS_256BIT], params->midx);

        logDebugBigNumberTid(1,"smem[=0+1]\n",(uint32_t *)vsmem);

        if (tid==0) {
	   //TODO change be movu256
           memcpy(z, smem_ptr, sizeof(uint32_t) * NWORDS_256BIT);
           //movu256(z, smem_ptr);
           logDebugBigNumberTid(1,"Z : \n",smem_ptr);
        }
    }

      
}

__global__ void addmu256_reduce_shfl_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t sumX[] =  {0,0,0,0,0,0,0,0};
    uint32_t sumY[] =  {0,0,0,0,0,0,0,0};
    uint32_t i, size1,size2;

    extern __shared__ uint32_t smem[];

    uint32_t __restrict__ *z;
    uint32_t laneIdx = tid % warpSize;
    uint32_t warpIdx = tid / warpSize;
  
    if(idx >= params->in_length) {
      return;
    }

    movu256(sumX, &in_vector[idx * U256K_OFFSET]);
    if (params->premod){
      modu255(sumX, sumX, params->midx);
    }

    if (params->premul){
       size1 = blockDim.x >> 6;
       size2 = blockDim.x >= 32 ? 16 : blockDim.x/2;
    } else {
       size1 = 16;
       //asm("clz.b32    %0,%1;\n\t"
         //:"=r"(size2) : "r"(blockDim.x >> 6));
       size2 = blockDim.x >> 6;
    }

    logInfoTid("Size1 :%d\n",size1);
    logInfoTid("Size2 :%d\n",size2);
    // last step
    if (gridDim.x == 1){
       z = (uint32_t *) out_vector;
    } else {
       z = (uint32_t *) &in_vector[idx/blockDim.x * U256K_OFFSET];  // 
    }

    logInfoBigNumberTid(1,"X[0]\n",sumX);

    // block wide warp reduce
    #pragma unroll
    for (i = size1; i > 0; i >>= 1){
      shflxoru256(sumY, sumX, i);
      logInfoTid("idx:%d\n",i);
      logInfoBigNumberTid(1,"sumX\n",sumX);
      logInfoBigNumberTid(1,"sumY\n",sumY);

      addmu256(sumX, sumX, sumY, params->midx);

      logInfoBigNumberTid(1,"sumX+\n",sumX);
    }

    if (laneIdx == 0) {
       movu256(&smem[warpIdx*NWORDS_256BIT], sumX);
       logInfoTid("save idx:%d\n",warpIdx);
       logInfoBigNumberTid(1,"val\n",sumX);
    }

    __syncthreads();

    if (tid < size2*2) {
      logInfoTid("blockDim :%d\n",blockDim.x);
      logInfoTid("LaneIdx :%d\n",laneIdx);
      logInfoTid("Size :%d\n",size2);
      movu256(sumX,&smem[laneIdx*NWORDS_256BIT]);
      logInfoBigNumberTid(size2*2-idx,"Save\n",&smem[laneIdx*NWORDS_256BIT]);
    } else {
      set0u256(sumX);
    }
    logInfoBigNumberTid(1,"Second\n",sumX);
    #pragma unroll
    // last warp reduce
    for (i=size2; i > 0; i >>=1){
      shflxoru256(sumY, sumX, i);
      logInfoTid("idx:%d\n",i);
      logInfoBigNumberTid(1,"sumY\n",sumY);
      logInfoBigNumberTid(1,"sumX\n",sumX);
      addmu256(sumX, sumX, sumY, params->midx);
      logInfoBigNumberTid(1,"sumX+\n",sumX);
    }

    if (tid==0) {
     //TODO change be movu256
     //memcpy(z, sumX, sizeof(uint32_t) * NWORDS_256BIT);
     movu256(z, sumX);
     logInfoBigNumberTid(1,"Z : \n",sumX);
    }
}



/*
    Modular Sub kernel

*/
__global__ void submu256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t i;

    uint32_t __restrict__ *x;
    uint32_t __restrict__ *y;
    uint32_t __restrict__ *z;
   
    if(tid >= params->in_length/params->stride) {
      return;
    }

    x = (uint32_t *) &in_vector[tid * params->stride * U256K_OFFSET + U256_XOFFSET * params->stride/2];
    y = (uint32_t *) &in_vector[tid * params->stride * U256K_OFFSET + U256_YOFFSET * params->stride/2];
    z = (uint32_t *) &out_vector[tid * params->stride/2 * U256K_OFFSET];
    
    if (params->premod){
      #pragma unroll
      for (i=0; i< params->stride/2; i++){
        modu255(&x[i*NWORDS_256BIT],&x[i*NWORDS_256BIT], params->midx);
        modu255(&y[i*NWORDS_256BIT],&y[i*NWORDS_256BIT], params->midx);
      }
    }

   #pragma unroll
   for (i=0; i< params->stride/2; i++){
      submu256(&z[i*NWORDS_256BIT],(const uint32_t *)&x[i*NWORDS_256BIT], (const uint32_t *)&y[i*NWORDS_256BIT], params->midx);
   }   
} 
/*
    Modulo

*/
__global__ void modu256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    const uint32_t __restrict__ *x;
    uint32_t __restrict__ *z;
 
    if(tid >= params->in_length) {
      return;
    }

    x = (const uint32_t *) &in_vector[tid * U256K_OFFSET];
    z = (uint32_t *) &out_vector[tid * U256K_OFFSET];
    
    modu255(z, x, params->midx);
}

/*
   Montgomery multiplication
*/
__global__ void mulmontu256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid >= params->in_length/2) {
      return;
    }

    uint32_t __restrict__ *A, *B, *U;
    uint32_t i,j; 
 
    A = (uint32_t *) &in_vector[tid * 2 * U256K_OFFSET + U256_XOFFSET];
    B = (uint32_t *) &in_vector[tid * 2 * U256K_OFFSET + U256_YOFFSET];
    U = (uint32_t *) &out_vector[tid * U256K_OFFSET];
   
    mulmontu256(U, (const uint32_t *)A, (const uint32_t *) B, params->midx);

   return;
}

/*
  Montgomery multiplication extended field number
*/
__global__ void mulmontu256_2_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid >= params->in_length/4) {
      return;
    }

    uint32_t __restrict__ *A, *B, *U;
    uint32_t i,j; 
 
    A = (uint32_t *) &in_vector[tid * 4 * U256K_OFFSET + U256_XOFFSET];
    B = (uint32_t *) &in_vector[tid * 4 * U256K_OFFSET + U256_YOFFSET];
    U = (uint32_t *) &out_vector[tid * 2 * U256K_OFFSET];
   
    mulmontu256_2(U, (const uint32_t *)A, (const uint32_t *) B, params->midx);

   return;
}

/*
  Right logical shift kernel
*/
__global__ void shr1u256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t i;
    uint32_t b_shifted;

    uint32_t __restrict__ *x;
    uint32_t __restrict__ *y;
    uint32_t __restrict__ *z;
   
    if(tid >= params->in_length) {
      return;
    }

    //memset causes blocking operations on current device
    x = (uint32_t *) &in_vector[tid * U256K_OFFSET + U256_XOFFSET];
    z = (uint32_t *) &out_vector[tid * U256K_OFFSET];
    memset(z, 0, NWORDS_256BIT*sizeof(uint32_t));
 
    logInfoBigNumberTid(1,"X: \n",x);
    #pragma unroll
    for (i=0; i< NWORDS_256BIT*32; i++){   
      b_shifted = shr1u256(x);
      z[i/32] |= (b_shifted << (i % 32));
      //logInfoTid("b : %d\n",b_shifted);
    }
    logInfoBigNumberTid(1,"Z: \n",z);
}

/*
  Left logical shift kernel
*/
__global__ void shl1u256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int i;
    uint32_t b_shifted;

    uint32_t __restrict__ *x;
    uint32_t __restrict__ *y;
    uint32_t __restrict__ *z;
   
    if(tid >= params->in_length) {
      return;
    }

    //memset causes blocking operations on current device
    x = (uint32_t *) &in_vector[tid * U256K_OFFSET + U256_XOFFSET];
    z = (uint32_t *) &out_vector[tid * U256K_OFFSET];
    memset(z, 0, NWORDS_256BIT*sizeof(uint32_t));

    logInfoBigNumberTid(1,"X: \n",x);
    #pragma unroll
    for (i= NWORDS_256BIT*32-1; i>=0; i--){   
      b_shifted = shl1u256(x);
      z[i/32] |= (b_shifted << (i % 32));
    }
    logInfoBigNumberTid(1,"Z: \n",z);
}

/*
  Left logical shift kernel
*/
__global__ void shlu256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int i;

    uint32_t __restrict__ *x;
    uint32_t __restrict__ y[NWORDS_256BIT];
    uint32_t __restrict__ *z;
   
    if(tid >= params->in_length) {
      return;
    }

    //memset causes blocking operations on current device
    x = (uint32_t *) &in_vector[tid * U256K_OFFSET + U256_XOFFSET];
    z = (uint32_t *) &out_vector[tid * U256K_OFFSET];
    memset(z, 0, NWORDS_256BIT*sizeof(uint32_t));

    logInfoBigNumberTid(1,"X: \n",x);
    #pragma unroll
    for (i= NWORDS_256BIT*32-1; i>=0; i--){   
      movu256(y,x);
      shlu256(y,i);
      logInfoBigNumberTid(1,"X: \n",x);
      logInfoBigNumberTid(1,"Z: \n",y);
    }
}

/*
  Right logical shift kernel
*/
__global__ void almmontinvu256_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t i;

    uint32_t __restrict__ *x;
    uint32_t __restrict__ *z;
   
    if(tid >= params->in_length) {
      return;
    }

    //memset causes blocking operations on current device
    x = (uint32_t *) &in_vector[tid * U256K_OFFSET + U256_XOFFSET];
    z = (uint32_t *) &out_vector[tid * U256K_OFFSET];
 
    logInfoTid("MONT INV : %d\n", tid);
    logInfoBigNumberTid(1,"X: \n",x);
    #pragma unroll
    almmontinvu256(z,x,params->midx);
    logInfoBigNumberTid(1,"Z: \n",z);
}


/*
   aA = X[0] * Y[0]
   bB = X[1] * Y[1]
   Z[0] = aA + bB * residue
   Z[1] = (X[0] + X[1]) * (Y[0] + Y[1]) - aA - bB
*/
__device__ void mulmontu256_2(uint32_t __restrict__ *U, const uint32_t __restrict__ *A, const uint32_t __restrict__ *B, mod_t midx)
{

#ifndef CU_ASM
    uint32_t tmulx[NWORDS_256BIT],tmuly[NWORDS_256BIT],tmulz[NWORDS_256BIT];
    uint32_t tmp4[NWORDS_256BIT];

    mulmontu256(tmulx, A,B,midx);                  
    mulmontu256(tmuly, &A[NWORDS_256BIT],&B[NWORDS_256BIT],midx); 

    addmu256(tmulz,A,&A[NWORDS_256BIT], midx);                
    addmu256(tmp4,B,&B[NWORDS_256BIT], midx);                
    mulmontu256(tmulz, tmulz,tmp4,midx); 
    submu256(U, tmulx, tmuly, midx);   
    addmu256(&U[NWORDS_256BIT], tmulx, tmuly, midx);                
    submu256(&U[NWORDS_256BIT], tmulz, &U[NWORDS_256BIT], midx);                

#else

     uint32_t const __restrict__ *P_u256 = mod_info_ct[midx].p;
     uint32_t const __restrict__ *PN_u256 = mod_info_ct[midx].p_;

     asm(ASM_MULG2_INIT
         ASM_MONTMULU256(tmulx,ax,bx)  
         ASM_MODU256(tmulx)
         ASM_MONTMULU256(tmuly,ay,by)  
         ASM_MODU256(tmuly)
         ASM_ADDU256(tmulz, ax, ay)    
         ASM_SUBMU256(rx, tmulx, tmuly)   
         ASM_ADDMU256(tmulx, tmulx, tmuly)  
         ASM_ADDU256(tmuly, bx, by)
         ASM_MONTMULU256(ry, tmulz, tmuly)
         ASM_MODU256(ry)
         ASM_SUBMU256(ry, ry, tmulx)
         ASM_MULG2_PACK);
#endif
}


/*
   ab =  X[0] * X[1]
   t1 =  X[0] + nonres * X[1] 
   a2  = X[0] + X[1]
   Z[0] = t1 * a2 - ab + nonres * ab
   Z[1] = ab + ab
*/
__device__ void sqmontu256_2(uint32_t __restrict__ *U, const uint32_t __restrict__ *A, mod_t midx)
{
#if 0
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t const __restrict__ *nonres = mod_info_ct[midx].nonres;
    uint32_t tmp1[NWORDS_256BIT],tmp2[NWORDS_256BIT],tmp3[NWORDS_256BIT];

    /*
    logInfoBigNumberTid(1,"X[0]:\n",(uint32_t *)A);
    logInfoBigNumberTid(1,"X[1]:\n",(uint32_t *)&A[NWORDS_256BIT]);
    */

    mulmontu256(tmp1, A,&A[NWORDS_256BIT], midx);             // Z[1] = 2 * X[0] * X[1]

    sqmontu256(tmp2, A, midx);
    sqmontu256(tmp3, &A[NWORDS_256BIT], midx);

    addmu256(&U[NWORDS_256BIT],tmp1,tmp1,midx);
    submu256(U,tmp2, tmp3,midx);                                 // Z[0] = X[0] * X[0] - (X[1] * X[1])
#else
#ifndef CU_ASM
    uint32_t tmulx[NWORDS_256BIT],tmuly[NWORDS_256BIT],tmulz[NWORDS_256BIT];

    mulmontu256(tmulx, A,A,midx);                  
    mulmontu256(tmuly, &A[NWORDS_256BIT],&A[NWORDS_256BIT],midx); 

    addmu256(tmulz,A,&A[NWORDS_256BIT], midx);                
    mulmontu256(tmulz, tmulz,tmulz,midx); 
    submu256(U, tmulx, tmuly, midx);   
    addmu256(&U[NWORDS_256BIT], tmulx, tmuly, midx);                
    submu256(&U[NWORDS_256BIT], tmulz, &U[NWORDS_256BIT], midx);                

#else

#if 0
     t_uint64 const __restrict__ *dP_u256 = (t_uint64 *)mod_info_ct[midx].p;
     t_uint64 const *dA = (t_uint64 *)A;
     t_uint64 const *dB = (t_uint64 *)A;
     t_uint64 const *dU = (t_uint64 *)U;
#else
     uint32_t const __restrict__ *P_u256 = mod_info_ct[midx].p;
     uint32_t const *B = A;
#endif
     uint32_t const __restrict__ *PN_u256 = mod_info_ct[midx].p_;

     asm(ASM_MULG2_INIT
         ASM_MONTMULU256(tmulx,ax,bx)  
         ASM_MODU256(tmulx)
         ASM_MONTMULU256(tmuly,ay,by)  
         ASM_MODU256(tmuly)
         ASM_ADDU256(tmulz, ax, ay)    
         ASM_SUBMU256(rx, tmulx, tmuly)   
         ASM_ADDMU256(tmulx, tmulx, tmuly)  
         ASM_MONTMULU256(ry, tmulz, tmulz)
         ASM_MODU256(ry)
         ASM_SUBMU256(ry, ry, tmulx)
         ASM_MULG2_PACK);
#endif
#endif
    
}


/*
   Montgomery Multiplication(xr^(-1),y^r(-1)) = xr^(-1) * yr^(-1) * r (mod N)  for 256 bit numbers
     FIOS implementatin

   NOTE. Function requires that x, y < N
   NOTE. If x or y are not in Montgomery format, output is 
    in standard format multiplication of x * y
     ex: MontMul(xr^(-1),  y) = xr^(-1) * y * r = x * y

   NOTE : According to Tolg Acar's thesis*:
      SOS   2s^2+s MUL, 4s^2+4s+2 ADD 
      FIOS  2s^2+s MUL, 5s^2+3s+2 ADD

   * www.microsoft.com/en-us/research/wp-content/uploads/1998/06/97Acar.pdf

*/
__device__ void mulmontu256(uint32_t __restrict__ *U, const uint32_t __restrict__ *A, const uint32_t __restrict__ *B, mod_t midx)
{ 
    //logInfoBigNumberTid(1,"B\n",(uint32_t *)B);
    uint32_t const __restrict__ *PN_u256 = mod_info_ct[midx].p_;

#ifndef CU_ASM
    uint32_t i;
    uint32_t S, C=0, C1, C2,C3;
    uint32_t __restrict__ M, X[2];
    uint32_t __restrict__ __align__(16) T[]={0,0,0,0,0,0,0,0,0,0};
    uint32_t const __restrict__ *P_u256 = mod_info_ct[midx].p;

    //logInfoBigNumberTid(1,"A\n",(uint32_t *)A);

    //movu256(Ar,(uint32_t *)A);
    //movu256(Br,(uint32_t *)B);

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    #pragma unroll
    for(i=0; i<NWORDS_256BIT; i++)
    {
      // (C,S) = t[0] + a[0]*b[i], worst case 2 words
      madcu32(&C,&S,A[0],B[i],T[0]);
      //logInfoTid("0 - C:%u\n",C);
      //logInfoTid("0 - S:%u\n",S);

      // ADD(t[1],C)
      //propcu32(T, C, 1);
      addcu32(&C3, &T[1], T[1], C);
      //logInfoTid("C3: %u\n",C3);
      //logInfoBigNumberTid(1,"T\n",T);

      // m = S*n'[0] mod W, where W=2^32
      // Note: X[Upper,Lower] = S*n'[0], m=X[Lower]
      mulu32lo(&M, S, PN_u256[0]);
      //logInfoTid("M[0]: %u\n", M);

      // (C,S) = S + m*n[0], worst case 2 words
      madcu32(&C,&S,M,P_u256[0],S);
      //logInfoTid("1 - C: %u\n",C );
      //logInfoTid("1 - S: %u\n", S);
  
      // FIRST IT
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[1], B[i]);
      addcu32(&C1, &S, T[1], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);

      // ADD(t[2],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[2], T[2], C);
      C3 =C2 + C1;

      // (C,S) = S + m*n[1]
      madcu32(&C,&T[0],M, P_u256[1],S);

      //j=2
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[2], B[i]);
      addcu32(&C1, &S, T[2], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);
 
      // ADD(t[j+1],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[3], T[3], C);
      C3 =C2 + C1;
 
      // (C,S) = S + m*n[j]
      madcu32(&C,&T[1],M, P_u256[2],S);
 
      // j = 3
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[3], B[i]);
      addcu32(&C1, &S, T[3], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);

      // ADD(t[j+1],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[4], T[4], C);
      C3 =C2 + C1;

      // (C,S) = S + m*n[j]
      madcu32(&C,&T[2],M, P_u256[3],S);

      // j = 4
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[4], B[i]);
      addcu32(&C1, &S, T[4], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);

      // ADD(t[j+1],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[5], T[5], C);
      C3 =C2 + C1;

      // (C,S) = S + m*n[j]
      madcu32(&C,&T[3],M, P_u256[4],S);

      // j = 5
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[5], B[i]);
      addcu32(&C1, &S, T[5], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);

      // ADD(t[j+1],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[6], T[6], C);
      C3 =C2 + C1;

      // (C,S) = S + m*n[j]
      madcu32(&C,&T[4],M, P_u256[5],S);

      // j = 6
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[6], B[i]);
      addcu32(&C1, &S, T[6], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);

      // ADD(t[j+1],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[7], T[7], C);
      C3 =C2 + C1;

      // (C,S) = S + m*n[j]
      madcu32(&C,&T[5],M, P_u256[6],S);

      // j = 7
      // (C,S) = t[j] + a[j]*b[i] + C, worst case 2 words
      mulu32(X, A[7], B[i]);
      addcu32(&C1, &S, T[7], C);
      addcu32(&C2, &S, S, X[0]);
      addcu32(&X[0], &C, C1, X[1]);
      addcu32(&C1, &C, C, C2);

      // ADD(t[j+1],C)
      //C +=C3;
      addcu32(&C2, &C, C, C3);
      C1 +=C2;
      addcu32(&C2, &T[8], T[8], C);
      C3 =C2 + C1;

      // (C,S) = S + m*n[j]
      madcu32(&C,&T[6],M, P_u256[7],S);

      //propcu32_extend(T,C3);
      // (C,S) = t[s] + C
      addcu32(&C,&T[7], T[NWORDS_256BIT], C);
      //logInfoTid("6 - C: %u\n",C );
      //logInfoTid("6 - S: %u\n",S );
      // t[s-1] = S
      // t[s] = t[s+1] + C
      addcu32(&X[0],&T[NWORDS_256BIT], T[NWORDS_256BIT+1], C);
      // t[s+1] = 0
      T[NWORDS_256BIT+1] = 0;
   }

   /* Step 3: if(u>=n) return u-n else return u */
   if (ltu256(P_u256,T)){
      subu256(U,T,P_u256);
   } else {
       movu256(U,T);
   }

#else
 t_uint64 *dA = (t_uint64 *) A;
 t_uint64 *dB = (t_uint64 *) B;
 t_uint64 *dU = (t_uint64 *) U;
 t_uint64 const __restrict__ *dP_u256 = (t_uint64 *) mod_info_ct[midx].p;
 //t_uint64 const __restrict__ *dPN_u256 = (t_uint64 *) mod_info_ct[midx].p_;

 asm(ASM_MUL_INIT_64 
     ASM_MONTMULU256(r,a,b)
     ASM_MODU256(r)
     ASM_MUL_PACK_64);

#endif
 //logInfoBigNumberTid(1,"U\n",(uint32_t *)U);

 return;

}

/*
  Montgomery Modular Inverse - Revisited
  E. Savas, C.K.Koc
  IEEE trasactions on Computers Vol49, No 7. July 2000
*/
__device__ void sqmontu256(uint32_t __restrict__ *U, const uint32_t __restrict__ *A, mod_t midx)
{
   //TODO : implement proper squaring
#ifndef CU_ASM
   mulmontu256(U,A,A,midx);
#else

 t_uint64 *dA = (t_uint64 *) A;
 t_uint64 *dB = (t_uint64 *) A;
 t_uint64 *dU = (t_uint64 *) U;
 t_uint64 const __restrict__ *dP_u256 = (t_uint64 *) mod_info_ct[midx].p;
 uint32_t const __restrict__ *PN_u256 =  mod_info_ct[midx].p_;

#if 1
 asm(ASM_MONTSQ_INIT_64 
     ASM_MONTMULU256(r,a,a)
     ASM_MODU256(r)
     ASM_MONTSQ_PACK_64);
#else
 asm(ASM_MUL_INIT_64 
     ASM_MONTMULU256(r,a,b)
     ASM_MODU256(r)
     ASM_MUL_PACK_64);
#endif

#endif
}

__device__ uint32_t almmontinvu256(uint32_t __restrict__ *y, const uint32_t __restrict__ *x, mod_t midx)
{
  const uint32_t __restrict__ *P = mod_info_ct[midx].p;

  uint32_t u[NWORDS_256BIT], v[NWORDS_256BIT];
  uint32_t s[] = {1,0,0,0,0,0,0,0};
  uint32_t r1[] = {0,0,0,0,0,0,0,0};
  uint32_t k = 0;
  uint32_t t0,t1,t2;
  uint32_t tmp[NWORDS_256BIT];

  movu256(u, (uint32_t *)P);
  movu256(v, (uint32_t *)x);

  //Phase 1 - ALmost inverse r = a^(-1) * 2 ^k, n<=k<=2n
  // u is  < 256bits
  // v is < 256 bits, < u
  // s is  1     
  // r1 is 0
#if 1
  //inv_t data_table[4];
  //uint32_t data_table_r[] = {0,1,0,3,0,1,0,2};

  //init_invtable(data_table, u, v, s, r1);

  while(eq0u256(v) == 0){
     t0 = u[0] & 0x1; 
     t1 = (v[0] & 0x1) << 1;
     subu256(tmp,v,u);
     t2 = (tmp[NWORDS_256BIT-1] & 0x80000000) >> 29;
     t0 = t0 + t1 + t2;

     logInfoTid("t0 : %d\n",t0);
     logInfoBigNumberTid(1,"u: \n",u);
     logInfoBigNumberTid(1,"v: \n",v);
     //almmontinv_step_h(&data_table[data_table_r[t0]]);	  
     if (t0 % 2 == 0) {
        shr1u256(u);
        shl1u256(s);
     } else if (t0 == 3) {
        //subu256(v,v,u);
        movu256(v, (uint32_t *)tmp);
        shr1u256(v);
        addu256(s,s,r1);
        shl1u256(r1);
     } else if (t0 == 7) {
        //subu256(u,u,v);
        negu256(u, (uint32_t *)tmp);
        shr1u256(u);
        addu256(r1,r1,s);
        shl1u256(s);
     } else{ 
        shr1u256(v);
        shl1u256(r1);
     }
     k++;
     //if (k==1000) break;
     if (k == 1000) {
        //int tid = threadIdx.x + blockDim.x * blockIdx.x;
        //logInfo("tid : %d\n",tid);
	break;
     }
  }

#else
  logInfoBigNumberTid(1,"U: \n",u);
  logInfoBigNumberTid(1,"V: \n",v);
  while(eq0u256(v) == 0){
     logInfoBigNumberTid(1,"u: \n",u);
     logInfoBigNumberTid(1,"v: \n",v);
     if (u[0] & 0x1 == 0){
        shr1u256(u);
        shl1u256(s);
     } else if (v[0] & 0x1 == 0){
        shr1u256(v);
        shl1u256(r1);
     } else if (ltu256(v,u)) {
        subu256(u,u,v);
        shr1u256(u);
        addu256(r1,r1,s);
        shl1u256(s);
     } else {
        subu256(v,v,u);
        shr1u256(v);
        addu256(s,s,r1);
        shl1u256(r1);
     }
     k++;
  }
#endif
  
  if (ltu256(P,r1)){
      subu256(r1,r1,P);
  }
  subu256(y, P,r1);

  return k;
}

__device__ uint32_t invmontu256(uint32_t __restrict__ *y, const uint32_t __restrict__ *x, mod_t midx)
{
   uint32_t k;
   uint32_t t[] = {1,0,0,0,0,0,0,0};
   uint32_t t_idx;

   const uint32_t *R[2];
   R[0] = mod_info_ct[midx].r2;
   R[1] = mod_info_ct[midx].r2modp;
   uint32_t shift[2];

   k = almmontinvu256(y, x, midx);

   t_idx = 2*NWORDS_256BIT*NBITS_WORD/k-1;
   shift[0] = 2*NWORDS_256BIT * NBITS_WORD - k;
   shift[1] = NWORDS_256BIT * NBITS_WORD - k;

   shlu256(t,shift[t_idx]);
   mulmontu256(y, y, R[t_idx],midx);
   mulmontu256(y, y, t,midx);
}

__device__ uint32_t invmontu256_2(uint32_t __restrict__ *y, const uint32_t __restrict__ *x, mod_t midx)
{
  uint32_t t0[NWORDS_256BIT], t1[NWORDS_256BIT];
  const uint32_t Zero[] = {0,0,0,0,0,0,0,0};

  sqmontu256(t0, x, midx);
  sqmontu256(t1, &x[NWORDS_256BIT], midx);
  addmu256(t0,t0,t1,midx);
  invmontu256(t0,t0,midx);

  mulmontu256(y, x, t0, midx);
  mulmontu256(&y[NWORDS_256BIT], &x[NWORDS_256BIT], t0, midx);
  submu256(&y[NWORDS_256BIT],Zero,&y[NWORDS_256BIT],midx);
}

__device__ void div2u256(uint32_t __restrict__ *z, const uint32_t __restrict__ *x)
{
  movu256(z,(uint32_t *)x);
  shr1u256(z);
}
/*
   x mod N

   NOTE : It requires that prime is at least 253 bit number. In reality is 254 bits the prime
    i am using
   */
__device__ void modu256(uint32_t __restrict__ *z, const uint32_t __restrict__ *x, mod_t midx)
{
   const uint32_t __restrict__ *p = mod_info_ct[midx].p;

   movu256(z,(uint32_t *)x);

   // x(255 bit number worst case) - p (253 bit number) = z (255 bit number) : ex 31(5 b) - 4(3 b) =27 (5 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
   // x(255 bit ) - p (253 bit number) = z (255 bit number) : ex 27(5 b) - 4(3 b) =23 (5 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
   // x(255 bit ) - p (253 bit number) = z (255 bit number) : ex 23(5 b) - 4(3 b) = 19 (5 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
   // x(255 bit ) - p (253 bit number) = z (254 bit number) : ex 19(5 b) - 4(3 b) = 15 (4 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
   // x(254 bit ) - p (253 bit number) = z (254 bit number) : ex 15(5 b) - 4(3 b) = 11 (4 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
   // x(254 bit ) - p (253 bit number) = z (254 bit number) : ex 11(5 b) - 4(3 b) = 7 (3 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
   // x(254 bit ) - p (253 bit number) = z (254 bit number) : ex 7(5 b) - 4(3 b) = 3 (3 b) 
   if (!ltu256(z,p)){
      subu256(z,z,p);
   } else { return; }
 
   assert(0);
}

/*
   x mod N

   NOTE : It requires that prime is at least 253 bit number and less than 256 bit (msb must be 0). In reality is 254 bits the prime
    i am using. modu255 is more efficient that modu256
   */
__device__ void modu255(uint32_t __restrict__ *z, const uint32_t __restrict__ *x, mod_t midx)
{
   const uint32_t __restrict__ *p = mod_info_ct[midx].p;

   #if 1
   movu256(z,(uint32_t *)x);
   #else
   asm("mov.u32     %0,  %8;\n\t"
       "mov.u32     %1,  %9;\n\t"
       "mov.u32     %2,  %10;\n\t"
       "mov.u32     %3,  %11;\n\t"
       "mov.u32     %4,  %12;\n\t"
       "mov.u32     %5,  %13;\n\t"
       "mov.u32     %6,  %14;\n\t"
       "mov.u32     %7,  %15;\n\t"
    : "=r"(z[0]), "=r"(z[1]), "=r"(z[2]), "=r"(z[3]),
      "=r"(z[4]), "=r"(z[5]), "=r"(z[6]), "=r"(z[7])
    : "r"(x[0]), "r"(x[1]), "r"(x[2]), "r"(x[3]),
      "r"(x[4]), "r"(x[5]), "r"(x[6]), "r"(x[7]));
  #endif

  if (!subgtu256(z,p)) return;
  else if (!subgtu256(z,p)) return;
  else if (!subgtu256(z,p)) return;
  else if (!subgtu256(z,p)) return;
  else if (!subgtu256(z,p)) return;
  else if (!subgtu256(z,p)) return;
  else if (!subgtu256(z,p)) return;

  assert(0);
}

/*
   x >> 1 for 256 bit number
*/
__device__ uint32_t shr1u256(uint32_t __restrict__ *x)
{
   uint32_t c; 

   asm("{                                    \n\t"
       "bfe.u32            %8,   %9,  0,1;  \n\t"       
       "shf.r.clamp.b32   %0, %9, %10, 1;   \n\t"
       "shf.r.clamp.b32   %1, %10, %11, 1 ;  \n\t"
       "shf.r.clamp.b32   %2, %11, %12, 1;   \n\t"
       "shf.r.clamp.b32   %3, %12, %13, 1;   \n\t"
       "shf.r.clamp.b32   %4, %13, %14, 1;   \n\t"
       "shf.r.clamp.b32   %5, %14, %15, 1;   \n\t"
       "shf.r.clamp.b32   %6, %15, %16, 1;   \n\t"
       "shr.b32           %7, %16, 1;        \n\t"
       "}                               \n\t"
       : "=r"(x[0]), "=r"(x[1]), "=r"(x[2]), "=r"(x[3]), 
         "=r"(x[4]), "=r"(x[5]), "=r"(x[6]), "=r"(x[7]), "=r"(c)
       : "r"(x[0]), "r"(x[1]), "r"(x[2]), "r"(x[3]), 
         "r"(x[4]), "r"(x[5]), "r"(x[6]), "r"(x[7]));

      return c;

}

/*
   x << 1 for 256 bit number
*/
__device__ uint32_t shl1u256(uint32_t __restrict__ *x)
{
   uint32_t c; 

   asm("{                                    \n\t"
       "bfe.u32            %8,   %16,  31,1;  \n\t"       // c = x[7] & (1<<31)
       "shf.l.clamp.b32   %7, %15, %16, 1;   \n\t"
       "shf.l.clamp.b32   %6, %14, %15, 1 ;  \n\t"
       "shf.l.clamp.b32   %5, %13, %14, 1;   \n\t"
       "shf.l.clamp.b32   %4, %12, %13, 1;   \n\t"
       "shf.l.clamp.b32   %3, %11, %12, 1;   \n\t"
       "shf.l.clamp.b32   %2, %10, %11, 1;   \n\t"
       "shf.l.clamp.b32   %1, %9, %10, 1;   \n\t"
       "shl.b32           %0, %9, 1;        \n\t"
       "}                               \n\t"
       : "=r"(x[0]), "=r"(x[1]), "=r"(x[2]), "=r"(x[3]), 
         "=r"(x[4]), "=r"(x[5]), "=r"(x[6]), "=r"(x[7]), "=r"(c)
       : "r"(x[0]), "r"(x[1]), "r"(x[2]), "r"(x[3]), 
         "r"(x[4]), "r"(x[5]), "r"(x[6]), "r"(x[7]));

      return c;

}


/*
   x << count for 256 bit number
*/
__device__ void shlu256(uint32_t *x, uint32_t count)
{
   uint32_t word_idx =  count >> NBITS_WORD_LOG2;
   uint32_t bit_idx = NBITS_WORD - (count & NBITS_WORD_MOD);
   uint32_t bit_count = NBITS_WORD - bit_idx;
   uint32_t i;

   logInfoTid("count :%d\n",count);
   logInfoTid("word_idx :%d\n",word_idx);
   logInfoTid("bit_idx :%d\n",bit_idx);
   logInfoTid("bit_count :%d\n",bit_count);
   asm("{                                    \n\t"
       "shf.l.clamp.b32   %7, %14, %15, %16;   \n\t"
       "shf.l.clamp.b32   %6, %13, %14, %16;   \n\t"
       "shf.l.clamp.b32   %5, %12, %13, %16;   \n\t"
       "shf.l.clamp.b32   %4, %11, %12, %16;   \n\t"
       "shf.l.clamp.b32   %3, %10, %11, %16;   \n\t"
       "shf.l.clamp.b32   %2, %9, %10, %16;   \n\t"
       "shf.l.clamp.b32   %1, %8, %9, %16;   \n\t"
       "shl.b32            %0, %8, %16;        \n\t"
       "}                               \n\t"
       : "=r"(x[0]), "=r"(x[1]), "=r"(x[2]), "=r"(x[3]), 
         "=r"(x[4]), "=r"(x[5]), "=r"(x[6]), "=r"(x[7]) 
       : "r"(x[0]), "r"(x[1]), "r"(x[2]), "r"(x[3]), 
         "r"(x[4]), "r"(x[5]), "r"(x[6]), "r"(x[7]),
	 "r"(bit_count));

   for (i=word_idx; i < NWORDS_256BIT; i++){
      x[NWORDS_256BIT-1-i+word_idx] = x[NWORDS_256BIT-1-i];	   
   }
   for (i=0; i < word_idx; i++){
      x[i] = 0;
   }
}
/*
   (x & (1<< bsel)) >> bsel  for 256 bit number
*/
__device__ uint32_t bselu256(const uint32_t __restrict__ *x, uint32_t bsel)
{
   uint32_t c;
   uint32_t word = bsel >> NBITS_WORD_LOG2; // bsel/32 gives the word number
   uint32_t bit = bsel & NBITS_WORD_MOD; // bsel % 32 gives bit number

   asm("{                                       \n\t"
         "bfe.u32            %0,   %1,  %2, 1;  \n\t"      
       "}                                       \n\t"
       : "=r"(c)
       : "r"(x[word]), "r"(bit));
    
   return c;
}

__device__ uint32_t bselMu256(const uint32_t __restrict__ *x, uint32_t bsel)
{
   uint32_t c,i, rc=0; 
   uint32_t word = bsel >> NBITS_WORD_LOG2; // bsel/32 gives the word number
   uint32_t bit = bsel & NBITS_WORD_MOD; // bsel % 32 gives bit number

   #pragma unroll
   for (i=0; i< DEFAULT_U256_BSELM; i++){
     asm("{                                       \n\t"
           "bfe.u32            %0,   %1,  %2, 1;  \n\t"      
         "}                                       \n\t"
         : "=r"(c)
         : "r"(x[NWORDS_256BIT*i+word]), "r"(bit));
    
     rc += (c << i);  
   }

   return rc;
}



/*
  returns number of leading zeros in a 256 bit number
*/
__device__ uint32_t clzMu256(const uint32_t __restrict__ *x)
{
   uint32_t i,j, c, rc, mrc=256; 
  
   #pragma unroll 
   for (i=0; i< DEFAULT_U256_BSELM; i++){
     c = 32;    
     rc = 0;
     for (j=NWORDS_256BIT; j >= 1 && c == 32; j--){
        asm("{                                    \n\t"
            "   clz.b32           %0,%2;          \n\t"
            "   add.u32           %1, %3, %0;     \n\t"      
            "}                   \n\t"
            :"=r"(c), "=r"(rc) : "r"(x[NWORDS_256BIT*i+j-1]), "r"(rc));
     }
     if (rc < mrc) { mrc = rc; }
   }
   return mrc;
}

__device__ uint32_t clzu256(const uint32_t __restrict__ *x)
{
   uint32_t j, c, rc, mrc=255; 
   
   c = 32;    
   rc = 0;
   for (j=NWORDS_256BIT; j >= 1 && c == 32; j--){
      asm("{                                    \n\t"
          "   clz.b32           %0,%2;          \n\t"
          "   add.u32           %1, %3, %0;     \n\t"      
          "}                   \n\t"
          :"=r"(c), "=r"(rc) : "r"(x[j-1]), "r"(rc));
   }

   return rc;
}

__forceinline__ __device__ void almmontinv_step_h(inv_t *table)
{
  // x0 = x0 - x1;
  // x0 = x0 >> 1
  // x4 = x2 + x3;
  // x3 = x3 << 1
  subu256(table->x0, table->x0, table->x1);
  shr1u256(table->x0);
  addu256(table->x4, table->x2,table->x3);
  shl1u256(table->x3);
}

__forceinline__ __device__ void init_invtable(inv_t *data_table, uint32_t *u, uint32_t *v, uint32_t *s, uint32_t *r1)
{
  uint32_t zero[] = {0,0,0,0,0,0,0,0};
  // x0 = x0 - x1;
  // x0 = x0 >> 1
  // x4 = x2 + x3;
  // x3 = x3 << 1
  data_table[0].x0 = u;
  data_table[0].x1 = zero;
  data_table[0].x2 = zero;
  data_table[0].x3 = s;
  data_table[0].x4 = s;

  data_table[1].x0 = v;
  data_table[1].x1 = zero;
  data_table[1].x2 = zero;
  data_table[1].x3 = r1;
  data_table[1].x4 = r1;

  data_table[2].x0 = u;
  data_table[2].x1 = v;
  data_table[2].x2 = r1;
  data_table[2].x3 = s;
  data_table[2].x4 = r1;

  data_table[3].x0 = v;
  data_table[3].x1 = u;
  data_table[3].x2 = s;
  data_table[3].x3 = r1;
  data_table[3].x4 = s;
}

// returns 1 if x - y >= y and x = x-y
// returns 0 if x - y <= y
__forceinline__ __device__ uint32_t subgtu256(uint32_t __restrict__ *x, const uint32_t __restrict__ *y)
{
   uint32_t z[NWORDS_256BIT];
   uint32_t r, flag;

   // 
   subu256(z,x,y);

   asm("clz.b32    %0,%1;\n\t"
       :"=r"(r) : "r"(z[NWORDS_256BIT-1]));
   flag = r > 0;
   if ((r == 32) && eq0u256(z)){
     flag = 0;
   } 
   if (flag){
     movu256(x,z);
   }
   return flag;
}


__forceinline__ __device__ void shflxoru256(uint32_t *d_out, uint32_t *d_in, uint32_t srcLane )
{
    ulonglong4 in, *out;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    in = *(ulonglong4 *)d_in;
    out = (ulonglong4 *)d_out;

    out->x = __shfl_xor_sync(0xffffffff, in.x, srcLane);
    out->y = __shfl_xor_sync(0xffffffff, in.y, srcLane);
    out->z = __shfl_xor_sync(0xffffffff, in.z, srcLane);
    out->w = __shfl_xor_sync(0xffffffff, in.w, srcLane);
}


