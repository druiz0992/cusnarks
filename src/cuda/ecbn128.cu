/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : ecbn128.cu
//
// Date       : 05/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementation of CUDA EC Kernel processing
//
//
//  General  Kernel input vector format:
//     N groups, where each group is made of 1 256 bit scalar number
//    and one Elliptic Point with two 256 bit coordinates
// 
//     X[0], PX[0], PY/Z[0], X[1], PX[1], PY/Z[1],..., X[N-1], PX[N-1], PY/Z[N-1]
//
//  Kernels
// {addec_kernel, doublec_kernel, scmulec_kernel, addec_reduce_kernel, scmulec_reduce_kernel};
//
//   
// ------------------------------------------------------------------

*/

#include <assert.h>
#include <iostream>
#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "rng.h"
#include "cusnarks_kernel.h"
#include "ecbn128.h"
#include "ecbn128_device.h"

using namespace std;

static kernel_cb ecbn128_kernel_callbacks[] = //{addecldr_kernel, doublecldr_kernel, scmulecldr_kernel, madecldr_kernel,
                                               {addecjac_kernel, doublecjac_kernel, scmulecjac_kernel, madecjac_kernel};

ECBN128::ECBN128 (uint32_t len) : CUSnarks( len * ECK_JAC_INDIMS, NWORDS_256BIT * sizeof(uint32_t) * len *  ECK_JAC_INDIMS,
		                            len * ECK_JAC_OUTDIMS,  NWORDS_256BIT * sizeof(uint32_t) * len * ECK_JAC_OUTDIMS, 
                                            ecbn128_kernel_callbacks, 0)
{
}

ECBN128::ECBN128 (uint32_t len, const uint32_t seed) :  CUSnarks(len * ECK_JAC_INDIMS, NWORDS_256BIT * sizeof(uint32_t) * len * ECK_JAC_INDIMS,
				                                 len * ECK_JAC_OUTDIMS, NWORDS_256BIT * sizeof(uint32_t) * len * ECK_JAC_OUTDIMS,
						       ecbn128_kernel_callbacks, seed)
{
}

#if 0
// samples[n] = k[0], Px[0], Py[0],...,k[N-1], Px[N-1], Py[N-1]
void ECBN128::rand(uint32_t *samples, uint32_t n_samples)
{
  // TODO : Implement random EC point. Problem is that 
  // i need to compute random scalar 256 bits (OK), and 
  // and one additional random scalar 256 bits K (OK).
  // EC Point = K * G(Gx,Gy) => For this, I need to 
  // call kernel to compute operation and convert point
  // back to affine coordinates (implement inverse). Instead of doing this, I will
  // generate random points in python for now

  uint32_t *k = new uint32_t[n_samples];
  CUSnarks::rand(k, n_samples);

  CUSnarks::kernelLaunch();

  for (uint32_t i=0; i < 0; i++){
    memcpy(&samples[i*ECK_INDIMS*NWORDS_256BIT + ECP_SCLOFFSET] , k[i], sizeof(uint32_t) * NWORDS_256BIT);
    memcpy(&samples[i*ECK_INDIMS*NWORDS_256BIT + ECP_JAC_INXOFFSET], , sizeof(uint32_t) * NWORDS_256BIT);
    memcpy(&samples[i*ECK_INDIMS*NWORDS_256BIT + ECP_JAC_INYOFFSET], , sizeof(uint32_t) * NWORDS_256BIT);
  }
  delete [] k;
}
#endif
