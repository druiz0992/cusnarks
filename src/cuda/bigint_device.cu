#include "hip/hip_runtime.h"
/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : bigint_kernel.cu
//
// Date       : 05/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementation of biginteger kernel and device functions
// ------------------------------------------------------------------

*/

/*
   addition of two 256 bit number modulo p Z[i] = X[i] + Y[i] (mod p)

   Input vector contains intercalated X, Y and Z numbers (X[0], Y[0], Z[0], X[1], Y[1], Z[1],..
    X[N-1], Y[N-1], Z[N-1]) where X, Y and Z are 256 bit numbers represented as an array of uint32_t
   
*/

#include <stdio.h>

#include "types.h"
#include "bigint_device.h"

/*
    Modular addition kernel

    Arguments :
      in_vector : Input vector of up to N 256 bit elements X[0], X[1], X[2] ... X[N-1].
      out_vector : Results of addition operation Y[0] = X[0] + X[1] mod p, Y[1] = X[2] + X[3] mod p...
      p : 256 bit module in 8 word uint32 array
      len : number of elements in output vector to be xferred. 
          Cannot be greater than half amount reseved during constructor, but not checked
*/
__global__ void addm_kernel(uint32_t *in_vector, uint32_t *p, uint32_t len, uint32_t *out_vector)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    const uint32_t *x;
    const uint32_t *y;
    uint32_t * z;
    uint32_t c = 0;
    uint32_t i;
 
    if(tid >= len) {
      return;
    }

    x = (uint32_t *) &in_vector[tid * 2 * NWORDS_256BIT + XOFFSET];
    y = (uint32_t *) &in_vector[tid * 2 * NWORDS_256BIT + YOFFSET];
    z = (uint32_t *) &out_vector[tid * NWORDS_256BIT];
    
    uaddm256(x, y, z, p);

    return;
}

__global__ void montmul_kernel(uint32_t *in_vector, uint32_t *p, uint32_t len, uint32_t *out_vector)
{
    return;
}

/*
   (z,c) = x * y + a
   z is 2 x uint32_t
   c is carry
*/
__forceinline__ __device__ void umadc32(uint32_t x, uint32_t y, uint32_t a, uint32_t *z, uint32_t *c)
{

   asm(".reg .u64      %prod;                 \n\t"
       ".reg .u64      %sum;                  \n\t"
       ".cvt.u64.u32   %sum   %4;             \n\t"
       "mad.wide.u32   %prod, %2,    %3, %sum;\n\t"
       "cvt.u32.u64    %0,    %prod;          \n\t"
       "shr.u64        %prod, %prod, 32;      \n\t"
       "cvt.u32.u64    %1 %prod;              \n\t"
       : "=r"(z[0]), "=r"(z[1]) 
       : "r"(x), "r"(y), "r"(a));

   // (C,S) = t[0] + a[0] * b[i] -> No carry in
   asm("mad.lo.cc.u32  %0, %3, %4, %5;      \n\t"
       "madc.hi.cc.u32 %1, %3, %4, 0;       \n\t"
       "addc.u32       %tmp, %1, 0;         \n\t"
       "set.lt.u32     %2, %1, %tmp;        \n\t"
       : "=r"(z[0]), "=r"(z[1]), "=r"(c[0]) 
       : "r"(x), "r"(y), "r"(a));
}

__forceinline__ __device__ void ucprop32(uint32_t *x, uint32_t *c)
{
   asm("move.u32    %cin, %3;  \n\t"
       "move.u32    %tmp, %2;  \n\t"
       "add.u32     %0, %cin"
       "set.lt.u32  %1, %0, %tmp;        \n\t"
       : "=r"(x[0]), "=r"(c[0]) 
       : "r"(x[0]), "r"(c[0]));
}
__forceinline__ __device__ void uadd256(const uint32_t *x, const uint32_t *y, uint32_t *z)
{
  // z[i] = x[i] + y[i] for 8x32 bit words
  asm("add.cc.u32        %0, %8, %12;\n\t"              // sum with carry out
      "addc.cc.u32       %1, %9,  %13;\n\t"             // sum with carry in and carry out
      "addc.cc.u32       %2, %10, %14;\n\t"
      "addc.cc.u32       %3, %11, %15;\n\t"
      "addc.cc.u32       %4, %12, %16;\n\t"
      "addc.cc.u32       %5, %13, %17;\n\t"
      "addc.cc.u32       %6, %14, %18;\n\t"
      "addc.u32          %7, %15, %19;\n\t"            // sum with carry in
      : "=r"(z[0]), "=r"(z[1]), "=r"(z[2]), "=r"(z[3]),
        "=r"(z[4]), "=r"(z[5]), "=r"(z[6]), "=r"(z[7])
      : "r"(x[0]), "r"(y[0]), "r"(x[1]), "r"(y[1]),
        "r"(x[2]), "r"(y[2]), "r"(x[3]), "r"(y[3]),
        "r"(x[4]), "r"(y[4]), "r"(x[5]), "r"(y[5]),
        "r"(x[6]), "r"(y[6]), "r"(x[7]), "r"(y[7]));
}

__forceinline__ __device__ void uaddm256(const uint32_t *x, const uint32_t *y, uint32_t *z, const uint32_t *p)
{
  uint32_t do_modf;
  uint32_t z_tmp[NWORDS_256BIT];

  // z[i] = x[i] + y[i] 
  uadd256(x, y, z);

  // z_tmp[i] = z[i] - p[i]
  usub256(z, p, z_tmp);
  
  // do_modf = most significant bit of z_tmp is 1
  asm("bfe.u32	%0, %1, 31, 1;\n\t"              
      : "=r"(do_modf)
      : "r"(z_tmp[7]));

  // if do_modf, return z_tmp. Else, return <
  if (do_modf){
     asm("mov.u32     %0,  %8;\n\t"
         "mov.u32     %1,  %9;\n\t"
         "mov.u32     %2,  %10;\n\t"
         "mov.u32     %3,  %11;\n\t"
         "mov.u32     %4,  %12;\n\t"
         "mov.u32     %5,  %13;\n\t"
         "mov.u32     %6,  %14;\n\t"
         "mov.u32     %7,  %15;\n\t"
      : "=r"(z[0]), "=r"(z[1]), "=r"(z[2]), "=r"(z[3]),
        "=r"(z[4]), "=r"(z[5]), "=r"(z[6]), "=r"(z[7])
      : "r"(z_tmp[0]), "r"(z_tmp[1]), "r"(z_tmp[2]), "r"(z_tmp[3]),
        "r"(z_tmp[4]), "r"(z_tmp[5]), "r"(z_tmp[6]), "r"(z_tmp[7]));
  }    
  
}

__forceinline__ __device__ void usub256(const uint32_t *x, const uint32_t *y, uint32_t *z)
{
  // z[i] = x[i] - y[i] for 8x32 bit words
  asm("sub.cc.u32        %0, %8, %12;\n\t"              // sub with borrow out
      "subc.cc.u32       %1, %9,  %13;\n\t"             // sub with borrow out and borrow in
      "subc.cc.u32       %2, %10, %14;\n\t"
      "subc.cc.u32       %3, %11, %15;\n\t"
      "subc.cc.u32       %4, %12, %16;\n\t"
      "subc.cc.u32       %5, %13, %17;\n\t"
      "subc.cc.u32       %6, %14, %18;\n\t"
      "subc.u32          %7, %15, %19;\n\t"            // sum with carry in
      : "=r"(z[0]), "=r"(z[1]), "=r"(z[2]), "=r"(z[3]),
        "=r"(z[4]), "=r"(z[5]), "=r"(z[6]), "=r"(z[7])
      : "r"(x[0]), "r"(y[0]), "r"(x[1]), "r"(y[1]),
        "r"(x[2]), "r"(y[2]), "r"(x[3]), "r"(y[3]),
        "r"(x[4]), "r"(y[4]), "r"(x[5]), "r"(y[5]),
        "r"(x[6]), "r"(y[6]), "r"(x[7]), "r"(y[7]));

}


/*
__global__ void ciosV2(KernelArray<unsigned int>d_a1, KernelArray<unsigned int>d_b1, KernelArray<unsigned int>d_ans, KernelArray<unsigned int>d_n, KernelArray<unsigned int>d_n1, int d_s, int blkSize)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int t[33] = { 0 };
    unsigned long long temp;
    __shared__ unsigned int shared_n[32], shared_n1[32], shared_s;
    shared_s = d_s;
    for (int i = 0; i < shared_s; i++){
        shared_n[i] = d_n._array[i];
        shared_n1[i] = d_n1._array[i];
    }
    __syncthreads();
    for (int i = 0; i < shared_s; i++){
        unsigned long long c = 0;
        for (int j = 0; j < shared_s; j++){
            temp = t[j] + (unsigned long long)d_a1._array[j * (1024 *
            blkSize) + idx] * (unsigned long long)d_b1._array[i * (1024 * blkSize) + idx] +
            c;
            t[j] = temp & 4294967295;
            c = temp >> 32;
        }
        temp = (unsigned long long)t[shared_s] + c;
        t[shared_s] = temp & 4294967295;
        t[shared_s + 1] = temp >> 32;
        unsigned long long m = ((unsigned long long)t[0] * (unsigned long
        long)shared_n1[0]) & 4294967295;
        temp = (unsigned long long)t[0] + m*(unsigned long
        long)shared_n[0];
        c = temp >> 32;
        for (int j = 1; j < shared_s; j++){
            temp = (unsigned long long)t[j] + m*(unsigned long
            long)shared_n[j] + c;
            t[j - 1] = temp & 4294967295;
            c = temp >> 32;
        }
        temp = (unsigned long long)t[shared_s] + c;
        t[shared_s - 1] = temp & 4294967295;
        c = temp >> 32;
        t[shared_s] = t[shared_s + 1] + c;
    }
    unsigned int u[33];
    for (int j = 0; j < shared_s + 1; j++){
        u[j] = t[j];
    }
    int b = 0;
    long long sub;
    for (int i = 0; i < shared_s; i++){
        sub = (long long)u[i] - shared_n[i] - b;
        if (sub < 0){
            t[i] = sub + 4294967296;
            b = 1;
        }
        else{
            t[i] = sub;
            b = 0;
        }
    }
    sub = (long long)u[shared_s] - b;
    u[shared_s] = sub;
    if (sub >= 0){
        int counter = 0;
        for (int i = 0; i < 32; i++){
            d_ans._array[i * 1024 * blkSize + idx] = t[counter++];
        }
    }
    else{
        int counter = 0;
        for (int i = 0; i < 32; i++){
            d_ans._array[i * 1024 * blkSize + idx] = u[counter++];
        }
    }
}

__global__ void BigInt_MontMul(KernelArray<unsigned int>d_a1, KernelArray<unsigned int>d_b1, KernelArray<unsigned int>d_ans, KernelArray<unsigned int>d_n, KernelArray<unsigned int>d_n1, int d_s, int blkSize)
{
    for (int i = 0; i < shared_s; i++){
        unsigned long long c = 0;
        for (int j = 0; j < shared_s; j++){
            temp = t[j] + (unsigned long long)d_a1._array[j * (1024 *
            blkSize) + idx] * (unsigned long long)d_b1._array[i * (1024 * blkSize) + idx] +
            c;
            t[j] = temp & 4294967295;
            c = temp >> 32;
        }
        temp = (unsigned long long)t[shared_s] + c;
        t[shared_s] = temp & 4294967295;
        t[shared_s + 1] = temp >> 32;
        unsigned long long m = ((unsigned long long)t[0] * (unsigned long
        long)shared_n1[0]) & 4294967295;
        temp = (unsigned long long)t[0] + m*(unsigned long
        long)shared_n[0];
        c = temp >> 32;
        for (int j = 1; j < shared_s; j++){
            temp = (unsigned long long)t[j] + m*(unsigned long
            long)shared_n[j] + c;
            t[j - 1] = temp & 4294967295;
            c = temp >> 32;
        }
        temp = (unsigned long long)t[shared_s] + c;
        t[shared_s - 1] = temp & 4294967295;
        c = temp >> 32;
        t[shared_s] = t[shared_s + 1] + c;
    }
    unsigned int u[33];
    for (int j = 0; j < shared_s + 1; j++){
        u[j] = t[j];
    }
    int b = 0;
    long long sub;
    for (int i = 0; i < shared_s; i++){
        sub = (long long)u[i] - shared_n[i] - b;
        if (sub < 0){
            t[i] = sub + 4294967296;
            b = 1;
        }
        else{
            t[i] = sub;
            b = 0;
        }
    }
    sub = (long long)u[shared_s] - b;
    u[shared_s] = sub;
    if (sub >= 0){
        int counter = 0;
        for (int i = 0; i < 32; i++){
            d_ans._array[i * 1024 * blkSize + idx] = t[counter++];
        }
    }
    else{
        int counter = 0;
        for (int i = 0; i < 32; i++){
            d_ans._array[i * 1024 * blkSize + idx] = u[counter++];
        }
    }
}

*/
