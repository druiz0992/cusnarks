#include "hip/hip_runtime.h"
/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : ecbn128_device.cu
//
// Date       : 12/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementatoin of EC Cuda functionality
// 
// NOTE : EC Points do not require to be premoded premod. They shoould
//  already be < than prime
// ------------------------------------------------------------------

*/

#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "ecbn128_device.h"
#include "u256_device.h"

__global__ void addecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1, *x2, *xr, *z1, *z2, *zr;
 
    if(tid >= params->in_length/6) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * 2 * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];
    x2 = (uint32_t *) &in_vector[(tid * 2 + 1) * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];
    
    addecldr(xr, x1, x2, x1, params->midx);

    return;
}
__global__ void doublecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1,*xr, *z1,*zr;
 
    if(tid >= params->in_length/3) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];
    
    doublecldr(xr, x1, params->midx);

  return;
}
__global__ void scmulecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *x1, *z1, *scl, *xr, *zr;
 
   if(tid >= params->in_length/3) {
     return;
   }

   x1  = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];
   scl = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_SCLOFFSET];

   xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];

   
   ldrstep(xr, x1, scl,  params->midx);

   return;
}

__global__ void madecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
  return;
}


/* 
  in_vector : k[0], px[0], py[0], k[1], px[1], py[1],...  Input EC points in Affine coordinates
  out vecto : px[0], py[0], pz[0], px[1], py[1],pz[1],...              Output EC points in Jacobian coordinates
*/
__global__ void addecjac_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1, *x2, *xr, *z1, *z2, *zr;
 
    if(tid >= params->in_length/6) {
      return;
    }
    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    x1 = (uint32_t *) &in_vector[tid * 2 * ECK_JAC_INOFFSET + ECP_JAC_INXOFFSET];
    x2 = (uint32_t *) &in_vector[(tid * 2 + 1) * ECK_JAC_INOFFSET + ECP_JAC_INXOFFSET];
    xr = (uint32_t *) &out_vector[tid * ECK_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET];
    
    addecjac(xr, x1, x2, params->midx);

    return;

}

__global__ void doublecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1,*xr, *z1,*zr;
 
    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/3) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * ECK_JAC_INOFFSET + ECP_JAC_INXOFFSET];
    xr = (uint32_t *) &out_vector[tid * ECK_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET];
    
    if (params->premod){
      modu256(x1,x1, params->midx);
      modu256(&x1[NWORDS_256BIT],&x1[NWORDS_256BIT], params->midx);
    }

    doublecjac(xr, x1, params->midx);

    return;
}

__global__ void scmulecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *x1, *z1, *scl, *xr, *zr;
 
   if(tid >= params->in_length/3) {
     return;
   }

   x1  = (uint32_t *) &in_vector[tid * ECK_JAC_INOFFSET + ECP_JAC_INXOFFSET];
   scl = (uint32_t *) &in_vector[tid * ECK_JAC_INOFFSET + ECP_SCLOFFSET];

   xr = (uint32_t *) &out_vector[tid * ECK_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET];
   
   scmulecjac(xr, x1, scl,  params->midx);

   return;
}

__global__ void madecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
  return;
}
    
__forceinline__ __device__
 void addecldr(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, 
            const uint32_t __restrict__ *x2, const uint32_t __restrict__ *xp, mod_t midx)
{
   // Xr = -4*b Z1 * Z2 * (X1 * Z2 + X2 * Z1) + (X1 * X2)^2 
   // Zr = xp * (X1 * Z2 - X2 * Z1)^2

   // 7 M, 2 SQ, 3 ADD
   const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT];
   const uint32_t __restrict__ *z2 = &x2[NWORDS_256BIT];
   uint32_t __restrict__ *zr =&zr[NWORDS_256BIT];

   uint32_t tmp1[NWORDS_256BIT];
   uint32_t __restrict__ *_4b = misc_const_ct[midx]._4b;
   

   mulmontu256(tmp1, x2  , z1  , midx);      
   mulmontu256(xr  , x1  , z2  , midx);      
   submu256(   zr  , xr  , tmp1, midx);
   addmu256(   tmp1, tmp1, xr  , midx);
   mulmontu256(tmp1, tmp1, z2  , midx);    
   mulmontu256(tmp1, tmp1, z1  , midx);    
   mulmontu256(xr  , x1  , x2  , midx);      
   mulmontu256(xr  , xr  , xr  , midx);          // TODO : implement squaring
   mulmontu256(tmp1, tmp1,_4b  , midx);
   submu256(   xr,   tmp1, xr  , midx);
   mulmontu256(zr,   zr  , zr  , midx);     // TODO : implement squaring
   mulmontu256(zr,   zr  , xp  , midx);   

  return;
}

__forceinline__ __device__
 void doublecldr(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, mod_t midx)
{
  // Xr = X1^4 - 8*b * X1*Z1^3
  // Zr = 4*Z1 * (X1^3 + b*Z1^3) 

  // 7 M, 3 SQ, 2 Add
  const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT];
  uint32_t __restrict__ *zr =&zr[NWORDS_256BIT];

  uint32_t tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT];
  uint32_t __restrict__ *_4 = misc_const_ct[midx]._4;
  uint32_t __restrict__ *b = ecbn128_params_ct[midx].b;
  uint32_t __restrict__ *_8b = misc_const_ct[midx]._8b;

  mulmontu256(xr,  z1,   z1,   midx);      // TODO squaring
  mulmontu256(zr,  xr,   z1,   midx);      // Zr = Z1^3
  mulmontu256(xr,  zr,   x1,   midx);      
  mulmontu256(xr,  xr,  _8b,   midx);      // Xr = 8b * X1 * Z1^3
  mulmontu256(tmp1, x1,   x1,   midx);      // TODO squaring
  mulmontu256(tmp2, tmp1,  tmp1,  midx);     // TODO squaring
  submu256(   xr,  tmp2, xr,   midx);

  mulmontu256(zr,  zr,   b,    midx);      // Zr = b*Z1^3
  mulmontu256(tmp1, tmp1,  x1,   midx);     
  addmu256(   zr, tmp1,   zr,   midx);
  mulmontu256(zr, zr,   _4,    midx);
  mulmontu256(zr, zr,   z1,    midx); 

  return;
}

// NOTE : EC points are in affine coordinates => Pz = 1 (in montgomery someting else)
// NOTE : EC points in montgomery, scl normal 
__forceinline__ __device__
 void ldrstep(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, uint32_t *scl, mod_t midx)
{
  uint32_t b0, idxn, idxp;
  uint32_t __restrict__ *_1 = misc_const_ct[midx]._1;

  uint32_t __restrict__ R[4*NWORDS_256BIT];

  //R[] = {[1,0],[X,1]} 
  memcpy(R, _1, NWORDS_256BIT * sizeof(uint32_t));
  memcpy(&R[3 * NWORDS_256BIT], _1, NWORDS_256BIT * sizeof(uint32_t));
  memcpy(&R[2 * NWORDS_256BIT], x1, NWORDS_256BIT * sizeof(uint32_t));

  while (!eq0u256(scl)){
     b0 = shr1u256(scl);
     idxn = ~b0 * 2 * NWORDS_256BIT;
     idxp =  b0 * 2 * NWORDS_256BIT;
     /*
     if (b0) { R0 = R0 + R1; R1 = R1 + R1;
     else {    R1 = R0 + R1; R0 = R0 + R0}
     */
     addecldr(&R[idxn], &R[idxn], &R[idxp], x1, midx);
     doublecldr(&R[idxp], &R[idxp], midx);
  }
   // TODO
   // Retrieve y(P) . Not sure if i need to convert to affine now. If I don't,
   // then I have three coordinates and it doesn't fit in my allocated space
   //
   // P = (x1, y1) , Q = (x2, y2), P-Q = (x,y)
   // Q = k P => x(R0) = X(Q), x(R1) = X(P-Q)
   //
   // y(P) = y1 = [2b + (a + x * x1) * (x + x1) - x2(x - x1) ^ 2] / (2*y)

  memcpy(xr,R,2 * NWORDS_256BIT * sizeof(uint32_t));

  return;
}

/*
  EC point addition
  
  Algorithm (https://en.wikibooks.org/wiki/Cryptography/Prime_Curve/Jacobian_Coordinates):
  IN : P1(X1,Y1,Z1), P2(X2,Y2,Z2)
  OUT: P3(X3,Y3,Z3)

    U1 = X1*Z2^2
    U2 = X2*Z1^2
    S1 = Y1*Z2^3
    S2 = Y2*Z1^3
    if (U1 == U2)
      if (S1 != S2)
        return POINT_AT_INFINITY
      else 
        return POINT_DOUBLE(X1, Y1, Z1)
    H = U2 - U1
    R = S2 - S1
    X3 = R^2 - H^3 - 2*U1*H^2
    Y3 = R*(U1*H^2 - X3) - S1*H^3
    Z3 = H*Z1*Z2
    return (X3, Y3, Z3)
*/
__forceinline__ __device__
 void addecjac(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, const uint32_t *x2, mod_t midx)
{
  const uint32_t __restrict__ *y1 = &x1[NWORDS_256BIT];
  const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT*2];
  const uint32_t __restrict__ *y2 = &x2[NWORDS_256BIT];
  const uint32_t __restrict__ *z2 = &x2[NWORDS_256BIT*2];
  uint32_t __restrict__ *yr = &xr[NWORDS_256BIT];
  uint32_t __restrict__ *zr = &xr[NWORDS_256BIT*2];
  uint32_t __restrict__ *_inf = misc_const_ct[midx]._inf;
  uint32_t __restrict__ *_2 = misc_const_ct[midx]._2;
 
  uint32_t __restrict__ tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT], tmp3[NWORDS_256BIT], tmp4[NWORDS_256BIT];

  mulmontu256(xr, z1, z1, midx);  // xr = z1sq 
  mulmontu256(zr, xr, x2, midx);  // zr = u2 = x2 * z1sq
  mulmontu256(xr, xr, z1, midx);  // xr = z1cube
  mulmontu256(xr, xr, y2, midx);  // xr = s2 = z1cube * y2
  mulmontu256(yr, z2, z2, midx);  // yr = z2sq
  mulmontu256(tmp1, x1, yr, midx);  // tmp1 = u1 = x1 * z2sq
  mulmontu256(yr, yr, z2, midx);  // yr = z2cube
  mulmontu256(yr, yr, y1, midx);  // yr = s1 = z2cube * y1

  // TODO Check if I can call add to compute x + x (instead of double)
  //  if not, I should call double below. I don't want to to avoid warp divergnce
  if (equ256((const uint32_t *)tmp1, (const uint32_t *)zr) &&   // u1 == u2
       !equ256( (const uint32_t *) yr, (const uint32_t *) xr)){  // s1 != s2
          memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
	  return;  //  if U1 == U2 and S1 == S2 => P1 == P2 (call double)
  }

  submu256(tmp2, zr, tmp1, midx);     // H = tmp2 = u2 - u1
  mulmontu256(zr, z1, z2, midx);      // zr = z1 * z2
  mulmontu256(zr, zr, tmp2, midx);       // zr = z1 * z2  * h

  mulmontu256(tmp3, tmp2, tmp2, midx);     // Hsq = tmp3 = H * H 
  mulmontu256(tmp2, tmp3, tmp2, midx);     // Hcube = tmp2 = Hsq * H 
  mulmontu256(tmp1, tmp1, tmp3, midx);     // tmp1 = u1 * Hsq

  submu256(tmp3, xr, yr, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, xr=free, yr=s1, zr=zr
  mulmontu256(yr, yr, tmp2, midx);     // yr = Hcube * s1
  mulmontu256(xr, tmp3, tmp3, midx);     // xr = R * R
  submu256(xr, xr, tmp2, midx);        // xr = x3= (R*R)-Hcube, yr = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  mulmontu256(tmp4, tmp1, _2, midx);     // tmp4 = u1*hsq *_2
  submu256(xr, xr, tmp4, midx);               // x3 = xr
  submu256(tmp1, tmp1, xr, midx);       // tmp1 = u1*hs1 - x3
  mulmontu256(tmp1, tmp1, tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)
  submu256(yr, tmp1, yr, midx);
}

/*
  EC point addition
  
  Algorithm (https://en.wikibooks.org/wiki/Cryptography/Prime_Curve/Jacobian_Coordinates):
  IN : P1(X1,Y1,Z1)
  OUT: P'(X',Y',Z')

   if (Y == 0)
      return POINT_AT_INFINITY
   S = 4*X*Y^2
   M = 3*X^2 + a*Z^4
   X' = M^2 - 2*S
   Y' = M*(S - X') - 8*Y^4
   Z' = 2*Y*Z
   return (X', Y', Z')
*/
__forceinline__ __device__
 void doublecjac(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, mod_t midx)
{
  const uint32_t __restrict__ *y1 = &x1[NWORDS_256BIT];
  const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT*2];
  uint32_t __restrict__ *yr = &xr[NWORDS_256BIT];
  uint32_t __restrict__ *zr = &xr[NWORDS_256BIT*2];
  uint32_t __restrict__ *_inf = misc_const_ct[midx]._inf;
  uint32_t __restrict__ *_8 = misc_const_ct[midx]._8;
  uint32_t __restrict__ *_4 = misc_const_ct[midx]._4;
  uint32_t __restrict__ *_3 = misc_const_ct[midx]._3;
  uint32_t __restrict__ *_2 = misc_const_ct[midx]._2;
 
  uint32_t __restrict__ tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT];

  if (eq0u256(y1)){ 
      memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
      return;  
  }
  mulmontu256(zr, y1, y1, midx);  // zr = ysq
  mulmontu256(yr, zr, zr, midx);  // yr = ysqsq
  mulmontu256(yr, yr, _8, midx);  // yr = ysqsq *_8
  mulmontu256(zr, zr, x1, midx);  // S = zr = x * ysq
  mulmontu256(zr, zr, _4, midx);  // S = zr = S * _4

  mulmontu256(xr, x1, x1, midx);  // M1 = xr = x * x
  mulmontu256(tmp1, xr, _3, midx);  // M = tmp1 = M1 * _3
  mulmontu256(xr, tmp1, tmp1, midx);  // X3 = xr = M * M,  yr = Ysqsq * _8, zr = S; tmp1 = M
  mulmontu256(tmp2, zr, _2, midx);   // tmp2 = S * _2
  submu256(xr, xr, tmp2, midx);      // X3 = xr; yr = Ysqsq * _8, zr = S, tmp1 = M, 
  submu256(tmp2, zr, xr, midx);   //  tmp2 = S - X3
  mulmontu256(tmp2, tmp2, tmp1, midx); // tmp2 = M * (S - X3)
  submu256(yr, tmp2, yr, midx);
  mulmontu256(zr, y1, x1, midx);
  mulmontu256(zr, zr, _2, midx);

}

__forceinline__ __device__
 void scmulecjac(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, uint32_t *scl, mod_t midx)
{
  uint32_t b0;

  uint32_t __restrict__ N[2*NWORDS_256BIT]; // N = P
  uint32_t __restrict__ *Q = xr; // Q = 0
  uint32_t __restrict__ *_inf = misc_const_ct[midx]._inf;

  memcpy(N, x1, 2 * NWORDS_256BIT * sizeof(uint32_t));
  memcpy(Q, _inf, 2* NWORDS_256BIT * sizeof(uint32_t));

  while (!eq0u256(scl)){
     b0 = shr1u256(scl);
     /*
     if (b0) { Q = Q + N }
     N = N + N
     */
     if (b0) {
       addecjac(Q, Q, N, midx);
     }
     doublecjac(N,N, midx);
  }

  return;
}
