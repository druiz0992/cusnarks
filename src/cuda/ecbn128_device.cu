#include "hip/hip_runtime.h"
/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : ecbn128_device.cu
//
// Date       : 12/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementatoin of EC Cuda functionality
// 
// NOTE : EC Points do not require to be premoded premod. They shoould
//  already be < than prime
// ------------------------------------------------------------------

*/

#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "log.h"
#include "utils_device.h"
#include "u256_device.h"
#include "z1_device.h"
#include "z2_device.h"
#include "ecbn128_device.h"

/* 
  in_vector : k[0], px[0], py[0], k[1], px[1], py[1],...  Input EC points in Affine coordinates
  out vecto : px[0], py[0], pz[0], px[1], py[1],pz[1],...              Output EC points in Jacobian coordinates
*/
__global__ void addecjacaff_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //Z1_t x1, x2, xr;

    if(tid >= params->in_length/4) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z1_t x1(&in_vector[tid * 2 * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t x2(&in_vector[(tid * 2 + 1) * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjacaff<Z1_t, uint256_t>(&xr, &x1, &x2, params->midx);

    return;

}

__global__ void addec2jacaff_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid >= params->in_length/8) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z2_t x1(&in_vector[tid * 2 * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t x2(&in_vector[(tid * 2 + 1) * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjacaff<Z2_t, uint512_t>(&xr, &x1, &x2, params->midx);

    return;

}

__global__ void addecjac_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //Z1_t x1, x2, xr;

    if(tid >= params->in_length/6) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z1_t x1(&in_vector[tid * 2 * ECP_JAC_OUTOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t x2(&in_vector[(tid * 2 + 1) * ECP_JAC_OUTOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjac<Z1_t, uint256_t>(&xr,0, &x1,0, &x2,0, params->midx);

    return;

}

__global__ void addec2jac_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //Z1_t x1, x2, xr;

    if(tid >= params->in_length/12) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z2_t x1(&in_vector[tid * 2 * ECP2_JAC_OUTOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t x2(&in_vector[(tid * 2 + 1) * ECP2_JAC_OUTOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjac<Z2_t, uint512_t>(&xr,0, &x1,0, &x2,0, params->midx);

    return;

}
__global__ void doublecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/3) {
      return;
    }

    Z1_t x1(&in_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
    
    doublecjac<Z1_t, uint256_t>(&xr, &x1, params->midx);

    return;
}

__global__ void doublec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/6) {
      return;
    }

    Z2_t x1(&in_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
    
    doublecjac<Z2_t, uint512_t>(&xr, &x1, params->midx);

    return;
}
__global__ void doublecjacaff_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/2) {
      return;
    }

    Z1_t x1(&in_vector[tid * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
    
    doublecjacaff<Z1_t, uint256_t>(&xr, &x1, params->midx);

    return;
}

__global__ void doublec2jacaff_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/4) {
      return;
    }

    Z2_t x1(&in_vector[tid * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
    
    doublecjacaff<Z2_t, uint512_t>(&xr, &x1, params->midx);

    return;
}


__global__ void scmulecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= params->in_length/3) {
     return;
   }

   scl = (uint32_t *) &in_vector[tid * NWORDS_256BIT + ECP_SCLOFFSET];
   Z1_t x1(&in_vector[ params->in_length/3 * NWORDS_256BIT+ tid * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
   Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z1_t, uint256_t>(&xr,0, &x1,0, scl,  params->midx);

   return;
}

__global__ void sc1mulecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= (params->in_length-1)/2) {
     return;
   }

   scl = (uint32_t *) &in_vector[ECP_SCLOFFSET];
   Z1_t x1(&in_vector[NWORDS_256BIT + tid * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
   Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z1_t, uint256_t>(&xr,0, &x1,0, scl,  params->midx);

   return;
}

__global__ void scmulec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= params->in_length/5) {
     return;
   }

   scl = (uint32_t *) &in_vector[tid * NWORDS_256BIT + ECP_SCLOFFSET];
   Z2_t x1(&in_vector[ params->in_length/5 * NWORDS_256BIT+ tid * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
   Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z2_t, uint512_t>(&xr,0, &x1,0, scl,  params->midx);

   return;
}

__global__ void sc1mulec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= (params->in_length-1)/4) {
     return;
   }

   scl = (uint32_t *) &in_vector[ECP_SCLOFFSET];
   Z2_t x1(&in_vector[NWORDS_256BIT+ tid * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
   Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z2_t, uint512_t>(&xr,0, &x1,0, scl,  params->midx);

   return;
}

__global__ void madecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    uint32_t debug_idx = 0;

    extern __shared__ uint32_t smem[];
    Z1_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl;
   
    if(idx >= params->in_length/params->stride) {
      return;
    }
    logInfoTid(idx,"Min Padding : %d\n",params->padding_idx);
    logInfoTid(idx,"Max Padding : %d\n",params->in_length/ECP_JAC_OUTDIMS);
    if (params->padding_idx){
       uint32_t padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&in_vector[idx * ECP_JAC_INOFFSET],padding);
          movu256(&in_vector[idx * ECP_JAC_INOFFSET + NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP_JAC_INOFFSET + 2*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }

    Z1_t xo, xr;
    if (params->premul){
      xo.assign(&in_vector[params->in_length/3 * NWORDS_256BIT + idx  * (params->stride-1) * NWORDS_256BIT + ECP_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = (uint32_t *) &in_vector[idx * params->stride/3 *  NWORDS_256BIT];
      logInfoTid(idx,"LE : %d\n",params->in_length);
      logInfoTid(idx,"InVO : %d\n",(params->stride-1) * NWORDS_256BIT);
      logInfoTid(idx,"SclVO : %d\n",params->in_length/3* 2 * NWORDS_256BIT);
    } else {
      xo.assign(&in_vector[idx  * (params->stride) * NWORDS_256BIT + ECP_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = NULL;
      logInfoTid(idx,"LE : %d\n",params->in_length);
      logInfoTid(idx,"InVO : %d\n",(params->stride) * NWORDS_256BIT);
      logInfoTid(idx,"SclVO : %d\n",params->in_length/3* 2 *NWORDS_256BIT);
    }
    xr.assign(&in_vector[blockIdx.x * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);  // 
    if (gridDim.x == 1){
      xr.assign(out_vector);
    }

    madecjac<Z1_t, uint256_t>(&xr, &xo, scl, &zsmem, params);
}

__global__ void madec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    uint32_t debug_idx = 0;

    extern __shared__ uint32_t smem[];
    Z2_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl;
   
    if(idx >= params->in_length/params->stride) {
      return;
    }
    logInfoTid(idx,"Min Padding : %d\n",params->padding_idx);
    logInfoTid(idx,"Max Padding : %d\n",params->in_length/ECP2_JAC_OUTDIMS);
    if (params->padding_idx){
       uint32_t padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP2_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET+ NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 2*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 3*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 4*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 5*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }

    Z2_t xo, xr;
    if (params->premul){
      xo.assign(&in_vector[params->in_length/5 * NWORDS_256BIT + idx  * params->stride * NWORDS_256BIT + ECP2_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = (uint32_t *) &in_vector[idx * params->stride/5 *  NWORDS_256BIT];
      logInfoTid(idx,"LE : %d\n",params->in_length);
      logInfoTid(idx,"InVO : %d\n",(params->stride) * NWORDS_256BIT);
      logInfoTid(idx,"SclVO : %d\n",params->in_length/5* NWORDS_256BIT);
    } else {
      xo.assign(&in_vector[idx  * (params->stride) * NWORDS_256BIT + ECP2_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = NULL;
      logInfoTid(idx,"LE : %d\n",params->in_length);
      logInfoTid(idx,"InVO : %d\n",(params->stride) * NWORDS_256BIT);
      logInfoTid(idx,"SclVO : %d\n",params->in_length/6 *NWORDS_256BIT);
    }
    xr.assign(&in_vector[blockIdx.x * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);  // 
    if (gridDim.x == 1){
      xr.assign(out_vector);
    }

    madecjac<Z2_t, uint512_t>(&xr, &xo, scl, &zsmem, params);
}
__global__ void madecjac_shfl_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t poffset = 0;

    extern __shared__ uint32_t smem[];
    Z1_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl = NULL;
  
    if(idx >= params->in_length/params->stride) {
      return;
    }
    logInfoTid(idx,"Min Padding : %d\n",params->padding_idx);
    logInfoTid(idx,"Max Padding : %d\n",params->in_length/ECP_JAC_OUTDIMS);
    logInfoTid(idx,"OUt : %d\n",params->in_length/params->stride);
    if (params->padding_idx){
       uint32_t __align__(16) padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&in_vector[idx * ECP_JAC_OUTOFFSET],padding);
          movu256(&in_vector[idx * ECP_JAC_OUTOFFSET + NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }

    Z1_t xo;
    if (params->premul){
      scl = (uint32_t *) &in_vector[idx *  NWORDS_256BIT];
      poffset = params->in_length/3 * NWORDS_256BIT;
      xo.assign(&in_vector[poffset + idx * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]); // 0 .. N-1
      logInfoBigNumberTid(idx,1,"SCL in \n",scl);
    } else {
      xo.assign(&out_vector[idx * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]); // 0 .. N-1
    }

    Z1_t xr(&out_vector[blockIdx.x * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);  // 
  
    //if (gridDim.x == 1){
      //xr.assign(out_vector);
    //} 

    logInfoBigNumberTid(idx,2,"X in \n",&xo);
    //logInfoBigNumberTid(idx,32*3,"In \n",in_vector);
    madecjac_shfl<Z1_t, uint256_t>(&xr, &xo, scl, &zsmem, params);
}

__global__ void madec2jac_shfl_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t poffset = 0;

    extern __shared__ uint32_t smem[];
    Z2_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl = NULL;
  
    if(idx >= params->in_length/params->stride) {
      return;
    }
    logInfoTid(idx,"Min Padding : %d\n",params->padding_idx);
    logInfoTid(idx,"OUt : %d\n",params->in_length/params->stride);
    if (params->padding_idx){
       uint32_t padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP2_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 2*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 3*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 4*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 5*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }

    Z2_t xo;
    if (params->premul){
      scl = (uint32_t *) &in_vector[idx *  NWORDS_256BIT];
      poffset = params->in_length/5 * NWORDS_256BIT;
      xo.assign(&in_vector[poffset + idx * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]); // 0 .. N-1
    } else {
      xo.assign(&in_vector[idx * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]); // 0 .. N-1
    }

    Z2_t xr(&in_vector[blockIdx.x * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);  // 
  
    if (gridDim.x == 1){
      xr.assign(out_vector);
    } 

    logInfoBigNumberTid(idx,2,"X in \n",&xo);
    //logInfoBigNumberTid(idx,32*3,"In \n",in_vector);
    madecjac_shfl<Z2_t, uint512_t>(&xr, &xo, scl, &zsmem, params);
}


template<typename T1, typename T2>
__forceinline__ __device__ void madecjac(T1 *xr, T1 *xo, uint32_t *scl, T1 *smem_ptr, kernel_params_t *params)
{
    uint32_t i;
    uint32_t ndbg = T1::getN();
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    logInfoTid(idx,"stride :%d\n",params->stride/ECP_JAC_OUTDIMS);
    // scalar multipliation
    if (params->premul){
        #pragma unroll
        for (i =0; i < params->stride/ECP_JAC_OUTDIMS; i++){
          /*
          logInfoBigNumberTid(idx,1,"scl :\n",&scl[i*ECP_JAC_INOFFSET]);
          logInfoBigNumberTid(idx,ndbg,"Xin[x]:\n",xos(i*ECP_JAC_INDIMS));
          logInfoBigNumberTid(idx,,"Xin[y]:\n",&xi[i*ECP_JAC_INOFFSET + NWORDS_256BIT]);
          */

          scmulecjac<T1, T2>(xr,i*ECP_JAC_OUTDIMS, xo, i*ECP_JAC_INDIMS, &scl[i*NWORDS_256BIT],  params->midx);
          

          /*
          logInfoBigNumberTid(idx,1,"Xout[x]:\n",&xo[i*ECP_JAC_OUTOFFSET]);
          logInfoBigNumberTid(idx,1,"Xout[y]:\n",&xo[i*ECP_JAC_OUTOFFSET + NWORDS_256BIT]);
          logInfoBigNumberTid(idx,1,"Xout[z]:\n",&xo[i*ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
          */
        }
    }
   
    if (params->stride/ECP_JAC_OUTDIMS > 1){ 
      addecjac<T1,T2>(smem_ptr,tid*ECP_JAC_OUTDIMS, xr,0, xr,ECP_JAC_OUTDIMS, params->midx);
      /*
      logInfoBigNumberTid(idx,1,"smem[X]\n",smem_ptr);
      logInfoBigNumberTid(idx,1,"smem[Y]\n",&smem_ptr[NWORDS_256BIT]);
      logInfoBigNumberTid(idx,1,"smem[Z]\n",&smem_ptr[2*NWORDS_256BIT]);
      */

      #pragma unroll
      for (i =0; i < params->stride/ECP_JAC_OUTDIMS-2; i++){
        addecjac<T1,T2>(smem_ptr,tid*ECP_JAC_OUTDIMS, smem_ptr, 0,xr, (i+2)*ECP_JAC_OUTDIMS, params->midx);
        /*
        logInfoBigNumberTid(idx,1,"smem[X]\n",smem_ptr);
        logInfoBigNumberTid(idx,1,"smem[Y]\n",&smem_ptr[NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[Z]\n",&smem_ptr[2*NWORDS_256BIT]);
        */
      }
      __syncthreads();
    }
  
    //logDebugBigNumberTid(idx,1,"smem[i]\n",smem_ptr);
    // reduction global mem
    if (blockDim.x >= 1024 && tid < 512){
      /*
      logInfoBigNumberTid(idx,1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(idx,1,"+smem[512]\n",&smem[(tid+512)*NWORDS_256BIT]);
      */
      
      addecjac<T1,T2>(smem_ptr, tid*ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+512)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(idx,1,"smem[0]\n",smem_ptr);
      */
    }
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256){
      /*
      logInfoBigNumberTid(idx,1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(idx,1,"+smem[256]\n",&smem[(tid+256)*NWORDS_256BIT]);
      */
      addecjac<T1,T2>(smem_ptr, tid * ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+256)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(idx,1,"smem[=256]\n",smem_ptr);
      */
    }
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128){
      /*
      logInfoBigNumberTid(idx,1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(idx,1,"+smem[128]\n",&smem[(tid+128)*NWORDS_256BIT]);
      */
      addecjac<T1,T2>(smem_ptr, tid * ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+128)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(idx,1,"smem[=128+0]\n",smem_ptr);
      */
    }
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64){
      /*
      logInfoBigNumberTid(idx,1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(idx,1,"+smem[64]\n",&smem[(tid+64)*NWORDS_256BIT]);
      */
      addecjac<T1,T2>(smem_ptr, tid * ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+64)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(idx,1,"smem[=64+0]\n",smem_ptr);
      */
    }
    __syncthreads();
      
    // unrolling warp

    if (tid < 32)
    {
        //volatile uint32_t *vsmem = smem_ptr;
        uint32_t *zvsmem = smem_ptr->getu256();
        T1 vsmem(zvsmem);
 
        /*
        logInfoBigNumberTid(idx,1,"smem[pre32X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[pre32Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[pre32Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+32)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(idx,1,"smem[32X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[32Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[32Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);

        logInfoBigNumberTid(idx,1,"smem[pre16X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[pre16Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[pre16Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+16)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(idx,1,"smem[16X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[16Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[16Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+8)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(idx,1,"smem[8X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[8Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[8Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+4)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(idx,1,"smem[4X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[4Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[4Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+2)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(idx,1,"smem[2X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[2Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[2Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+1)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(idx,1,"smem[X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(idx,1,"smem[Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(idx,1,"smem[Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        if (tid==0) {
           xr->setu256(0,smem_ptr,0);
        }
    }

  return;
}

template<typename T1, typename T2>
__forceinline__ __device__ void madecjac_shfl(T1 *xr, T1 *xo, uint32_t *scl, T1 *smem_ptr, kernel_params_t *params)
{
    uint32_t i, size1, size2;
    uint32_t ndbg = T1::getN();
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t __align__(16) zsumX[ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint32_t)];
    uint32_t __align__(16) zsumY[ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint32_t)];
    uint32_t laneIdx = tid % warpSize;
    uint32_t warpIdx = tid / warpSize;
    T1 sumX(zsumX);
    T1 sumY(zsumY);
    T1 _inf;
    infz(&_inf, params->midx);

    size1 = 16;
    // ECP_JAC_INOFFSET = 3 * NWORDS_256BIT
    // ECP_JAC_INXOFFSET = 1 * NWORDS_256BIT
    // scalar multipliation
    if (params->premul){
        sumX.setu256(0,xo,0,1);
        sumX.setu256(1,xo,1,1);

        logInfoBigNumberTid(idx,1,"scl :\n",scl);
 
        scmulecjac<T1, T2>(&sumX,0, &sumX, 0, scl,  params->midx);
          
        logInfoBigNumberTid(idx,3*ndbg,"Xout[x,y,z]:\n",&sumX);

        size2 = blockDim.x >> 6;
    } else {
        size2 = blockDim.x >> 6;
        sumX.setu256(0,xo,0);
    }
   
    __syncthreads();

    // block wide warp reduce
    #pragma unroll
    for (i = size1; i > 0; i >>= 1){
      shflxoruecc<T1,T2>(&sumY, &sumX, i);
      logInfoTid(idx,"idx:%d\n",i);
      logInfoBigNumberTid(idx,3*ndbg,"sumX\n",&sumX);
      logInfoBigNumberTid(idx,3*ndbg,"sumY\n",&sumY);

      addecjac<T1,T2>(&sumX,0, &sumX,0, &sumY,0, params->midx);

      logInfoBigNumberTid(idx,3*ndbg,"sumX+\n",&sumX);
    }

    __syncthreads();
    if (laneIdx == 0) {
       smem_ptr->setu256(warpIdx*ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint256_t), &sumX,0);
       logInfoTid(idx,"save idx:%d\n",warpIdx);
       logInfoBigNumberTid(idx,ndbg*3,"val\n",&sumX);
    }

    __syncthreads();
  
    if (size2){

      logInfoBigNumberTid(idx,ndbg*3,"Smem\n",smem_ptr);
      if (tid < size2*2) {
        logInfoTid(idx,"blockDim :%d\n",blockDim.x);
        logInfoTid(idx,"LaneIdx :%d\n",laneIdx);
        logInfoTid(idx,"Size :%d\n",size2);
  
        sumX.setu256(0,smem_ptr,laneIdx*ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint256_t));
      } else {
        sumX.setu256(0,&_inf,0);
      }
      logInfoBigNumberTid(idx,ndbg*3,"Second\n",&sumX);
  
      #pragma unroll
      // last warp reduce
      for (i=size2; i > 0; i >>=1){
        shflxoruecc<T1,T2>(&sumY, &sumX, i);
        logInfoTid(idx,"idx:%d\n",i);
        logInfoBigNumberTid(idx,ndbg*3,"sumY\n",&sumY);
        logInfoBigNumberTid(idx,ndbg*3,"sumX\n",&sumX);
        addecjac<T1,T2>(&sumX,0, &sumX,0, &sumY,0, params->midx);
        logInfoBigNumberTid(idx,3*ndbg,"sumX+\n",&sumX);
      }
    }

    __syncthreads();
    if (tid==0) {
     //TODO change be movu256
     xr->setu256(0,&sumX,0);
     logInfoBigNumberTid(idx,ndbg*3,"Z-sumX : \n",&sumX);
    }

  return;
}


/*
  EC point addition
  
  Algorithm (https://en.wikibooks.org/wiki/Cryptography/Prime_Curve/Jacobian_Coordinates):
  IN : P1(X1,Y1,Z1), P2(X2,Y2,Z2)
  OUT: P3(X3,Y3,Z3)

    U1 = X1*Z2^2
    U2 = X2*Z1^2
    S1 = Y1*Z2^3
    S2 = Y2*Z1^3
    if (U1 == U2)
      if (S1 != S2)
        return POINT_AT_INFINITY
      else 
        return POINT_DOUBLE(X1, Y1, Z1)
    H = U2 - U1
    R = S2 - S1
    X3 = R^2 - H^3 - 2*U1*H^2
    Y3 = R*(U1*H^2 - X3) - S1*H^3
    Z3 = H*Z1*Z2
    return (X3, Y3, Z3)

    TODO . check if I can remove
    NOTE X1 cannot be 0. X2 can from two sources: 
      - reduction -> when there is not enough input data, i append with 0 and put it in second addition term
      - scalar multiplication -> if first bit of scalar is 0, I add 0
*/
template<typename T1, typename T2>
__forceinline__ __device__ void addecjac(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t x1offset, T1 *zx2, uint32_t x2offset, mod_t midx)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  T1 x1(zx1->getu256(0+x1offset)), y1(zx1->getu256(1+x1offset)), z1(zx1->getu256(2+x1offset));
  T1 x2(zx2->getu256(0+x2offset)), y2(zx2->getu256(1+x2offset)), z2(zx2->getu256(2+x2offset));
  T1 xr(zxr->getu256(0+zoffset)),  yr(zxr->getu256(1+zoffset)), zr(zxr->getu256(2+zoffset));
  T1 _inf;

  infz(&_inf, midx);
 
  uint32_t ndbg=T1::getN();
  uint32_t __restrict__ ztmp[7*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(ztmp), tmp2(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp3(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]),
                 tmp4(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_x(&ztmp[4*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_y(&ztmp[5*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_z(&ztmp[6*sizeof(T2)/sizeof(uint32_t)]);

  // TODO : Change definition of inf to 0, 1, 0 instead of 1,0,1 as it is now
  /*
  logInfoBigNumberTid(tid,ndbg,"x1\n",x1.getu256());
  logInfoBigNumberTid(tid,ndbg,"y1\n",y1.getu256());
  logInfoBigNumberTid(tid,ndbg,"z1\n",z1.getu256());
  logInfoBigNumberTid(tid,ndbg,"x2\n",x2.getu256());
  logInfoBigNumberTid(tid,ndbg,"y2\n",y2.getu256());
  logInfoBigNumberTid(tid,ndbg,"z2\n",z2.getu256());
  */

  if (eq0z(&y2)){ 
      zxr->setu256(zoffset,zx1,x1offset);
      //logInfoTid(tid,"R1=inf\n",tid);
      return;  
  }
  squarez(&tmp_x, &z1,         midx);  // tmp_x = z1sq 
  mulz(&tmp_z, &tmp_x, &x2, midx);  // tmp_z = u2 = x2 * z1sq
  mulz(&tmp_x, &tmp_x, &z1, midx);  // tmp_x = z1cube
  mulz(&tmp_x, &tmp_x, &y2, midx);  // tmp_x = s2 = z1cube * y2
  squarez(&tmp_y, &z2,        midx);  // tmp_y = z2sq
  mulz(&tmp1, &x1, &tmp_y, midx);  // tmp1 = u1 = x1 * z2sq
  mulz(&tmp_y, &tmp_y, &z2, midx);  // tmp_y = z2cube
  mulz(&tmp_y, &tmp_y, &y1, midx);  // tmp_y = s1 = z2cube * y1

  //  if U1 == U2 and S1 == S2 => P1 = P2 -> double
  //  if U1 == U2 and S1 != S2 => P1 = -P2 -> return 0
  //  instead of calling double,  i proceed. It is better to avoid warp divergence
  if (eqz(&tmp1, &tmp_z) &&   // u1 == u2
       !eqz( &tmp_y, &tmp_x)){  // s1 != s2
          zxr->setu256(zoffset,&_inf,x1offset);
          //logInfoTid(tid,"R2=inf\n",tid);
	  return;  

  }

  subz(&tmp2, &tmp_z, &tmp1, midx);     // H = tmp2 = u2 - u1
  mulz(&tmp_z, &z1, &z2, midx);      // tmp_z = z1 * z2
  mulz(&zr, &tmp_z, &tmp2, midx);       // zr = z1 * z2  * h

  /*
  logInfoBigNumberTid(tid,ndbg,"H\n",&tmp2);
  logInfoBigNumberTid(tid,ndbg,"z1 * z2\n",&tmp_z);
  logInfoBigNumberTid(tid,ndbg,"z1 * z2  * h\n",&zr);
  */

  squarez(&tmp3, &tmp2,        midx);     // Hsq = tmp3 = H * H 
  mulz(&tmp2, &tmp3, &tmp2, midx);     // Hcube = tmp2 = Hsq * H 
  mulz(&tmp1, &tmp1, &tmp3, midx);     // tmp1 = u1 * Hsq

  /*
  logInfoBigNumberTid(tid,ndbg,"Hsq\n",&tmp3);
  logInfoBigNumberTid(tid,ndbg,"H3\n",&tmp2);
  logInfoBigNumberTid(tid,ndbg,"Hsq * u1\n",&tmp1);
  */

  subz(&tmp3, &tmp_x, &tmp_y, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, tmp_x=free, tmp_y=s1, zr=zr
  mulz(&tmp_y, &tmp_y, &tmp2, midx);     // tmp_y = Hcube * s1
  squarez(&tmp_x, &tmp3, midx);     // tmp_x = R * R

  /*
  logInfoBigNumberTid(tid,ndbg,"R\n",&tmp3);
  logInfoBigNumberTid(tid,ndbg,"Hcube* s1\n",&tmp_y);
  logInfoBigNumberTid(tid,ndbg,"Rsq * u1\n",&tmp_x);
  */

  subz(&tmp_x, &tmp_x, &tmp2, midx);        // tmp_x = x3= (R*R)-Hcube, tmp_y = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  // TODO muluk256
  mul2z(&tmp4, &tmp1, midx);     // tmp4 = u1*hsq *_2

  /*
  logInfoBigNumberTid(tid,ndbg,"Rsq - H3\n",&tmp_x);
  logInfoBigNumberTid(tid,ndbg,"Hsq * 2 * u1\n",&tmp4);
  */

  subz(&xr, &tmp_x, &tmp4, midx);               // x3 = xr
  subz(&tmp1, &tmp1, &xr, midx);       // tmp1 = u1*hs1 - x3
  mulz(&tmp1, &tmp1, &tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)
  subz(&yr, &tmp1, &tmp_y, midx);

  /*
  logInfoBigNumberTid(tid,ndbg,"X : \n",&xr);
  logInfoBigNumberTid(tid,ndbg,"Y : \n",&yr);
  logInfoBigNumberTid(tid,ndbg,"Z : \n",&zr);
  */
}

template<typename T1, typename T2>
__forceinline__ __device__ void addecjacmixed(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t x1offset, T1 *zx2, uint32_t x2offset, mod_t midx)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  T1 x1(zx1->getu256(0+x1offset)), y1(zx1->getu256(1+x1offset));
  T1 x2(zx2->getu256(0+x2offset)), y2(zx2->getu256(1+x2offset)), z2(zx2->getu256(2+x2offset));
  T1 xr(zxr->getu256(0+zoffset)),  yr(zxr->getu256(1+zoffset)), zr(zxr->getu256(2+zoffset));
  T1 _inf;

  infz(&_inf, midx);
 
  uint32_t ndbg=T1::getN();
  uint32_t __restrict__ ztmp[4*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(ztmp), tmp3(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp_x(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_z(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]);

  // TODO : Change definition of inf to 0, 1, 0 instead of 1,0,1 as it is now
  /*
  logInfoBigNumberTid(tid,ndbg,"x1\n",x1.getu256());
  logInfoBigNumberTid(tid,ndbg,"y1\n",y1.getu256());
  logInfoBigNumberTid(tid,ndbg,"x2\n",x2.getu256());
  logInfoBigNumberTid(tid,ndbg,"y2\n",y2.getu256());
  logInfoBigNumberTid(tid,ndbg,"z2\n",z2.getu256());
  */

  if (eq0z(&y2)){ 
      zxr->setu256(zoffset,zx1,x1offset);
      //logInfoTid(tid,"R1=inf\n",tid);
      return;  
  }
  squarez(&tmp_x, &z2,         midx);  // tmp_x = z2sq 
  mulz(&tmp_z, &tmp_x, &x1, midx);  // tmp_z = u1 = x1 * z2sq
  mulz(&tmp_x, &tmp_x, &z2, midx);  // tmp_x = z2cube
  mulz(&tmp_x, &tmp_x, &y1, midx);  // tmp_x = s1 = z2cube * y1

  //  if U1 == U2 and S1 == S2 => P1 = P2 -> double
  //  if U1 == U2 and S1 != S2 => P1 = -P2 -> return 0
  //  instead of calling double,  i proceed. It is better to avoid warp divergence
  if (eqz(&x2, &tmp_z) &&   // u1 == u2
       !eqz( &y2, &tmp_x)){  // s1 != s2
          zxr->setu256(zoffset,&_inf,x1offset);
          //logInfoTid(tid,"R2=inf\n",tid);
	  return;  

  }

  subz(&tmp1, &x2, &tmp_z, midx);     // H = tmp1 = u2 - u1
  mulz(&zr, &z2, &tmp1, midx);       // zr = z1 * z2  * h
  squarez(&tmp3, &tmp1,        midx);     // Hsq = tmp3 = H * H 

  /*
  logInfoBigNumberTid(tid,ndbg,"H\n",&tmp1);
  logInfoBigNumberTid(tid,ndbg,"z2 * h\n",&zr);
  logInfoBigNumberTid(tid,ndbg,"Hsq\n",&tmp3);
  */

  mulz(&tmp1, &tmp3, &tmp1, midx);     // Hcube = tmp1= Hsq * H 
  mulz(&tmp3, &tmp_z, &tmp3, midx);     // tmp3 = u1 * Hsq

  /*
  logInfoBigNumberTid(tid,ndbg,"H3\n",&tmp1);
  logInfoBigNumberTid(tid,ndbg,"Hsq * u1\n",&tmp3);
  */

  subz(&tmp_z, &y2, &tmp_x, midx);        // R = tmp_z = S2 - S1 
  squarez(&xr, &tmp_z, midx);     // xr = Rsq

  /*
  logInfoBigNumberTid(tid,ndbg,"R\n",&tmp_z);
  logInfoBigNumberTid(tid,ndbg,"Rsq\n",&xr);
  */

  subz(&xr, &xr, &tmp1, midx);     // xr = Rsq - Hcube

  /*
  logInfoBigNumberTid(tid,ndbg,"Rsq - Hcube\n",&xr);
  */

  subz(&xr, &xr, &tmp3, midx);     // xr = Rsq - Hcube - u1*Hsq
  subz(&xr, &xr, &tmp3, midx);     // xr = Rsq - Hcube - 2*u1*Hsq

  /*
  logInfoBigNumberTid(tid,ndbg,"X\n",&xr);
  */

  subz(&yr, &tmp3, &xr, midx);          // yr = u1*Hsq - xr
  mulz(&yr, &yr, &tmp_z, midx);        //  yr = R*(u1*Hsq - xr)

  mulz(&tmp1, &tmp1, &tmp_x, midx);     // tmp1 = Hcube * s1
  subz(&yr, &yr, &tmp1, midx);

  /*
  logInfoBigNumberTid(tid,ndbg,"X : \n",&xr);
  logInfoBigNumberTid(tid,ndbg,"Y : \n",&yr);
  logInfoBigNumberTid(tid,ndbg,"Z : \n",&zr);
  */
}

/*
  input is in affine coordinates -> P(Z) = 1
  I can do Q = Q+Y or Q = Y + Q
    NOTE X1, X2 cannot be 0
*/

template <typename T1, typename T2>
__forceinline__ __device__ void addecjacaff(T1  *zxr, T1 *zx1, T1 *zx2, mod_t midx)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  T1 y1(zx1->getu256(ECP_JAC_YOFFSET_BASE)), y2(zx2->getu256(ECP_JAC_YOFFSET_BASE)),
     xr(zxr->getu256(ECP_JAC_XOFFSET_BASE)),
     yr(zxr->getu256(ECP_JAC_YOFFSET_BASE)), zr(zxr->getu256(ECP_JAC_ZOFFSET_BASE));

  T1 _inf;
  uint32_t ndbg = T1::getN();

  infz(&_inf, midx);

 
  uint32_t __restrict__ ztmp[4*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(ztmp), tmp2(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp3(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]), 
                 tmp4(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]);

  // TODO Check if I can call add to compute x + x (instead of double)
  //  if not, I should call double below. I don't want to to avoid warp divergnce
  if (eqz(zx1, zx2) &&   // u1 == u2
       !eqz( &y1,  &y2)){  // s1 != s2
          zxr->setu256(0,&_inf,0);
	  return;  //  if U1 == U2 and S1 == S2 => P1 == P2 (call double)
  }

  /*
  logInfoBigNumberTid(tid,ndbg,"x1\n",zx1);
  logInfoBigNumberTid(tid,ndbg,"y1\n",&y1);
  logInfoBigNumberTid(tid,ndbg,"x2\n",zx2);
  logInfoBigNumberTid(tid,ndbg,"y2\n",&y2);

  logInfoBigNumberTid(tid,1,"x22\n",zx2->get2u256());
  logInfoBigNumberTid(tid,1,"x12\n",zx1->get2u256());
  */

  subz(&zr, zx2, zx1, midx);     // H = tmp2 = u2 - u1
  
  /*
  logInfoBigNumberTid(tid,ndbg,"H\n",&zr);
  */

  squarez(&tmp3, &zr,        midx);     // Hsq = tmp3 = H * H 
  mulz(&tmp2, &tmp3, &zr, midx);     // Hcube = tmp2 = Hsq * H 
  mulz(&tmp1, zx1, &tmp3, midx);     // tmp1 = u1 * Hsq

  /*
  logInfoBigNumberTid(tid,ndbg,"Hsq\n",&tmp3);
  logInfoBigNumberTid(tid,ndbg,"Hcube\n",&tmp2);
  logInfoBigNumberTid(tid,ndbg,"u1 * Hsq\n",&tmp1);
  */

  subz(&tmp3, &y2, &y1, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, xr=free, yr=s1, zr=zr
  mulz(&yr, &y1, &tmp2, midx);     // yr = Hcube * s1
  squarez(zxr, &tmp3, midx);     // xr = R * R

  /*
  logInfoBigNumberTid(tid,ndbg,"R\n",&tmp3);
  logInfoBigNumberTid(tid, ndbg,"s1\n",&yr);
  logInfoBigNumberTid(tid,ndbg,"Rsq\n",&xr);
  */
  subz(zxr, zxr, &tmp2, midx);        // xr = x3= (R*R)-Hcube, yr = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  // TODO muluk256
  mul2z(&tmp4, &tmp1, midx);     // tmp4 = u1*hsq *_2

  /*
  logInfoBigNumberTid(tid,ndbg,"Rsq - Hcube\n",&xr);
  logInfoBigNumberTid(tid,ndbg,"u1 * Hsq * 2\n",&tmp4);
  */

  subz(zxr, &xr, &tmp4, midx);               // x3 = xr
  subz(&tmp1, &tmp1, zxr, midx);       // tmp1 = u1*hs1 - x3
  //logInfoBigNumberTid(tid,ndbg,"u1*hsq - x3\n",&tmp1);
  mulz(&tmp1, &tmp1, &tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)
  //logInfoBigNumberTid(tid,ndbg,"r * (u1*hsq - x3)\n",&tmp1);
  subz(&yr, &tmp1, &yr, midx);

  /*
  logInfoBigNumberTid(tid,ndbg,"X3\n",&xr);
  logInfoBigNumberTid(tid,ndbg,"Y3\n",&yr);
  logInfoBigNumberTid(tid,ndbg,"Z3\n",&zr);
  */
}

/*
  EC point addition
  
  Algorithm (https://en.wikibooks.org/wiki/Cryptography/Prime_Curve/Jacobian_Coordinates):
  IN : P1(X1,Y1,Z1)
  OUT: P'(X',Y',Z')

   if (Y == 0)
      return POINT_AT_INFINITY
   S = 4*X*Y^2
   M = 3*X^2 + a*Z^4
   X' = M^2 - 2*S
   Y' = M*(S - X') - 8*Y^4
   Z' = 2*Y*Z
   return (X', Y', Z')
*/
template<typename T1, typename T2>
__forceinline__ __device__ void doublecjac(T1 *zxr, T1 *zx1, mod_t midx)
{
  #if 0
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  T1 y1(zx1->getu256(1)), z1(zx1->getu256(2));
  T1 yr(zxr->getu256(1)), zr(zxr->getu256(2));
  T1 _inf;
  uint32_t ndbg = T1::getN();

  infz(&_inf,midx);


  uint32_t __restrict__ ztmp[5*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(ztmp), tmp2(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp_x(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_y(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_z(&ztmp[4*sizeof(T2)/sizeof(uint32_t)]);

  // TODO : review this comparison, and see if I can do better. or where I should put it
  // as i check this in several places
  if (eq0z(&y1)){ 
      zxr->setu256(0,&_inf,0);
      //memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
      return;  
  }
  squarez(&tmp_z, &y1,            midx);  // tmp_z = ysq
  squarez(&tmp_y, &tmp_z, midx);  // tmp_y = ysqsq
  // TODO muluk256
  mul8z(&tmp_y, &tmp_y, midx);  // tmp_y = ysqsq *_8
  mulz(&tmp_z, &tmp_z, zx1, midx);  // S = tmp_z = x * ysq
  // TODO muluk256
  mul4z(&tmp_z, &tmp_z, midx);  // S = tmp_z = S * _4

  squarez(&tmp_x, zx1, midx);  // M1 = tmp_x = x * x
  // TODO muluk256
  mul3z(&tmp1, &tmp_x, midx);  // M = tmp1 = M1 * _3
  squarez(&tmp_x, &tmp1, midx);  // X3 = tmp_x = M * M,  tmp_y = Ysqsq * _8, tmp_z = S; tmp1 = M
  // TODO muluk256
  mul2z(&tmp2, &tmp_z, midx);   // tmp2 = S * _2
  subz(zxr, &tmp_x, &tmp2, midx);      // X3 = tmp_x; tmp_y = Ysqsq * _8, tmp_z = S, tmp1 = M, 
  subz(&tmp2, &tmp_z, zxr, midx);   //  tmp2 = S - X3
  mulz(&tmp2, &tmp2, &tmp1, midx); // tmp2 = M * (S - X3)
  mulz(&tmp_z, &y1, &z1, midx);
  // TODO muluk256
  mul2z(&zr, &tmp_z, midx);
  subz(&yr, &tmp2, &tmp_y, midx);

  /*
  logInfoBigNumberTid(tid,ndbg,"X : \n",zxr);
  logInfoBigNumberTid(tid,ndbg,"Y : \n",&yr);
  logInfoBigNumberTid(tid,ndbg,"Z : \n",&zr);
  */
  #else

 int tid = threadIdx.x + blockDim.x * blockIdx.x;

  T1 y1(zx1->getu256(1)), z1(zx1->getu256(2));
  T1 yr(zxr->getu256(1)), zr(zxr->getu256(2));
  T1 _inf;
  uint32_t ndbg = T1::getN();

  infz(&_inf,midx);


  uint32_t __restrict__ ztmp[2*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp_y(ztmp), tmp_z(&ztmp[sizeof(T2)/sizeof(uint32_t)]);

  // TODO : review this comparison, and see if I can do better. or where I should put it
  // as i check this in several places
  if (eq0z(&y1)){ 
      zxr->setu256(0,&_inf,0);
      //memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
      return;  
  }
  squarez(&tmp_z, &y1,            midx);  // tmp_z = ysq
  squarez(&tmp_y, &tmp_z, midx);  // tmp_y = ysqsq

  addz(&tmp_y, &tmp_y, &tmp_y, midx);  // tmp_y = ysqsq + ysqsq
  addz(&tmp_y, &tmp_y, &tmp_y, midx);  // tmp_y = 2ysqsq + 2ysqsq
  addz(&tmp_y, &tmp_y, &tmp_y, midx);  // tmp_y = 4ysqsq + 4ysqsq

  mulz(&tmp_z, &tmp_z, zx1, midx);  
  addz(&tmp_z, &tmp_z, &tmp_z, midx);  
  addz(&tmp_z, &tmp_z, &tmp_z, midx);  // S = tmp_z = 2X1Ysq + 2X1Ysq

  mulz(&zr, &y1, &z1, midx);     //  Z3 = Y * Z
  addz(&zr, &zr, &zr, midx);

  squarez(&yr, zx1, midx);           
  addz(zxr, &yr, &yr, midx);       
  addz(&yr, zxr, &yr, midx);       // M = yr = 3Xsq

  squarez(zxr, &yr, midx);       // X3 = Msq

  subz(zxr, zxr, &tmp_z, midx);   // X3 = Msq - S
  subz(zxr, zxr, &tmp_z, midx);      // X3 = Msq - 2S

  subz(&tmp_z, &tmp_z, zxr, midx);   //  tmp_z = S - X3
  mulz(&yr, &yr, &tmp_z, midx);     //  Y3 = M * (S - X3)
  subz(&yr, &yr, &tmp_y, midx);    // Y3 = M * (S - X3) - 8ysqsq


  /*
  logInfoBigNumberTid(tid,ndbg,"X : \n",zxr);
  logInfoBigNumberTid(tid,ndbg,"Y : \n",&yr);
  logInfoBigNumberTid(tid,ndbg,"Z : \n",&zr);
  */
  #endif
}

/* 
   X1 cannot be 0
 */
template<typename T1, typename T2>
__forceinline__ __device__ void doublecjacaff(T1 *zxr, T1 *zx1, mod_t midx)
{
  uint32_t ztmp[3*sizeof(T2)/sizeof(uint32_t)];
 
  T1 y1(zx1->getu256(1)); 
  T1 yr(zxr->getu256(1)), zr(zxr->getu256(2)); 

  T1 tmp1(ztmp), tmp2(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp_y(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]);

  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  T1 _inf;
  uint32_t ndbg = T1::getN();

  infz(&_inf,midx);

  /*
  logInfoBigNumberTid(tid, ndbg,"x1\n",zx1->getu256());
  logInfoBigNumberTid(tid,ndbg,"y1\n",y1.getu256());
  */
  squarez(&zr, &y1, midx);  // zr = ysq
  squarez(&tmp_y, &zr, midx);  // yr = ysqsq

  /*
  logInfoBigNumberTid(tid,ndbg,"ysq\n",zr.getu256());
  logInfoBigNumberTid(tid,ndbg,"Yqsq\n",tmp_y.getu256());
  */
  // TODO muluk256
  mul8z(&tmp_y, &tmp_y, midx);  // tmp_y = ysqsq *_8
  mulz(&zr, &zr, zx1, midx);  // S = zr = x * ysq

  /*
  logInfoBigNumberTid(tid,ndbg,"8*Ysqsq\n",tmp_y.getu256());
  logInfoBigNumberTid(tid,ndbg,"S\n",zr.getu256());
  */
  // TODO muluk256
  mul4z(&zr, &zr, midx);  // S = zr = S * _4

  //logInfoBigNumberTid(tid,ndbg,"S*4\n",zr.getu256());

  squarez(zxr, zx1, midx);  // M1 = xr = x * x
  // TODO muluk256
  mul3z(&tmp1, zxr, midx);  // M = tmp1 = M1 * _3

  /*
  logInfoBigNumberTid(tid,ndbg,"Xsq\n",zxr->getu256());
  logInfoBigNumberTid(tid,ndbg,"M\n",tmp1.getu256());
  */
  squarez(zxr, &tmp1, midx);  // X3 = xr = M * M,  tmp_y = Ysqsq * _8, zr = S; tmp1 = M
  // TODO muluk256
  mul2z(&tmp2, &zr, midx);   // tmp2 = S * _2

  /* 
  logInfoBigNumberTid(tid,ndbg,"M*M\n",zxr->getu256());
  logInfoBigNumberTid(tid,ndbg,"S*2\n",tmp2.getu256());
  */

  subz(zxr, zxr, &tmp2, midx);      // X3 = xr; tmp_y = Ysqsq * _8, zr = S, tmp1 = M, 
  subz(&tmp2, &zr, zxr, midx);   //  tmp2 = S - X3

  /*
  logInfoBigNumberTid(tid,ndbg,"X3\n",zxr->getu256());
  logInfoBigNumberTid(tid,ndbg,"S-X3\n",tmp2.getu256());
  */

  mulz(&tmp2, &tmp2, &tmp1, midx); // tmp2 = M * (S - X3)
  //logInfoBigNumberTid(tid,ndbg,"M * (S-X3)\n",tmp2.getu256());
  // TODO muluk256
  mul2z(&zr, &y1, midx);
  subz(&yr, &tmp2, &tmp_y, midx);

  /*
  logInfoBigNumberTid(tid,ndbg,"y3\n",yr.getu256());
  logInfoBigNumberTid(tid,ndbg,"z3\n",zxr->getu256());
  */
}

template<typename T1, typename T2>
__forceinline__ __device__ void scmulecjac(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t xoffset, uint32_t *scl, mod_t midx)
{
  uint32_t b0;
  uint32_t ndbg = T1::getN();
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  uint32_t i, iter=0;

  uint32_t __restrict__ zN[3*sizeof(T2)/sizeof(uint32_t)]; // N = P
  uint32_t *_1 = misc_const_ct[midx]._1;
  T1 _inf;
  T1 N(zN);
  T1 Q(zxr->getu256(zoffset));
  T1 y1(zx1->getu256(xoffset+1));

  infz(&_inf, midx);

  // TODO : review this comparison
  if (eq0z(&y1)){ 
      zxr->setu256(zoffset,&_inf,0);
      return;  
  }

  //N.setu256(0,zx1,xoffset);
  N.setu256(0,zx1,xoffset,1);
  N.setu256(1,zx1,xoffset+1,1);
  setkz(&N,2,_1);


  Q.setu256(0,&_inf, 0);
  
  if (eq0u256(scl)) { return; }

  // TODO : Either implement left to right, or count where msb is and substitute while by unrolled
  // loop

  // TODO : MAD several numbers at once using shamir's trick

  logInfoBigNumberTid(tid,1,"SCL mul: \n",scl);
  logInfoBigNumberTid(tid,3*ndbg,"Q: \n",&Q);
  logInfoBigNumberTid(tid,3*ndbg,"N: \n",&N);

  #if 0
    uint32_t __restrict__ scl_cpy[NWORDS_256BIT];
    movu256(scl_cpy, scl);
    for (i=0; i< 32; i++){
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
    }
  #else
    uint32_t offset;
    uint32_t msb = clzMu256(scl);
    logInfoTid(tid,"msb : %d \n",msb);
    //#pragma unroll
    for (i=msb>>U256_MBSCLUSTER; i< (1 << (NWORDS_256BIT - U256_MBSCLUSTER)); i++){
        offset = i<<U256_MBSCLUSTER;
        scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset,   midx);
        scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+1, midx);
        scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+2, midx);
        scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+3, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+4, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+5, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+6, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+7, midx);
     }
  #endif

  logInfoBigNumberTid(tid,3*ndbg,"R-N: \n",&N);
  logInfoBigNumberTid(tid,3*ndbg,"R-Q: \n",&Q);
  return;
}

template<typename T1, typename T2>
__device__ void scmulecjac_step_r2l(T1 *Q,T1 *N, uint32_t *scl, mod_t midx )
{
   uint32_t  b0 = shr1u256(scl);
   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   logInfoTid(tid,"B0 : %d\n",b0);
   if (b0) {
      addecjac<T1, T2> (Q,0, N,0, Q,0, midx);
   }
   doublecjac<T1, T2>(N,N, midx);
}

template<typename T1, typename T2>
__device__ void scmulecjac_step_l2r(T1 *Q,T1 *N, uint32_t *scl, uint32_t offset, mod_t midx )
{
   uint32_t  b0 = bselMu256(scl,255-offset);
   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   logInfoTid(tid,"B0 : %d\n",b0);
   doublecjac<T1, T2>(Q,Q, midx);
   if (b0) {
      addecjacmixed<T1, T2> (Q,0, N,0, Q,0, midx);
   }
}

template<typename T1, typename T2>
__forceinline__ __device__ void shflxoruecc(T1 *d_out,T1 *d_in, uint32_t srcLane )
{
    ulonglong4 *in, *out;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t i;

    #pragma unroll
    for (i=0; i<ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint256_t);i++){
    
      in = (ulonglong4 *)d_in->getsingleu256(i);
      out = (ulonglong4 *)d_out->getsingleu256(i);

      out->x = __shfl_xor_sync(0xffffffff, in->x, srcLane);
      out->y = __shfl_xor_sync(0xffffffff, in->y, srcLane);
      out->z = __shfl_xor_sync(0xffffffff, in->z, srcLane);
      out->w = __shfl_xor_sync(0xffffffff, in->w, srcLane);
    }
}
/////////
// Temporary implemenation of future functionality. Leave here for now...


#if 0
__forceinline__ __device__
 void addecjacaff(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, const uint32_t *x2, mod_t midx)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const uint32_t __restrict__ *y1 = &x1[NWORDS_256BIT];
  const uint32_t __restrict__ *y2 = &x2[NWORDS_256BIT];
  uint32_t __restrict__ *yr = &xr[NWORDS_256BIT];
  uint32_t __restrict__ *zr = &xr[NWORDS_256BIT*2];
  uint32_t __restrict__ *_inf = misc_const_ct[midx]._inf;
  uint32_t __restrict__ *_2 = misc_const_ct[midx]._2;
 
  uint32_t __restrict__ tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT], tmp3[NWORDS_256BIT], tmp4[NWORDS_256BIT];

  // TODO Check if I can call add to compute x + x (instead of double)
  //  if not, I should call double below. I don't want to to avoid warp divergnce
  if (eqz((const uint32_t *)x1, (const uint32_t *)x2) &&   // u1 == u2
       !eqz( (const uint32_t *) y1, (const uint32_t *) y2)){  // s1 != s2
          memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
	  return;  //  if U1 == U2 and S1 == S2 => P1 == P2 (call double)
  }

  /*
  if (tid == 0){
     logInfoBigNumber("x1\n",(uint32_t *)x1);
     logInfoBigNumber("y1\n",(uint32_t *)y1);
     logInfoBigNumber("x2\n",(uint32_t *)x2);
     logInfoBigNumber("y2\n",(uint32_t *)y2);
  }
  */
  subz(zr, x2, x1, midx);     // H = tmp2 = u2 - u1
  if (tid == 0){
    logInfoBigNumber("H\n",(uint32_t *)zr);
  }

  squarez(tmp3, zr,        midx);     // Hsq = tmp3 = H * H 
  mulz(tmp2, tmp3, zr, midx);     // Hcube = tmp2 = Hsq * H 
  mulz(tmp1, x1, tmp3, midx);     // tmp1 = u1 * Hsq

  /*
  if (tid == 0){
    logInfoBigNumber("Hsq\n",(uint32_t *)tmp3);
    logInfoBigNumber("Hcube\n",(uint32_t *)tmp2);
    logInfoBigNumber("u1 * Hsq\n",(uint32_t *)tmp1);
  }
  */

  subz(tmp3, y2, y1, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, xr=free, yr=s1, zr=zr
  mulz(yr, y1, tmp2, midx);     // yr = Hcube * s1
  squarez(xr, tmp3, midx);     // xr = R * R

  /*
  if (tid == 0){
    logInfoBigNumber("R\n",(uint32_t *)tmp3);
    logInfoBigNumber("s1\n",(uint32_t *)yr);
    logInfoBigNumber("Rsq\n",(uint32_t *)xr);
  }
  */
  subz(xr, xr, tmp2, midx);        // xr = x3= (R*R)-Hcube, yr = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  // TODO muluk256
  mul2z(tmp4, tmp1, midx);     // tmp4 = u1*hsq *_2

  /*
  if (tid == 0){
    logInfoBigNumber("Rsq - Hcube\n",(uint32_t *)xr);
    logInfoBigNumber("u1 * Hsq * 2\n",(uint32_t *)tmp4);
  }
  */
  subz(xr, xr, tmp4, midx);               // x3 = xr
  subz(tmp1, tmp1, xr, midx);       // tmp1 = u1*hs1 - x3
  mulz(tmp1, tmp1, tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)
  subz(yr, tmp1, yr, midx);

  /*
  if (tid == 0){
    logInfoBigNumber("X3\n",(uint32_t *)xr);
    logInfoBigNumber("u1 * hsq - x3\n",(uint32_t *)tmp1);
    logInfoBigNumber("Y3\n",(uint32_t *)yr);
  }
  */
}
#endif

#if 0
__global__ void addecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1, *x2, *xr;
 
    if(tid >= params->in_length/6) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * 2 * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];
    x2 = (uint32_t *) &in_vector[(tid * 2 + 1) * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];
    
    addecldr(xr, x1, x2, x1, params->midx);

    return;
}
__global__ void doublecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1,*xr;
 
    if(tid >= params->in_length/3) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];
    
    doublecldr(xr, x1, params->midx);

  return;
}
__global__ void scmulecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *x1, *scl, *xr;
 
   if(tid >= params->in_length/3) {
     return;
   }

   x1  = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];
   scl = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_SCLOFFSET];

   xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];

   
   ldrstep(xr, x1, scl,  params->midx);

   return;
}

__global__ void madecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
  return;
}

    
__forceinline__ __device__
 void addecldr(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, 
            const uint32_t __restrict__ *x2, const uint32_t __restrict__ *xp, mod_t midx)
{
   // Xr = -4*b Z1 * Z2 * (X1 * Z2 + X2 * Z1) + (X1 * X2)^2 
   // Zr = xp * (X1 * Z2 - X2 * Z1)^2

   // 7 M, 2 SQ, 3 ADD
   const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT];
   const uint32_t __restrict__ *z2 = &x2[NWORDS_256BIT];
   uint32_t __restrict__ *zr =&zr[NWORDS_256BIT];

   uint32_t tmp1[NWORDS_256BIT];
   uint32_t __restrict__ *_4b = misc_const_ct[midx]._4b;
   

   mulz(tmp1, x2  , z1  , midx);      
   mulz(xr  , x1  , z2  , midx);      
   subz(   zr  , xr  , tmp1, midx);
   addz(   tmp1, tmp1, xr  , midx);
   mulz(tmp1, tmp1, z2  , midx);    
   mulz(tmp1, tmp1, z1  , midx);    
   mulz(xr  , x1  , x2  , midx);      
   squarez(xr  , xr         , midx);    
   // multiply by 12. 
   //  Using Montgomery: 136 mul + 346 add.
   //  Chaining 12 additions : 0 mul + 84 adds + modulus!!!
   // TODO : Use muluk256 function
   mulkz(tmp1, tmp1,_4b  , midx);  
   subz(   xr,   tmp1, xr  , midx);
   squarez(zr,   zr         , midx);     
   mulz(zr,   zr  , xp  , midx);   

  return;
}

__forceinline__ __device__
 void doublecldr(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, mod_t midx)
{
  // Xr = X1^4 - 8*b * X1*Z1^3
  // Zr = 4*Z1 * (X1^3 + b*Z1^3) 

  // 7 M, 3 SQ, 2 Add
  const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT];
  uint32_t __restrict__ *zr =&zr[NWORDS_256BIT];

  uint32_t tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT];
  uint32_t __restrict__ *_4 = misc_const_ct[midx]._4;
  uint32_t __restrict__ *b = ecbn128_params_ct[midx].b;
  uint32_t __restrict__ *_8b = misc_const_ct[midx]._8b;

  squarez(xr,  z1,           midx);    
  mulz(zr,  xr,   z1,   midx);      // Zr = Z1^3
  mulz(xr,  zr,   x1,   midx);      
  // TODO muluk256
  mulkz(xr,  xr,  _8b,   midx);      // Xr = 8b * X1 * Z1^3
  squarez(tmp1, x1,         midx);      
  squarez(tmp2, tmp1,       midx);    
  subz(   xr,  tmp2, xr,   midx);

  // TODO muluk256
  mulkz(zr,  zr,   b,    midx);      // Zr = b*Z1^3
  mulz(tmp1, tmp1,  x1,   midx);     
  addz(   zr, tmp1,   zr,   midx);
  // TODO muluk256
  mulkz(zr, zr,   _4,    midx);
  mulz(zr, zr,   z1,    midx); 

  return;
}

// NOTE : EC points are in affine coordinates => Pz = 1 (in montgomery someting else)
// NOTE : EC points in montgomery, scl normal 
__forceinline__ __device__
 void ldrstep(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, uint32_t *scl, mod_t midx)
{
  uint32_t b0, idxn, idxp;
  uint32_t __restrict__ *_1 = misc_const_ct[midx]._1;

  uint32_t __restrict__ R[4*NWORDS_256BIT];

  //R[] = {[1,0],[X,1]} 
  memcpy(R, _1, NWORDS_256BIT * sizeof(uint32_t));
  memcpy(&R[3 * NWORDS_256BIT], _1, NWORDS_256BIT * sizeof(uint32_t));
  memcpy(&R[2 * NWORDS_256BIT], x1, NWORDS_256BIT * sizeof(uint32_t));

  while (!eq0u256(scl)){
     b0 = shr1u256(scl);
     idxn = ~b0 * 2 * NWORDS_256BIT;
     idxp =  b0 * 2 * NWORDS_256BIT;
     /*
     if (b0) { R0 = R0 + R1; R1 = R1 + R1;
     else {    R1 = R0 + R1; R0 = R0 + R0}
     */
     addecldr(&R[idxn], &R[idxn], &R[idxp], x1, midx);
     doublecldr(&R[idxp], &R[idxp], midx);
  }
   // TODO
   // Retrieve y(P) . Not sure if i need to convert to affine now. If I don't,
   // then I have three coordinates and it doesn't fit in my allocated space
   //
   // P = (x1, y1) , Q = (x2, y2), P-Q = (x,y)
   // Q = k P => x(R0) = X(Q), x(R1) = X(P-Q)
   //
   // y(P) = y1 = [2b + (a + x * x1) * (x + x1) - x2(x - x1) ^ 2] / (2*y)

  memcpy(xr,R,2 * NWORDS_256BIT * sizeof(uint32_t));

  return;
}

#endif


