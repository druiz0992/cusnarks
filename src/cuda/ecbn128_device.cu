#include "hip/hip_runtime.h"
/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : ecbn128_device.cu
//
// Date       : 12/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementatoin of EC Cuda functionality
// ------------------------------------------------------------------

*/

#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "ecbn128_device.h"
#include "u256_device.h"

__global__ void addecc_kernel(uint32_t *out_vector, uint32_t *in_vector, const uint32_t *p, uint32_t np, uint32_t len, uint32_t premod)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t *x1, *x2, *xr, *z1, *z2, *zr;
 
    if(tid >= len/6) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * 2 * ECK_OFFSET + ECP_XOFFSET];
    //z1 = (uint32_t *) &in_vector[tid * 2 * ECK_OFFSET + ECP_ZOFFSET];
    x2 = (uint32_t *) &in_vector[(tid * 2 + 1) * ECK_OFFSET + ECP_XOFFSET];
    //z2 = (uint32_t *) &in_vector[(tid * 2 + 1) * ECK_OFFSET + ECP_ZOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_OFFSET + ECP_XOFFSET];
    //zr = (uint32_t *) &out_vector[tid * ECK_OFFSET + ECP_ZOFFSET];
    
    if (premod){
      modu256(x1,x1,p);
      modu256(&x1[NWORDS_256BIT],&x1[NWORDS_256BIT],p);
      //modu256(z1,z1,p);
      modu256(x2,x2,p);
      modu256(&x2[NWORDS_256BIT],&x2[NWORDS_256BIT],p);
      //modu256(z2,z2,p);
    }

    addecc(xr, x1, x2, p, np);

    return;
}
__global__ void doublecc_kernel(uint32_t *out_vector, uint32_t *in_vector, const uint32_t *p, uint32_t len, uint32_t premod)
{
  return;
}
__global__ void scmulecc_kernel(uint32_t *out_vector, uint32_t *in_vector, const uint32_t *p, uint32_t len, uint32_t premod)
{
  return;
}
__global__ void addecc_reduce_kernel(uint32_t *out_vector, uint32_t *in_vector, const uint32_t *p, uint32_t len, uint32_t premod)
{
  return;
}
__global__ void scmulecc_reduce_kernel(uint32_t *out_vector, uint32_t *in_vector const uint32_t *p, uint32_t len, uint32_t premod)
{
  return;
}
    
__forceinline__ __device__ void addecc(uint32_t *xr, const uint32_t *x1, const uint32_t *x2, const uint32_t *p, const uint32_t *np)
{
   // Xr = -12 Z1 * Z2 * (X1 * Z2 + X2 * Z1) + (X1 * X2)^2 
   // Zr = x * (X1 * Z2 - X2 * Z1)^2
   const uint32_t *z1 = &x1[NWORDS_256BIT];
   const uint32_t *z2 = &x2[NWORDS_256BIT];
   uint32_t *zr =&zr[NWORDS_256BIT];

   uint32_t tmp1[NWORDS_256BIT];

   mulmontu256(tmp1, x2, z1, p, np);      
   mulmontu256(xr, x1, z2, p, np);      
   submu256(zr, xr, tmp1,  p);
   addmu256(tmp1, tmp1, xr, p);
   mulmontu256(tmp1, tmp1, z2, p, np);    
   mulmontu256(tmp1, tmp1, z1, p, np);    
   mulmontu256(xr, x1, x2, p, np);      
   mulmontu256(xr, xr, xr, p, np);     // TODO : implement squaring
   // TODO Multiply xr by -12
   addmu256(xr, tmp1, xr, p);
   mulmontu256(zr, zr, zr, p, np);     // TODO : implement squaring
   // TODO multiply zr by x

  return;
}
