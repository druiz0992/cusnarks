#include "hip/hip_runtime.h"
/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : ecbn128_device.cu
//
// Date       : 12/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementatoin of EC Cuda functionality
// 
// NOTE : EC Points do not require to be premoded premod. They shoould
//  already be < than prime
// ------------------------------------------------------------------

*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "log.h"
#include "utils_device.h"
#include "u256_device.h"
#include "z1_device.h"
#include "z2_device.h"
#include "ecbn128_device.h"

/* 
  in_vector : k[0], px[0], py[0], k[1], px[1], py[1],...  Input EC points in Affine coordinates
  out vecto : px[0], py[0], pz[0], px[1], py[1],pz[1],...              Output EC points in Jacobian coordinates
*/
__global__ void addecjacaff_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //Z1_t x1, x2, xr;

    if(tid >= params->in_length/4) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z1_t x1(&in_vector[tid * 2 * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t x2(&in_vector[(tid * 2 + 1) * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjacaff<Z1_t, uint256_t>(&xr, &x1, &x2, params->midx);

    return;

}

__global__ void addec2jacaff_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if(tid >= params->in_length/8) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z2_t x1(&in_vector[tid * 2 * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t x2(&in_vector[(tid * 2 + 1) * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjacaff<Z2_t, uint512_t>(&xr, &x1, &x2, params->midx);

    return;

}

__global__ void addecjac_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //Z1_t x1, x2, xr;

    if(tid >= params->in_length/6) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z1_t x1(&in_vector[tid * 2 * ECP_JAC_OUTOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t x2(&in_vector[(tid * 2 + 1) * ECP_JAC_OUTOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjac<Z1_t, uint256_t>(&xr,0, &x1,0, &x2,0, params->midx);

    return;

}

__global__ void addec2jac_kernel(uint32_t   *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //Z1_t x1, x2, xr;

    if(tid >= params->in_length/12) {
      return;
    }

    // x1 points to inPx[i]. x2 points to inPx[i+1]. xr points to outPx[i]
    Z2_t x1(&in_vector[tid * 2 * ECP2_JAC_OUTOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t x2(&in_vector[(tid * 2 + 1) * ECP2_JAC_OUTOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
   
    //TODO : this is not very nice, but it gets the job done. Try to come up
    // with something better 
    addecjac<Z2_t, uint512_t>(&xr,0, &x1,0, &x2,0, params->midx);

    return;

}
__global__ void doublecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/3) {
      return;
    }

    Z1_t x1(&in_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
    
    doublecjac<Z1_t, uint256_t>(&xr, &x1, params->midx);

    return;
}

__global__ void doublec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/6) {
      return;
    }

    Z2_t x1(&in_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
    
    doublecjac<Z2_t, uint512_t>(&xr, &x1, params->midx);

    return;
}
__global__ void doublecjacaff_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/2) {
      return;
    }

    Z1_t x1(&in_vector[tid * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
    Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
    
    doublecjacaff<Z1_t, uint256_t>(&xr, &x1, params->midx);

    return;
}

__global__ void doublec2jacaff_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // x1 points to inPx[i].  xr points to outPx[i]
    if(tid >= params->in_length/4) {
      return;
    }

    Z2_t x1(&in_vector[tid * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
    Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
    
    doublecjacaff<Z2_t, uint512_t>(&xr, &x1, params->midx);

    return;
}

#if LOG_LEVEL != LOG_LEVEL_NOLOG
__global__ void scmulecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
#else
__global__ void __launch_bounds__(256,2) scmulecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
#endif
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= params->in_length/3) {
     return;
   }

   scl = (uint32_t *) &in_vector[tid * NWORDS_256BIT + ECP_SCLOFFSET];
   Z1_t x1(&in_vector[ params->in_length/3 * NWORDS_256BIT+ tid * ECP_JAC_INOFFSET + ECP_JAC_INXOFFSET]);
   Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z1_t, uint256_t>(&xr,0, &x1,0, scl,  params);

   return;
}

__global__ void sc1mulecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   logInfoTid("Length : %d\n", params->in_length);
   logInfoTid("R Length : %d\n", params->in_length-ECP_JAC_INDIMS);
   logInfoTid("SCL O %d\n", ECP_SCLOFFSET);
   if(tid >= params->in_length-ECP_JAC_INDIMS) {
     return;
   }

   scl = (uint32_t *) &in_vector[tid * NWORDS_256BIT + ECP_SCLOFFSET];
   logInfoBigNumberTid(1,"SCL MONT\n", scl);
   // confert from montgomery if necessary
   if (params->premul){
      uint32_t One[NWORDS_256BIT] = {1,0,0,0,0,0,0,0};
      mulmontu256(scl, scl, One, params->midx);
   } 
   logInfoBigNumberTid(1,"SCL EXT\n", scl);

   Z1_t x1(&in_vector[(params->in_length-ECP_JAC_INDIMS)*NWORDS_256BIT + ECP_JAC_INXOFFSET]);
   Z1_t xr(&out_vector[tid * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z1_t, uint256_t>(&xr,0, &x1,0, scl,  params);

   return;
}

#if LOG_LEVEL != LOG_LEVEL_NOLOG
__global__ void scmulec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
#else
__global__ void __launch_bounds__(128,2) scmulec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
#endif
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= params->in_length/5) {
     return;
   }

   scl = (uint32_t *) &in_vector[tid * NWORDS_256BIT + ECP_SCLOFFSET];
   Z2_t x1(&in_vector[ params->in_length/5 * NWORDS_256BIT+ tid * ECP2_JAC_INOFFSET + ECP2_JAC_INXOFFSET]);
   Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);
  
   scmulecjac<Z2_t, uint512_t>(&xr,0, &x1,0, scl,params);

   return;
}

__global__ void sc1mulec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *scl;
 
   if(tid >= params->in_length-ECP2_JAC_INDIMS) {
     return;
   }

   scl = (uint32_t *) &in_vector[tid * NWORDS_256BIT + ECP_SCLOFFSET];
   // confert from montgomery if necessary
   if (params->premul){
      uint32_t One[NWORDS_256BIT] = {1,0,0,0,0,0,0,0};
      mulmontu256(scl, scl, One, params->midx);
   }
   Z2_t x1(&in_vector[(params->in_length-ECP2_JAC_INDIMS)*NWORDS_256BIT + ECP2_JAC_INXOFFSET]);
   Z2_t xr(&out_vector[tid * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);
   logInfoBigNumberTid(1,"SCL\n", scl);
   logInfoBigNumberTid(4,"X1\n", &x1);

   scmulecjac<Z2_t, uint512_t>(&xr,0, &x1,0, scl,  params);

   return;
}

__global__ void madecjac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    uint32_t debug_idx = 0;

    extern __shared__ uint32_t smem[];
    Z1_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl;
   
    if(idx >= params->in_length/params->stride) {
      return;
    }
    logInfoTid("Min Padding : %d\n",params->padding_idx);
    logInfoTid("Max Padding : %d\n",params->in_length/ECP_JAC_OUTDIMS);
    if (params->padding_idx){
       uint32_t padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&in_vector[idx * ECP_JAC_INOFFSET],padding);
          movu256(&in_vector[idx * ECP_JAC_INOFFSET + NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP_JAC_INOFFSET + 2*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }

    Z1_t xo, xr;
    if (params->premul){
      xo.assign(&in_vector[params->in_length/3 * NWORDS_256BIT + idx  * (params->stride-1) * NWORDS_256BIT + ECP_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = (uint32_t *) &in_vector[idx * params->stride/3 *  NWORDS_256BIT];
      logInfoTid("LE : %d\n",params->in_length);
      logInfoTid("InVO : %d\n",(params->stride-1) * NWORDS_256BIT);
      logInfoTid("SclVO : %d\n",params->in_length/3* 2 * NWORDS_256BIT);
    } else {
      xo.assign(&in_vector[idx  * (params->stride) * NWORDS_256BIT + ECP_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = NULL;
      logInfoTid("LE : %d\n",params->in_length);
      logInfoTid("InVO : %d\n",(params->stride) * NWORDS_256BIT);
      logInfoTid("SclVO : %d\n",params->in_length/3* 2 *NWORDS_256BIT);
    }
    xr.assign(&in_vector[blockIdx.x * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);  // 
    if (gridDim.x == 1){
      xr.assign(out_vector);
    }

    madecjac<Z1_t, uint256_t>(&xr, &xo, scl, &zsmem, params);
}

__global__ void madec2jac_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    uint32_t debug_idx = 0;

    extern __shared__ uint32_t smem[];
    Z2_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl;
   
    if(idx >= params->in_length/params->stride) {
      return;
    }
    logInfoTid("Min Padding : %d\n",params->padding_idx);
    logInfoTid("Max Padding : %d\n",params->in_length/ECP2_JAC_OUTDIMS);
    if (params->padding_idx){
       uint32_t padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP2_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET+ NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 2*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 3*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 4*NWORDS_256BIT],padding);
          movu256(&in_vector[idx * ECP2_JAC_OUTOFFSET + 5*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }

    Z2_t xo, xr;
    if (params->premul){
      xo.assign(&in_vector[params->in_length/5 * NWORDS_256BIT + idx  * params->stride * NWORDS_256BIT + ECP2_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = (uint32_t *) &in_vector[idx * params->stride/5 *  NWORDS_256BIT];
      logInfoTid("LE : %d\n",params->in_length);
      logInfoTid("InVO : %d\n",(params->stride) * NWORDS_256BIT);
      logInfoTid("SclVO : %d\n",params->in_length/5* NWORDS_256BIT);
    } else {
      xo.assign(&in_vector[idx  * (params->stride) * NWORDS_256BIT + ECP2_JAC_OUTXOFFSET]); // 0 .. N-1
      scl = NULL;
      logInfoTid("LE : %d\n",params->in_length);
      logInfoTid("InVO : %d\n",(params->stride) * NWORDS_256BIT);
      logInfoTid("SclVO : %d\n",params->in_length/6 *NWORDS_256BIT);
    }
    xr.assign(&in_vector[blockIdx.x * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);  // 
    if (gridDim.x == 1){
      xr.assign(out_vector);
    }

    madecjac<Z2_t, uint512_t>(&xr, &xo, scl, &zsmem, params);
}
__global__ void madecjac_shfl_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t poffset = 0;

    extern __shared__ uint32_t smem[];
    Z1_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl = NULL;
 
    if(idx >= params->in_length/params->stride) {
      return;
    }
    /*
    logInfoTid("Min Padding : %d\n",params->padding_idx);
    logInfoTid("Max Padding : %d\n",params->in_length/ECP_JAC_OUTDIMS);
    logInfoTid("OUt : %d\n",params->in_length/params->stride);
    if (params->padding_idx){
       uint32_t __align__(16) padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&out_vector[idx * ECP_JAC_OUTOFFSET],padding);
          movu256(&out_vector[idx * ECP_JAC_OUTOFFSET + NWORDS_256BIT],padding);
          movu256(&out_vector[idx * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT],padding);
       }
    }
    __syncthreads();
    */

    Z1_t xo;
    if (params->premul){
      scl = (uint32_t *) in_vector;
      poffset = params->in_length/3 * NWORDS_256BIT;
      xo.assign(&in_vector[poffset]);
      logInfoBigNumberTid(1,"SCL in \n",scl);
    } else {
      xo.assign(&out_vector[idx * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]); // 0 .. N-1
    }

    Z1_t xr(&out_vector[blockIdx.x * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);  // 
    //Z1_t xr(&out_vector[idx * ECP_JAC_OUTOFFSET + ECP_JAC_OUTXOFFSET]);  // 
    //Z1_t xr(&out_vector[0]);
  
    madecjac_shfl<Z1_t, uint256_t>(&xr, &xo, scl, &zsmem, params);
}

__global__ void madec2jac_shfl_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t poffset = 0;

    extern __shared__ uint32_t smem[];
    Z2_t zsmem(smem);  // 0 .. blockDim

    uint32_t __restrict__ *scl = NULL;
  
    if(idx >= params->in_length/params->stride) {
      return;
    }
    /*
    logInfoTid("Min Padding : %d\n",params->padding_idx);
    logInfoTid("OUt : %d\n",params->in_length/params->stride);
    if (params->padding_idx){
       uint32_t padding[] = {0,0,0,0,0,0,0,0};
       // add zeros between padding and next multiple of 32
       if (idx < params->in_length/ECP2_JAC_OUTDIMS && idx >= params->padding_idx){
          movu256(&out_vector[idx * ECP2_JAC_OUTOFFSET],padding);
          movu256(&out_vector[idx * ECP2_JAC_OUTOFFSET + NWORDS_256BIT],padding);
          movu256(&out_vector[idx * ECP2_JAC_OUTOFFSET + 2*NWORDS_256BIT],padding);
          movu256(&out_vector[idx * ECP2_JAC_OUTOFFSET + 3*NWORDS_256BIT],padding);
          movu256(&out_vector[idx * ECP2_JAC_OUTOFFSET + 4*NWORDS_256BIT],padding);
          movu256(&out_vector[idx * ECP2_JAC_OUTOFFSET + 5*NWORDS_256BIT],padding);
       }
       __syncthreads();
    }
    */

    Z2_t xo;
    if (params->premul){
      scl = (uint32_t *) in_vector;
      poffset = params->in_length/5 * NWORDS_256BIT;
      xo.assign(&in_vector[poffset]);
    } else {
      xo.assign(&out_vector[idx * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]); // 0 .. N-1
    }

    Z2_t xr(&out_vector[blockIdx.x * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);  // 
    //Z2_t xr(&out_vector[idx * ECP2_JAC_OUTOFFSET + ECP2_JAC_OUTXOFFSET]);  // 

    madecjac_shfl<Z2_t, uint512_t>(&xr, &xo, scl, &zsmem, params);
}


template<typename T1, typename T2>
__forceinline__ __device__ void madecjac(T1 *xr, T1 *xo, uint32_t *scl, T1 *smem_ptr, kernel_params_t *params)
{
    uint32_t i;
    //uint32_t ndbg = T1::getN();
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    logInfoTid("stride :%d\n",params->stride/ECP_JAC_OUTDIMS);
    // scalar multipliation
    if (params->premul){
        #pragma unroll
        for (i =0; i < params->stride/ECP_JAC_OUTDIMS; i++){
          /*
          logInfoBigNumberTid(1,"scl :\n",&scl[i*ECP_JAC_INOFFSET]);
          logInfoBigNumberTid(T1::getN(),"Xin[x]:\n",xos(i*ECP_JAC_INDIMS));
          logInfoBigNumberTid("Xin[y]:\n",&xi[i*ECP_JAC_INOFFSET + NWORDS_256BIT]);
          */

          scmulecjac<T1, T2>(xr,i*ECP_JAC_OUTDIMS, xo, i*ECP_JAC_INDIMS, &scl[i*NWORDS_256BIT],  params);
          

          /*
          logInfoBigNumberTid(1,"Xout[x]:\n",&xo[i*ECP_JAC_OUTOFFSET]);
          logInfoBigNumberTid(1,"Xout[y]:\n",&xo[i*ECP_JAC_OUTOFFSET + NWORDS_256BIT]);
          logInfoBigNumberTid(1,"Xout[z]:\n",&xo[i*ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
          */
        }
    }
   
    if (params->stride/ECP_JAC_OUTDIMS > 1){ 
      addecjac<T1,T2>(smem_ptr,tid*ECP_JAC_OUTDIMS, xr,0, xr,ECP_JAC_OUTDIMS, params->midx);
      /*
      logInfoBigNumberTid(1,"smem[X]\n",smem_ptr);
      logInfoBigNumberTid(1,"smem[Y]\n",&smem_ptr[NWORDS_256BIT]);
      logInfoBigNumberTid(1,"smem[Z]\n",&smem_ptr[2*NWORDS_256BIT]);
      */

      #pragma unroll
      for (i =0; i < params->stride/ECP_JAC_OUTDIMS-2; i++){
        addecjac<T1,T2>(smem_ptr,tid*ECP_JAC_OUTDIMS, smem_ptr, 0,xr, (i+2)*ECP_JAC_OUTDIMS, params->midx);
        /*
        logInfoBigNumberTid(1,"smem[X]\n",smem_ptr);
        logInfoBigNumberTid(1,"smem[Y]\n",&smem_ptr[NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[Z]\n",&smem_ptr[2*NWORDS_256BIT]);
        */
      }
      __syncthreads();
    }
  
    //logDebugBigNumberTid(1,"smem[i]\n",smem_ptr);
    // reduction global mem
    if (blockDim.x >= 1024 && tid < 512){
      /*
      logInfoBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(1,"+smem[512]\n",&smem[(tid+512)*NWORDS_256BIT]);
      */
      
      addecjac<T1,T2>(smem_ptr, tid*ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+512)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(1,"smem[0]\n",smem_ptr);
      */
    }
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256){
      /*
      logInfoBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(1,"+smem[256]\n",&smem[(tid+256)*NWORDS_256BIT]);
      */
      addecjac<T1,T2>(smem_ptr, tid * ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+256)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(1,"smem[=256]\n",smem_ptr);
      */
    }
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128){
      /*
      logInfoBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(1,"+smem[128]\n",&smem[(tid+128)*NWORDS_256BIT]);
      */
      addecjac<T1,T2>(smem_ptr, tid * ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+128)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(1,"smem[=128+0]\n",smem_ptr);
      */
    }
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64){
      /*
      logInfoBigNumberTid(1,"+smem[0]\n",smem_ptr);
      logInfoBigNumberTid(1,"+smem[64]\n",&smem[(tid+64)*NWORDS_256BIT]);
      */
      addecjac<T1,T2>(smem_ptr, tid * ECP_JAC_INDIMS,
               smem_ptr,tid * ECP_JAC_INDIMS,
               smem_ptr, (tid+64)*ECP_JAC_INDIMS, params->midx);
      /*
      logInfoBigNumberTid(1,"smem[=64+0]\n",smem_ptr);
      */
    }
    __syncthreads();
      
    // unrolling warp

    if (tid < 32)
    {
        //volatile uint32_t *vsmem = smem_ptr;
        uint32_t *zvsmem = smem_ptr->getu256();
        T1 vsmem(zvsmem);
 
        /*
        logInfoBigNumberTid(1,"smem[pre32X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[pre32Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[pre32Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+32)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(1,"smem[32X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[32Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[32Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);

        logInfoBigNumberTid(1,"smem[pre16X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[pre16Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[pre16Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+16)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(1,"smem[16X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[16Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[16Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+8)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(1,"smem[8X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[8Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[8Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+4)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(1,"smem[4X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[4Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[4Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+2)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(1,"smem[2X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[2Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[2Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        addecjac<T1, T2>(&vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,tid * ECP_JAC_OUTDIMS,
                 &vsmem,(tid+1)*ECP_JAC_OUTDIMS, params->midx);

        /*
        logInfoBigNumberTid(1,"smem[X]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET]);
        logInfoBigNumberTid(1,"smem[Y]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET+NWORDS_256BIT]);
        logInfoBigNumberTid(1,"smem[Z]\n",(uint32_t *)&vsmem[tid * ECP_JAC_OUTOFFSET + 2*NWORDS_256BIT]);
        */

        if (tid==0) {
           xr->setu256(0,smem_ptr,0);
        }
    }

  return;
}

template<typename T1, typename T2>
__forceinline__ __device__ void madecjac_shfl(T1 *xr, T1 *xo, uint32_t *scl, T1 *smem_ptr, kernel_params_t *params)
{
    uint32_t i,  size2;
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    uint32_t __align__(16) zsumX[ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint32_t)];
    uint32_t __align__(16) zsumY[ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint32_t)];
    uint32_t laneIdx = tid % warpSize;
    uint32_t warpIdx = tid / warpSize;
    uint32_t opt_ec_offset = params->in_length/params->stride - 
                               params->in_length/(params->stride * U256_BSELM) * U256_BSELM;
    T1 sumX(zsumX);
    T1 sumY(zsumY);

    size2 = blockDim.x >> 6;
    // scalar multiplication
    if (params->premul){
        sumX.setu256(0,xo,idx*2,1);
        sumX.setu256(1,xo,idx*2+1,1);
        if (idx < opt_ec_offset || params->padding_idx == 0){
          logInfoBigNumberTid(1,"scl :\n",&scl[idx * NWORDS_256BIT]);
 
          logInfoBigNumberTid(2*T1::getN(),"Xin[x,y,z]:\n",&sumX);
          scmulecjac<T1, T2>(&sumX,0, &sumX, 0, &scl[idx * NWORDS_256BIT],  params);
          
          logInfoBigNumberTid(3*T1::getN(),"Xout[x,y,z]:\n",&sumX);
        } else {
           T1 sumX2(xo->getu256(0));
           logInfoBigNumberTid(2*T1::getN(),"Xo[x,y]:\n",&sumX2);
      
           scmulecjac_opt<T1, T2>(&sumX,0,
                                  &sumX2, 2*(idx*U256_BSELM - opt_ec_offset), 
                                  &scl[idx*NWORDS_256BIT *U256_BSELM],  params);
           logInfoBigNumberTid(3*T1::getN(),"Final R : \n",&sumX);
        }
        
    } else {
        sumX.setu256(0,xo,0);
        logInfoBigNumberTid(3*T1::getN(),"Xout[x,y,z]:\n",&sumX);
    }


    __syncthreads();
 
    // block wide warp reduce
    #pragma unroll
    for (i = WARP_HALF_SIZE; i > 0; i >>= 1){
      shflxoruecc<T1,T2>(&sumY, &sumX, i);
      logInfoTid("idx:%d\n",i);
      logInfoBigNumberTid(3*T1::getN(),"sumX1\n",&sumX);
      logInfoBigNumberTid(3*T1::getN(),"sumY1\n",&sumY);
     
      addecjac<T1,T2>(&sumX,0, &sumX,0, &sumY,0, params->midx);
      logInfoBigNumberTid(3*T1::getN(),"sumX1+\n",&sumX);

    }

    __syncthreads();
    if (laneIdx == 0) {
       smem_ptr->setu256(warpIdx*ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint256_t), &sumX,0);
       logInfoTid("save idx:%d\n",warpIdx);
       logInfoBigNumberTid(3*T1::getN(),"val\n",&sumX);
    }

    __syncthreads();
  
    if (size2){

      logInfoBigNumberTid(3*T1::getN(),"Smem\n",smem_ptr);
      if (tid < size2*2) {
        logInfoTid("blockDim :%d\n",blockDim.x);
        logInfoTid("LaneIdx :%d\n",laneIdx);
        logInfoTid("Size :%d\n",size2);
  
        sumX.setu256(0,smem_ptr,laneIdx*ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint256_t));
      } else {
        T1 _inf;
        infz(&_inf, params->midx);
        sumX.setu256(0,&_inf,0);
      }
      logInfoBigNumberTid(3*T1::getN(),"Second\n",&sumX);
  
      #pragma unroll
      for (i=size2; i > 0; i >>=1){
        shflxoruecc<T1,T2>(&sumY, &sumX, i);
        logInfoTid("idx:%d\n",i);
        logInfoBigNumberTid(3*T1::getN(),"sumY\n",&sumY);
        logInfoBigNumberTid(3*T1::getN(),"sumX\n",&sumX);

        addecjac<T1,T2>(&sumX,0, &sumX,0, &sumY,0, params->midx);
        logInfoBigNumberTid(3*T1::getN(),"sumX+\n",&sumX);
      }
    }

    if (tid==0) {
     xr->setu256(0,&sumX,0);
     logInfoBigNumberTid(3*T1::getN(),"Z-sumX : \n",&sumX);
    } 

  return;
}


/*
  EC point addition
  
  Algorithm (https://en.wikibooks.org/wiki/Cryptography/Prime_Curve/Jacobian_Coordinates):
  IN : P1(X1,Y1,Z1), P2(X2,Y2,Z2)
  OUT: P3(X3,Y3,Z3)

    U1 = X1*Z2^2
    U2 = X2*Z1^2
    S1 = Y1*Z2^3
    S2 = Y2*Z1^3
    if (U1 == U2)
      if (S1 != S2)
        return POINT_AT_INFINITY
      else 
        return POINT_DOUBLE(X1, Y1, Z1)
    H = U2 - U1
    R = S2 - S1
    X3 = R^2 - H^3 - 2*U1*H^2
    Y3 = R*(U1*H^2 - X3) - S1*H^3
    Z3 = H*Z1*Z2
    return (X3, Y3, Z3)

    TODO . check if I can remove
    NOTE X1 cannot be 0. X2 can from two sources: 
      - reduction -> when there is not enough input data, i append with 0 and put it in second addition term
      - scalar multiplication -> if first bit of scalar is 0, I add 0
*/
template<typename T1, typename T2>
__forceinline__ __device__ void addecjac(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t x1offset, T1 *zx2, uint32_t x2offset, mod_t midx)
{
  T1 x1(zx1->getu256(0+x1offset)), y1(zx1->getu256(1+x1offset)), z1(zx1->getu256(2+x1offset));
  T1 x2(zx2->getu256(0+x2offset)), y2(zx2->getu256(1+x2offset)), z2(zx2->getu256(2+x2offset));
  T1 xr(zxr->getu256(0+zoffset)),  yr(zxr->getu256(1+zoffset)), zr(zxr->getu256(2+zoffset));
 
  uint32_t __restrict__ ztmp[5*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(ztmp), tmp3(&ztmp[1*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_x(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_y(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]),
                 tmp_z(&ztmp[4*sizeof(T2)/sizeof(uint32_t)]);

  // TODO : Change definition of inf to 0, 1, 0 instead of 1,0,1 as it is now
  /*
  logInfoBigNumberTid(T1::getN(),"x1\n",x1.getu256());
  logInfoBigNumberTid(T1::getN(),"y1\n",y1.getu256());
  logInfoBigNumberTid(T1::getN(),"z1\n",z1.getu256());
  logInfoBigNumberTid(T1::getN(),"x2\n",x2.getu256());
  logInfoBigNumberTid(T1::getN(),"y2\n",y2.getu256());
  logInfoBigNumberTid(T1::getN(),"z2\n",z2.getu256());
  */

  if (eq0z(&z2)){ 
      zxr->setu256(T1::getN()*zoffset,zx1,T1::getN()*x1offset);
      logInfoTid("R1=X1 x1offset : %d\n",x1offset);
      logInfoTid("R1=X1 x2offset : %d\n",x2offset);
      logInfoTid("R1=X1 zoffset : %d\n",zoffset);
      logInfoBigNumberTid(3*T1::getN(),"XR\n",zxr);
      logInfoBigNumberTid(3*T1::getN(),"X1\n",zx1);
      logInfoBigNumberTid(3*T1::getN(),"X2\n",zx2);
      return;  
  }
  if (eq0z(&z1)){ 
      /*
      uint32_t R[3*sizeof(T2)/sizeof(uint32_t)]; 
      T1 RR(R);
      RR.setu256(0,zxr,T1::getN()*zoffset);
      logInfoBigNumberTid(3*T1::getN(),"prev XR\n",&RR);
      RR.setu256(0,zx1,T1::getN()*x1offset);
      logInfoBigNumberTid(3*T1::getN(),"prev X1\n",&RR);
      RR.setu256(0,zx2,T1::getN()*x2offset);
      logInfoBigNumberTid(3*T1::getN(),"prev X2\n",&RR);
      */
      zxr->setu256(T1::getN()*zoffset,zx2,T1::getN()*x2offset);
      /*
      logInfoTid("R1=X2 x2offset : %d\n",x2offset);
      logInfoTid("R1=X2 x1offset : %d\n",x1offset);
      logInfoTid("R1=X2 zoffset  : %d\n",zoffset);
      RR.setu256(0,zxr,T1::getN()*zoffset);
      logInfoBigNumberTid(3*T1::getN(),"XR\n",&RR);
      RR.setu256(0,zx1,T1::getN()*x1offset);
      logInfoBigNumberTid(3*T1::getN(),"X1\n",&RR);
      RR.setu256(0,zx2,T1::getN()*x2offset);
      logInfoBigNumberTid(3*T1::getN(),"X2\n",&RR);
      */
      return;  
  }
  squarez(&tmp_x, &z1,         midx);  // tmp_x = z1sq 
  mulz(&tmp_z, &tmp_x, &x2, midx);  // tmp_z = u2 = x2 * z1sq
  mulz(&tmp_x, &tmp_x, &z1, midx);  // tmp_x = z1cube
  mulz(&tmp_x, &tmp_x, &y2, midx);  // tmp_x = s2 = z1cube * y2
  squarez(&tmp_y, &z2,        midx);  // tmp_y = z2sq
  mulz(&tmp1, &x1, &tmp_y, midx);  // tmp1 = u1 = x1 * z2sq
  mulz(&tmp_y, &tmp_y, &z2, midx);  // tmp_y = z2cube
  mulz(&tmp_y, &tmp_y, &y1, midx);  // tmp_y = s1 = z2cube * y1

  //  if U1 == U2 and S1 == S2 => P1 = P2 -> double
  //  if U1 == U2 and S1 != S2 => P1 = -P2 -> return 0
  //  instead of calling double,  i proceed. It is better to avoid warp divergence
  if (eqz(&tmp1, &tmp_z)) {    // u1 == u2
      if (!eqz( &tmp_y, &tmp_x)){  // s1 != s2
          T1 _inf;
          infz(&_inf, midx);
          zxr->setu256(zoffset,&_inf,x1offset);
	  return;  
      }
      doublecjac<T1, T2>(zxr,zx1, midx);
      return;
  }

  subz(&tmp_z, &tmp_z, &tmp1, midx);     // H = tmp2 = u2 - u1
  mulz(&zr, &z1, &z2, midx);      // tmp_z = z1 * z2
  mulz(&zr, &zr, &tmp_z, midx);       // zr = z1 * z2  * h

  /*
  logInfoBigNumberTid(T1::getN(),"H\n",&tmp2);
  logInfoBigNumberTid(T1::getN(),"z1 * z2\n",&tmp_z);
  logInfoBigNumberTid(T1::getN(),"z1 * z2  * h\n",&zr);
  */

  squarez(&tmp3, &tmp_z,        midx);     // Hsq = tmp3 = H * H 
  mulz(&tmp_z, &tmp3, &tmp_z, midx);     // Hcube = tmp2 = Hsq * H 
  mulz(&tmp1, &tmp1, &tmp3, midx);     // tmp1 = u1 * Hsq

  /*
  logInfoBigNumberTid(T1::getN(),"Hsq\n",&tmp3);
  logInfoBigNumberTid(T1::getN(),"H3\n",&tmp2);
  logInfoBigNumberTid(T1::getN(),"Hsq * u1\n",&tmp1);
  */

  subz(&tmp3, &tmp_x, &tmp_y, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, tmp_x=free, tmp_y=s1, zr=zr
  mulz(&tmp_y, &tmp_y, &tmp_z, midx);     // tmp_y = Hcube * s1
  squarez(&tmp_x, &tmp3, midx);     // tmp_x = R * R

  /*
  logInfoBigNumberTid(T1::getN(),"R\n",&tmp3);
  logInfoBigNumberTid(T1::getN(),"Hcube* s1\n",&tmp_y);
  logInfoBigNumberTid(T1::getN(),"Rsq * u1\n",&tmp_x);
  */

  subz(&tmp_x, &tmp_x, &tmp_z, midx);        // tmp_x = x3= (R*R)-Hcube, tmp_y = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  mul2z(&xr, &tmp1, midx);     // tmp4 = u1*hsq *_2

  /*
  logInfoBigNumberTid(T1::getN(),"Rsq - H3\n",&tmp_x);
  logInfoBigNumberTid(T1::getN(),"Hsq * 2 * u1\n",&tmp4);
  */

  subz(&xr, &tmp_x, &xr, midx);               // x3 = xr
  subz(&tmp1, &tmp1, &xr, midx);       // tmp1 = u1*hs1 - x3
  mulz(&tmp1, &tmp1, &tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)
  subz(&yr, &tmp1, &tmp_y, midx);

  /*
  logInfoBigNumberTid(T1::getN(),"X : \n",&xr);
  logInfoBigNumberTid(T1::getN(),"Y : \n",&yr);
  logInfoBigNumberTid(T1::getN(),"Z : \n",&zr);
  */
}

template<typename T1, typename T2>
__forceinline__ __device__ void addecjacmixed(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t x1offset, T1 *zx2, uint32_t x2offset, mod_t midx)
{
  T1 x1(zx1->getu256(0+x1offset)), y1(zx1->getu256(1+x1offset));
  T1 x2(zx2->getu256(0+x2offset)), y2(zx2->getu256(1+x2offset)), z2(zx2->getu256(2+x2offset));
  T1 xr(zxr->getu256(0+zoffset)),  yr(zxr->getu256(1+zoffset)), zr(zxr->getu256(2+zoffset));
 
  uint32_t __restrict__ ztmp[4*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(&ztmp[0*sizeof(T2)/sizeof(uint32_t)]), 
     tmp3(&ztmp[1*sizeof(T2)/sizeof(uint32_t)]),
     tmp_x(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]),
     tmp_z(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]);

  // TODO : Change definition of inf to 0, 1, 0 instead of 1,0,1 as it is now
  /*
  logInfoBigNumberTid(T1::getN(),"x1\n",x1.getu256());
  logInfoBigNumberTid(T1::getN(),"y1\n",y1.getu256());
  logInfoBigNumberTid(T1::getN(),"x2\n",x2.getu256());
  logInfoBigNumberTid(T1::getN(),"y2\n",y2.getu256());
  logInfoBigNumberTid(T1::getN(),"z2\n",z2.getu256());
  */

  if (eq0z(&z2)){ 
      zxr->setu256(T1::getN()*zoffset,zx1,T1::getN()*x1offset);
      logInfoTid("R1=inf %d\n", midx);
      return;  
  }
  squarez(&tmp_x, &z2,         midx);  // tmp_x = z2sq 
  mulz(&tmp_z, &tmp_x, &x1, midx);  // tmp_z = u1 = x1 * z2sq
  mulz(&tmp_x, &tmp_x, &z2, midx);  // tmp_x = z2cube
  mulz(&tmp_x, &tmp_x, &y1, midx);  // tmp_x = s1 = z2cube * y1

  //  if U1 == U2 and S1 == S2 => P1 = P2 -> double
  //  if U1 == U2 and S1 != S2 => P1 = -P2 -> return 0
  //  instead of calling double,  i proceed. It is better to avoid warp divergence
  if (eqz(&x2, &tmp_z)){    // u1 == u2
       if (!eqz( &y2, &tmp_x)){  // s1 != s2
          T1 _inf;
          infz(&_inf, midx);
          zxr->setu256(T1::getN()*zoffset,&_inf,0);
          logInfoTid("R2=inf %d\n", midx);
	  return;  
        } 
        //logInfoTid("R4=D %d\n", midx);
        doublecjac<T1, T2>(zxr,zx2, midx);
        return;
  }
  subz(&tmp1, &x2, &tmp_z, midx);     // H = tmp1 = u2 - u1
  mulz(&zr, &z2, &tmp1, midx);       // zr = z1 * z2  * h
  squarez(&tmp3, &tmp1,        midx);     // Hsq = tmp3 = H * H 

  /*
  logInfoBigNumberTid(T1::getN(),"H\n",&tmp1);
  logInfoBigNumberTid(T1::getN(),"z2 * h\n",&zr);
  logInfoBigNumberTid(T1::getN(),"Hsq\n",&tmp3);
  */

  mulz(&tmp1, &tmp3, &tmp1, midx);     // Hcube = tmp1= Hsq * H 
  mulz(&tmp3, &tmp_z, &tmp3, midx);     // tmp3 = u1 * Hsq

  /*
  logInfoBigNumberTid(T1::getN(),"H3\n",&tmp1);
  logInfoBigNumberTid(T1::getN(),"Hsq * u1\n",&tmp3);
  */

  subz(&tmp_z, &y2, &tmp_x, midx);        // R = tmp_z = S2 - S1 
  squarez(&xr, &tmp_z, midx);     // xr = Rsq

  /*
  logInfoBigNumberTid(T1::getN(),"R\n",&tmp_z);
  logInfoBigNumberTid(T1::getN(),"Rsq\n",&xr);
  */

  subz(&xr, &xr, &tmp1, midx);     // xr = Rsq - Hcube

  //logInfoBigNumberTid(T1::getN(),"Rsq - Hcube\n",&xr);

  subz(&xr, &xr, &tmp3, midx);     // xr = Rsq - Hcube - u1*Hsq
  subz(&xr, &xr, &tmp3, midx);     // xr = Rsq - Hcube - 2*u1*Hsq

  //logInfoBigNumberTid(T1::getN(),"X\n",&xr);

  subz(&yr, &tmp3, &xr, midx);          // yr = u1*Hsq - xr
  mulz(&yr, &yr, &tmp_z, midx);        //  yr = R*(u1*Hsq - xr)

  mulz(&tmp1, &tmp1, &tmp_x, midx);     // tmp1 = Hcube * s1
  subz(&yr, &yr, &tmp1, midx);

  /*
  logInfoBigNumberTid(T1::getN(),"X : \n",&xr);
  logInfoBigNumberTid(T1::getN(),"Y : \n",&yr);
  logInfoBigNumberTid(T1::getN(),"Z : \n",&zr);
  */
}

/*
  input is in affine coordinates -> P(Z) = 1
  I can do Q = Q+Y or Q = Y + Q
    NOTE X1, X2 cannot be 0
*/

template <typename T1, typename T2>
__forceinline__ __device__ void addecjacaff(T1  *zxr, T1 *zx1, T1 *zx2, mod_t midx)
{
  T1 y1(zx1->getu256(ECP_JAC_YOFFSET_BASE)), y2(zx2->getu256(ECP_JAC_YOFFSET_BASE)),
     xr(zxr->getu256(ECP_JAC_XOFFSET_BASE)),
     yr(zxr->getu256(ECP_JAC_YOFFSET_BASE)), zr(zxr->getu256(ECP_JAC_ZOFFSET_BASE));

  uint32_t __restrict__ ztmp[4*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp1(ztmp), tmp2(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp3(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]), 
                 tmp4(&ztmp[3*sizeof(T2)/sizeof(uint32_t)]);

  // TODO Check if I can call add to compute x + x (instead of double)
  //  if not, I should call double below. I don't want to to avoid warp divergnce
  if (eqz(zx1, zx2)){   // u1 == u2
      if (!eqz( &y1,  &y2)){  // s1 != s2
          T1 _inf;
          infz(&_inf, midx);
          zxr->setu256(0,&_inf,0);
	  return;  //  if U1 == U2 and S1 == S2 => P1 == P2 (call double)
     }
     doublecjacaff<T1, T2>(zxr,zx1, midx);
     return;
  }

  logInfoBigNumberTid(T1::getN(),"x1\n",zx1);
  logInfoBigNumberTid(T1::getN(),"y1\n",&y1);
  logInfoBigNumberTid(T1::getN(),"x2\n",zx2);
  logInfoBigNumberTid(T1::getN(),"y2\n",&y2);

  subz(&zr, zx2, zx1, midx);     // H = tmp2 = u2 - u1
  
  logInfoBigNumberTid(T1::getN(),"H\n",&zr);

  squarez(&tmp3, &zr,        midx);     // Hsq = tmp3 = H * H 
  mulz(&tmp2, &tmp3, &zr, midx);     // Hcube = tmp2 = Hsq * H 
  mulz(&tmp1, zx1, &tmp3, midx);     // tmp1 = u1 * Hsq

  logInfoBigNumberTid(T1::getN(),"Hsq\n",&tmp3);
  logInfoBigNumberTid(T1::getN(),"Hcube\n",&tmp2);
  logInfoBigNumberTid(T1::getN(),"u1 * Hsq\n",&tmp1);

  subz(&tmp3, &y2, &y1, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, xr=free, yr=s1, zr=zr
  mulz(&yr, &y1, &tmp2, midx);     // yr = Hcube * s1
  squarez(zxr, &tmp3, midx);     // xr = R * R

  logInfoBigNumberTid(T1::getN(),"R\n",&tmp3);
  logInfoBigNumberTid(T1::getN(),"s1\n",&yr);
  logInfoBigNumberTid(T1::getN(),"Rsq\n",&xr);

  subz(zxr, zxr, &tmp2, midx);        // xr = x3= (R*R)-Hcube, yr = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  // TODO muluk256
  mul2z(&tmp4, &tmp1, midx);     // tmp4 = u1*hsq *_2

  logInfoBigNumberTid(T1::getN(),"Rsq - Hcube\n",&xr);
  logInfoBigNumberTid(T1::getN(),"u1 * Hsq * 2\n",&tmp4);

  subz(zxr, &xr, &tmp4, midx);               // x3 = xr
  subz(&tmp1, &tmp1, zxr, midx);       // tmp1 = u1*hs1 - x3

  logInfoBigNumberTid(T1::getN(),"u1*hsq - x3\n",&tmp1);

  mulz(&tmp1, &tmp1, &tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)

  logInfoBigNumberTid(T1::getN(),"r * (u1*hsq - x3)\n",&tmp1);

  subz(&yr, &tmp1, &yr, midx);

  logInfoBigNumberTid(T1::getN(),"X3\n",&xr);
  logInfoBigNumberTid(T1::getN(),"Y3\n",&yr);
  logInfoBigNumberTid(T1::getN(),"Z3\n",&zr);
}

/*
  EC point addition
  
  Algorithm (https://en.wikibooks.org/wiki/Cryptography/Prime_Curve/Jacobian_Coordinates):
  IN : P1(X1,Y1,Z1)
  OUT: P'(X',Y',Z')

   if (Y == 0)
      return POINT_AT_INFINITY
   S = 4*X*Y^2
   M = 3*X^2 + a*Z^4
   X' = M^2 - 2*S
   Y' = M*(S - X') - 8*Y^4
   Z' = 2*Y*Z
   return (X', Y', Z')
*/
template<typename T1, typename T2>
__forceinline__ __device__ void doublecjac(T1 *zxr, T1 *zx1, mod_t midx)
{
  T1 y1(zx1->getu256(1)), z1(zx1->getu256(2));
  T1 yr(zxr->getu256(1)), zr(zxr->getu256(2));

  uint32_t __restrict__ ztmp[2*sizeof(T2)/sizeof(uint32_t)];
  T1 tmp_y(&ztmp[0*sizeof(T2)/sizeof(uint32_t)]), 
     tmp_z(&ztmp[1*sizeof(T2)/sizeof(uint32_t)]);

  // TODO : review this comparison, and see if I can do better. or where I should put it
  // as i check this in several places
  if (eq0z(&z1)){ 
      T1 _inf;
      infz(&_inf,midx);
      zxr->setu256(0,&_inf,0);
      //logInfoTid("R3 Inf : %d\n",midx);
      //memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
      return;  
  }
  squarez(&tmp_z, &y1,            midx);  // tmp_z = ysq
  squarez(&tmp_y, &tmp_z, midx);  // tmp_y = ysqsq
  
  addz(&tmp_y, &tmp_y, &tmp_y, midx);  // tmp_y = ysqsq + ysqsq
  addz(&tmp_y, &tmp_y, &tmp_y, midx);  // tmp_y = 2ysqsq + 2ysqsq
  addz(&tmp_y, &tmp_y, &tmp_y, midx);  // tmp_y = 4ysqsq + 4ysqsq

  mulz(&tmp_z, &tmp_z, zx1, midx);  
  addz(&tmp_z, &tmp_z, &tmp_z, midx);  
  addz(&tmp_z, &tmp_z, &tmp_z, midx);  // S = tmp_z = 2X1Ysq + 2X1Ysq

  mulz(&zr, &y1, &z1, midx);     //  Z3 = Y * Z
  addz(&zr, &zr, &zr, midx);

  squarez(&yr, zx1, midx);           
  addz(zxr, &yr, &yr, midx);       
  addz(&yr, zxr, &yr, midx);       // M = yr = 3Xsq

  squarez(zxr, &yr, midx);       // X3 = Msq

  subz(zxr, zxr, &tmp_z, midx);   // X3 = Msq - S
  subz(zxr, zxr, &tmp_z, midx);      // X3 = Msq - 2S

  subz(&tmp_z, &tmp_z, zxr, midx);   //  tmp_z = S - X3
  mulz(&yr, &yr, &tmp_z, midx);     //  Y3 = M * (S - X3)
  subz(&yr, &yr, &tmp_y, midx);    // Y3 = M * (S - X3) - 8ysqsq


  /*
  logInfoBigNumberTid(T1::getN(),"X : \n",zxr);
  logInfoBigNumberTid(T1::getN(),"Y : \n",&yr);
  logInfoBigNumberTid(T1::getN(),"Z : \n",&zr);
  */
}

/* 
   X1 cannot be 0
 */
template<typename T1, typename T2>
__forceinline__ __device__ void doublecjacaff(T1 *zxr, T1 *zx1, mod_t midx)
{
  uint32_t ztmp[3*sizeof(T2)/sizeof(uint32_t)];
 
  T1 y1(zx1->getu256(1)); 
  T1 yr(zxr->getu256(1)), zr(zxr->getu256(2)); 

  T1 tmp1(ztmp), tmp2(&ztmp[sizeof(T2)/sizeof(uint32_t)]),
                 tmp_y(&ztmp[2*sizeof(T2)/sizeof(uint32_t)]);

  /*
  logInfoBigNumberTid(T1::getN(),"x1\n",zx1->getu256());
  logInfoBigNumberTid(T1::getN(),"y1\n",y1.getu256());
  */
  squarez(&zr, &y1, midx);  // zr = ysq
  squarez(&tmp_y, &zr, midx);  // yr = ysqsq

  /*
  logInfoBigNumberTid(T1::getN(),"ysq\n",zr.getu256());
  logInfoBigNumberTid(T1::getN(),"Yqsq\n",tmp_y.getu256());
  */
  // TODO muluk256
  mul8z(&tmp_y, &tmp_y, midx);  // tmp_y = ysqsq *_8
  mulz(&zr, &zr, zx1, midx);  // S = zr = x * ysq

  /*
  logInfoBigNumberTid(T1::getN(),"8*Ysqsq\n",tmp_y.getu256());
  logInfoBigNumberTid(T1::getN(),"S\n",zr.getu256());
  */
  // TODO muluk256
  mul4z(&zr, &zr, midx);  // S = zr = S * _4

  //logInfoBigNumberTid(T1::getN(),"S*4\n",zr.getu256());

  squarez(zxr, zx1, midx);  // M1 = xr = x * x
  // TODO muluk256
  mul3z(&tmp1, zxr, midx);  // M = tmp1 = M1 * _3

  /*
  logInfoBigNumberTid(T1::getN(),"Xsq\n",zxr->getu256());
  logInfoBigNumberTid(T1::getN(),"M\n",tmp1.getu256());
  */
  squarez(zxr, &tmp1, midx);  // X3 = xr = M * M,  tmp_y = Ysqsq * _8, zr = S; tmp1 = M
  // TODO muluk256
  mul2z(&tmp2, &zr, midx);   // tmp2 = S * _2

  /* 
  logInfoBigNumberTid(T1::getN(),"M*M\n",zxr->getu256());
  logInfoBigNumberTid(T1::getN(),"S*2\n",tmp2.getu256());
  */

  subz(zxr, zxr, &tmp2, midx);      // X3 = xr; tmp_y = Ysqsq * _8, zr = S, tmp1 = M, 
  subz(&tmp2, &zr, zxr, midx);   //  tmp2 = S - X3

  /*
  logInfoBigNumberTid(T1::getN(),"X3\n",zxr->getu256());
  logInfoBigNumberTid(T1::getN(),"S-X3\n",tmp2.getu256());
  */

  mulz(&tmp2, &tmp2, &tmp1, midx); // tmp2 = M * (S - X3)
  //logInfoBigNumberTid(T1::getN(),"M * (S-X3)\n",tmp2.getu256());
  // TODO muluk256
  mul2z(&zr, &y1, midx);
  subz(&yr, &tmp2, &tmp_y, midx);

  /*
  logInfoBigNumberTid(T1::getN(),"y3\n",yr.getu256());
  logInfoBigNumberTid(T1::getN(),"z3\n",zxr->getu256());
  */
}

template<typename T1, typename T2>
__forceinline__ __device__ void scmulecjac(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t xoffset, uint32_t *scl, kernel_params_t *params)
{
  uint32_t i;
  mod_t midx = params->midx;

  uint32_t __restrict__ zN[3*sizeof(T2)/sizeof(uint32_t)]; // N = P
  uint32_t *_1 = misc_const_ct[midx]._1;
  T1 N(zN);
  T1 Q(zxr->getu256(zoffset));

  T1 _inf;
  infz(&_inf, midx);

  N.setu256(0,zx1,xoffset,1);
  N.setu256(1,zx1,xoffset+1,1);
  setkz(&N,2,_1);

  T1 x1(N.getu256(xoffset+0));
  T1 y1(N.getu256(xoffset+1));

  zxr->setu256(0,&_inf,0);

  //logInfoBigNumberTid(3*T1::getN(),"X: \n",&N);
  // TODO : revew this comparison
  if ( (eq0z(&x1) && eqz(&y1,_1)) || eq0u256(scl)){ 
      logInfoBigNumberTid(T1::getN(),"X1 : \n",&x1);
      logInfoBigNumberTid(T1::getN(),"Y1 : \n",&y1);
      logInfoBigNumberTid(3*T1::getN(),"ZX1 : \n",zx1);
      logInfoBigNumberTid(1,"scl : \n",scl);
      logInfoTid("eq0z(&x1) : %d\n",eq0z(&x1));
      logInfoTid("eqz(&y1, _1) : %d\n",eqz(&y1,_1));
      logInfoTid("eq0u256(&scl) : %d\n",eq0u256(scl));
      //zxr->setu256(zoffset,&_inf,0);
      logInfoBigNumberTid(3*T1::getN(),"Inf: \n",zxr);
      return;  
  }


  //Q.setu256(0,&_inf, 0);

  // TODO : Either implement left to right, or count where msb is and substitute while by unrolled
  // loop

  // TODO : MAD several numbers at once using shamir's trick

  logInfoBigNumberTid(1,"SCL mul: \n",scl);
  logInfoBigNumberTid(3*T1::getN(),"Q: \n",&Q);
  logInfoBigNumberTid(3*T1::getN(),"N: \n",&N);

  #if 0
    uint32_t __restrict__ scl_cpy[NWORDS_256BIT];
    movu256(scl_cpy, scl);
    for (i=0; i< 32; i++){
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
        scmulecjac_step_r2l<T1, T2>(&Q,&N, scl_cpy, midx);
    }
  #else
    uint32_t offset;
    uint32_t msb = clzu256(scl);

    logInfoTid("msb : %d \n",msb);
    //#pragma unroll
    //for (i=msb>>U256_MBSCLUSTER; i< (1 << (NWORDS_256BIT - U256_MBSCLUSTER)); i++){
    for (i=msb; i< 1 << (NWORDS_256BIT); i++){
        //offset = i<<U256_MBSCLUSTER;
        offset = i;
        scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset,   midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+1, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+2, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+3, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+4, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+5, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+6, midx);
        //scmulecjac_step_l2r<T1, T2>(&Q,&N, scl, offset+7, midx);
     }
  #endif

  logInfoBigNumberTid(3*T1::getN(),"R-N: \n",&N);
  logInfoBigNumberTid(3*T1::getN(),"R-Q: \n",&Q);

  return;
}



template<typename T1, typename T2>
__forceinline__ __device__ void scmulecjac_opt(T1 *zxr, uint32_t zoffset, T1 *zx1, uint32_t xoffset, uint32_t *scl, kernel_params_t *params)
{
  uint32_t i;

  uint32_t __restrict__ EC_table[((1<<U256_BSELM))*3*sizeof(T2)/sizeof(uint32_t)]; // N = P
  uint32_t offset;
  uint32_t msb = clzMu256(scl);

  T1 _inf;
  infz(&_inf, params->midx);

  T1 Q(zxr->getu256(zoffset));
  T1 T(EC_table);

  Q.setu256(0,&_inf, 0);
  logInfoBigNumberTid(3*T1::getN(),"Initial Q : \n",&Q);

  build_ec_table<T1, T2>(&T, zx1, xoffset, scl, params);
  logInfoTid("msb : %d\n",msb);

  for (i=msb; i< 1 << (NWORDS_256BIT); i++){
      scmulecjac_step_l2r2<T1, T2>(&Q,&T, scl, i, params->midx);
  }

  logInfoBigNumberTid(3*T1::getN(),"Final Q : \n",&Q);

  return;
}

template<typename T1, typename T2>
__device__ void scmulecjac_step_r2l(T1 *Q,T1 *N, uint32_t *scl, mod_t midx )
{
   uint32_t  b0 = shr1u256(scl);
   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   logInfoTid("B0 : %d\n",b0);
   if (b0) {
      addecjac<T1, T2> (Q,0, N,0, Q,0, midx);
   }
   doublecjac<T1, T2>(N,N, midx);
}

template<typename T1, typename T2>
__device__ void scmulecjac_step_l2r(T1 *Q,T1 *N, uint32_t *scl, uint32_t offset, mod_t midx )
{
   uint32_t  b0 = bselu256(scl,255-offset);
   //int tid = threadIdx.x + blockDim.x * blockIdx.x;
   logInfoTid("B0 : %d\n",b0);
   doublecjac<T1, T2>(Q,Q, midx);
   //logInfoBigNumberTid(3*T1::getN(),"Q-D : \n",Q);
   if (b0) {
      addecjacmixed<T1, T2> (Q,0, N,0, Q,0, midx);
      //logInfoBigNumberTid(3*T1::getN(),"Q-A : \n",Q);
      //addecjac<T1, T2> (Q,0, N,0, Q,0, midx);
   }
   
}

template<typename T1, typename T2>
__device__ void scmulecjac_step_l2r2(T1 *Q,T1 *N, uint32_t *scl, uint32_t offset, mod_t midx )
{
   uint32_t  b = bselMu256(scl,255-offset);

   logInfoTid("offset : %d ",offset);
   logInfoTid("b : %d\n",b);
   doublecjac<T1, T2>(Q,Q, midx);
   addecjac<T1, T2> (Q,0, N,b*3, Q,0, midx);
   //logInfoBigNumberTid(3*T1::getN(),"Q : \n",Q);
   
}
template<typename T1, typename T2>
__device__ void build_ec_table(T1 *d_out,T1 *d_in, uint32_t din_offset, uint32_t *scl, kernel_params_t *params)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  uint32_t j, last_pow2, k=0;
  T1 _inf;
  uint32_t __restrict__ *_1 = misc_const_ct[params->midx]._1;
  
  infz(&_inf, params->midx);

  d_out->setu256(0,&_inf,0);

  logInfoTid("din_offset : %d\n",din_offset);

  //RR.setu256(0,d_out,0);
  //logInfoBigNumberTid(3*T1::getN(),"",&RR);
  for (j=1; j< (1<<U256_BSELM); j++){
      // check if power of 2
      if ((j & (j-1)) == 0) {
         last_pow2 = j;
          
          // check if no more numbers, ecp is inf or scl 0. In all these cases, set input to inf
          if  ((din_offset + 2*k >= 2*params->in_length/params->stride) ||
               eq0u256(&scl[k*NWORDS_256BIT]) ||
               (eq0z(d_in, din_offset + k*2) && eqz(d_in, din_offset + k*2 + 1,_1)) ) {
                d_out->setu256(j*3 * T1::getN(),&_inf,0);
          }
          // else , add number to table
          else {
                d_out->setu256(j* 3,
                            d_in, din_offset + k*2,
                            1);
                d_out->setu256(j*3+1,
                            d_in, din_offset + (k*2+1),
                            2);
                setkz(d_out,j*3+2,_1);
         }
         k++;
      } else {
         addecjac<T1,T2>(d_out,j*3,
                         d_out, last_pow2*3,
                         d_out,(j-last_pow2)*3, params->midx);

      }
  }
  /*
    uint32_t __restrict__ R[3*sizeof(T2)/sizeof(uint32_t)]; 
    T1 RR(R);
  for (j=0; j< (1<<U256_BSELM); j++){
      RR.setu256(0,d_out,T1::getN()*j*3);
      logInfoTid("idx : %d \n",j);
      logInfoBigNumberTid(3*T1::getN(),"",&RR);
  }
  logInfoTid("din_offset : %d\n",din_offset);
  for (j=0; j< U256_BSELM; j++){
      RR.setu256(0,d_in,T1::getN()*(din_offset+j*2));
      logInfoTid("In idx : %d \n",j);
      logInfoBigNumberTid(2*T1::getN(),"",&RR);
  }
  */
}


template<typename T1, typename T2>
__forceinline__ __device__ void shflxoruecc(T1 *d_out,T1 *d_in, uint32_t srcLane )
{
    ulonglong4 *in, *out;
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t i;

    #pragma unroll
    for (i=0; i<ECP_JAC_OUTDIMS*sizeof(T2)/sizeof(uint256_t);i++){
    
      in = (ulonglong4 *)d_in->getsingleu256(i);
      out = (ulonglong4 *)d_out->getsingleu256(i);

      out->x = __shfl_xor_sync(0xffffffff, in->x, srcLane);
      out->y = __shfl_xor_sync(0xffffffff, in->y, srcLane);
      out->z = __shfl_xor_sync(0xffffffff, in->z, srcLane);
      out->w = __shfl_xor_sync(0xffffffff, in->w, srcLane);
    }
}
/////////
// Temporary implemenation of future functionality. Leave here for now...


#if 0
__forceinline__ __device__
 void addecjacaff(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, const uint32_t *x2, mod_t midx)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  const uint32_t __restrict__ *y1 = &x1[NWORDS_256BIT];
  const uint32_t __restrict__ *y2 = &x2[NWORDS_256BIT];
  uint32_t __restrict__ *yr = &xr[NWORDS_256BIT];
  uint32_t __restrict__ *zr = &xr[NWORDS_256BIT*2];
  uint32_t __restrict__ *_inf = misc_const_ct[midx]._inf;
  uint32_t __restrict__ *_2 = misc_const_ct[midx]._2;
 
  uint32_t __restrict__ tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT], tmp3[NWORDS_256BIT], tmp4[NWORDS_256BIT];

  // TODO Check if I can call add to compute x + x (instead of double)
  //  if not, I should call double below. I don't want to to avoid warp divergnce
  if (eqz((const uint32_t *)x1, (const uint32_t *)x2) &&   // u1 == u2
       !eqz( (const uint32_t *) y1, (const uint32_t *) y2)){  // s1 != s2
          memcpy(xr, _inf, 3 * NWORDS_256BIT * sizeof(uint32_t));
	  return;  //  if U1 == U2 and S1 == S2 => P1 == P2 (call double)
  }

  /*
  if (tid == 0){
     logInfoBigNumber("x1\n",(uint32_t *)x1);
     logInfoBigNumber("y1\n",(uint32_t *)y1);
     logInfoBigNumber("x2\n",(uint32_t *)x2);
     logInfoBigNumber("y2\n",(uint32_t *)y2);
  }
  */
  subz(zr, x2, x1, midx);     // H = tmp2 = u2 - u1
  if (tid == 0){
    logInfoBigNumber("H\n",(uint32_t *)zr);
  }

  squarez(tmp3, zr,        midx);     // Hsq = tmp3 = H * H 
  mulz(tmp2, tmp3, zr, midx);     // Hcube = tmp2 = Hsq * H 
  mulz(tmp1, x1, tmp3, midx);     // tmp1 = u1 * Hsq

  /*
  if (tid == 0){
    logInfoBigNumber("Hsq\n",(uint32_t *)tmp3);
    logInfoBigNumber("Hcube\n",(uint32_t *)tmp2);
    logInfoBigNumber("u1 * Hsq\n",(uint32_t *)tmp1);
  }
  */

  subz(tmp3, y2, y1, midx);        // R = tmp3 = S2 - S1 tmp1=u1*Hsq, tmp2=Hcube, xr=free, yr=s1, zr=zr
  mulz(yr, y1, tmp2, midx);     // yr = Hcube * s1
  squarez(xr, tmp3, midx);     // xr = R * R

  /*
  if (tid == 0){
    logInfoBigNumber("R\n",(uint32_t *)tmp3);
    logInfoBigNumber("s1\n",(uint32_t *)yr);
    logInfoBigNumber("Rsq\n",(uint32_t *)xr);
  }
  */
  subz(xr, xr, tmp2, midx);        // xr = x3= (R*R)-Hcube, yr = Hcube * S1, zr=zr, tmp1=u1*Hsq, tmp2 = Hcube, tmp3 = R

  // TODO muluk256
  mul2z(tmp4, tmp1, midx);     // tmp4 = u1*hsq *_2

  /*
  if (tid == 0){
    logInfoBigNumber("Rsq - Hcube\n",(uint32_t *)xr);
    logInfoBigNumber("u1 * Hsq * 2\n",(uint32_t *)tmp4);
  }
  */
  subz(xr, xr, tmp4, midx);               // x3 = xr
  subz(tmp1, tmp1, xr, midx);       // tmp1 = u1*hs1 - x3
  mulz(tmp1, tmp1, tmp3, midx);  // tmp1 = r * (u1 * hsq - x3)
  subz(yr, tmp1, yr, midx);

  /*
  if (tid == 0){
    logInfoBigNumber("X3\n",(uint32_t *)xr);
    logInfoBigNumber("u1 * hsq - x3\n",(uint32_t *)tmp1);
    logInfoBigNumber("Y3\n",(uint32_t *)yr);
  }
  */
}
#endif

#if 0
__global__ void addecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1, *x2, *xr;
 
    if(tid >= params->in_length/6) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * 2 * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];
    x2 = (uint32_t *) &in_vector[(tid * 2 + 1) * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];
    
    addecldr(xr, x1, x2, x1, params->midx);

    return;
}
__global__ void doublecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t __restrict__ *x1,*xr;
 
    if(tid >= params->in_length/3) {
      return;
    }

    x1 = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];

    xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];
    
    doublecldr(xr, x1, params->midx);

  return;
}
__global__ void scmulecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;

   uint32_t __restrict__ *x1, *scl, *xr;
 
   if(tid >= params->in_length/3) {
     return;
   }

   x1  = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_LDR_INXOFFSET];
   scl = (uint32_t *) &in_vector[tid * ECK_LDR_INOFFSET + ECP_SCLOFFSET];

   xr = (uint32_t *) &out_vector[tid * ECK_LDR_OUTOFFSET + ECP_LDR_OUTXOFFSET];

   
   ldrstep(xr, x1, scl,  params->midx);

   return;
}

__global__ void madecldr_kernel(uint32_t *out_vector, uint32_t *in_vector, kernel_params_t *params)
{
  return;
}

    
__forceinline__ __device__
 void addecldr(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, 
            const uint32_t __restrict__ *x2, const uint32_t __restrict__ *xp, mod_t midx)
{
   // Xr = -4*b Z1 * Z2 * (X1 * Z2 + X2 * Z1) + (X1 * X2)^2 
   // Zr = xp * (X1 * Z2 - X2 * Z1)^2

   // 7 M, 2 SQ, 3 ADD
   const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT];
   const uint32_t __restrict__ *z2 = &x2[NWORDS_256BIT];
   uint32_t __restrict__ *zr =&zr[NWORDS_256BIT];

   uint32_t tmp1[NWORDS_256BIT];
   uint32_t __restrict__ *_4b = misc_const_ct[midx]._4b;
   

   mulz(tmp1, x2  , z1  , midx);      
   mulz(xr  , x1  , z2  , midx);      
   subz(   zr  , xr  , tmp1, midx);
   addz(   tmp1, tmp1, xr  , midx);
   mulz(tmp1, tmp1, z2  , midx);    
   mulz(tmp1, tmp1, z1  , midx);    
   mulz(xr  , x1  , x2  , midx);      
   squarez(xr  , xr         , midx);    
   // multiply by 12. 
   //  Using Montgomery: 136 mul + 346 add.
   //  Chaining 12 additions : 0 mul + 84 adds + modulus!!!
   // TODO : Use muluk256 function
   mulkz(tmp1, tmp1,_4b  , midx);  
   subz(   xr,   tmp1, xr  , midx);
   squarez(zr,   zr         , midx);     
   mulz(zr,   zr  , xp  , midx);   

  return;
}

__forceinline__ __device__
 void doublecldr(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, mod_t midx)
{
  // Xr = X1^4 - 8*b * X1*Z1^3
  // Zr = 4*Z1 * (X1^3 + b*Z1^3) 

  // 7 M, 3 SQ, 2 Add
  const uint32_t __restrict__ *z1 = &x1[NWORDS_256BIT];
  uint32_t __restrict__ *zr =&zr[NWORDS_256BIT];

  uint32_t tmp1[NWORDS_256BIT], tmp2[NWORDS_256BIT];
  uint32_t __restrict__ *_4 = misc_const_ct[midx]._4;
  uint32_t __restrict__ *b = ecbn128_params_ct[midx].b;
  uint32_t __restrict__ *_8b = misc_const_ct[midx]._8b;

  squarez(xr,  z1,           midx);    
  mulz(zr,  xr,   z1,   midx);      // Zr = Z1^3
  mulz(xr,  zr,   x1,   midx);      
  // TODO muluk256
  mulkz(xr,  xr,  _8b,   midx);      // Xr = 8b * X1 * Z1^3
  squarez(tmp1, x1,         midx);      
  squarez(tmp2, tmp1,       midx);    
  subz(   xr,  tmp2, xr,   midx);

  // TODO muluk256
  mulkz(zr,  zr,   b,    midx);      // Zr = b*Z1^3
  mulz(tmp1, tmp1,  x1,   midx);     
  addz(   zr, tmp1,   zr,   midx);
  // TODO muluk256
  mulkz(zr, zr,   _4,    midx);
  mulz(zr, zr,   z1,    midx); 

  return;
}

// NOTE : EC points are in affine coordinates => Pz = 1 (in montgomery someting else)
// NOTE : EC points in montgomery, scl normal 
__forceinline__ __device__
 void ldrstep(uint32_t __restrict__ *xr, const uint32_t __restrict__ *x1, uint32_t *scl, mod_t midx)
{
  uint32_t b0, idxn, idxp;
  uint32_t __restrict__ *_1 = misc_const_ct[midx]._1;

  uint32_t __restrict__ R[4*NWORDS_256BIT];

  //R[] = {[1,0],[X,1]} 
  memcpy(R, _1, NWORDS_256BIT * sizeof(uint32_t));
  memcpy(&R[3 * NWORDS_256BIT], _1, NWORDS_256BIT * sizeof(uint32_t));
  memcpy(&R[2 * NWORDS_256BIT], x1, NWORDS_256BIT * sizeof(uint32_t));

  while (!eq0u256(scl)){
     b0 = shr1u256(scl);
     idxn = ~b0 * 2 * NWORDS_256BIT;
     idxp =  b0 * 2 * NWORDS_256BIT;
     /*
     if (b0) { R0 = R0 + R1; R1 = R1 + R1;
     else {    R1 = R0 + R1; R0 = R0 + R0}
     */
     addecldr(&R[idxn], &R[idxn], &R[idxp], x1, midx);
     doublecldr(&R[idxp], &R[idxp], midx);
  }
   // TODO
   // Retrieve y(P) . Not sure if i need to convert to affine now. If I don't,
   // then I have three coordinates and it doesn't fit in my allocated space
   //
   // P = (x1, y1) , Q = (x2, y2), P-Q = (x,y)
   // Q = k P => x(R0) = X(Q), x(R1) = X(P-Q)
   //
   // y(P) = y1 = [2b + (a + x * x1) * (x + x1) - x2(x - x1) ^ 2] / (2*y)

  memcpy(xr,R,2 * NWORDS_256BIT * sizeof(uint32_t));

  return;
}

#endif


