/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : bigint.cu
//
// Date       : 05/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementation of biginteger class. 
// ------------------------------------------------------------------

*/

/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU
This class will get translated into python via swig
*/

#include <assert.h>
#include <iostream>

#include "types.h"
#include "bigint.h"
#include "bigint_device.h"

using namespace std;
/*
    Constructor

    Arguments :
      array_host : array of 256 bit numbers located at the host side
      length : Number of 256 bit numbers
*/
BigInt::BigInt (uint32_t* array_host, uint32_t *p, uint32_t length) {

  this->array_host = array_host;
  this->len = length;

  uint32_t size = len * sizeof(uint32_t) * NWORDS_256BIT;

  hipError_t err = hipMalloc((void**) &this->array_device, size);
  assert(err == 0);

  err = hipMalloc((void**) &this->p, sizeof(uint32_t) * NWORDS_256BIT);
  assert(err == 0);

  err = hipMemcpy(this->array_device, this->array_host, size, hipMemcpyHostToDevice);
  assert(err == 0);

  err = hipMemcpy(this->p, p, sizeof(uint32_t) * NWORDS_256BIT, hipMemcpyHostToDevice);
  assert(err == 0);

}

void BigInt::addm() {
  //addm_kernel<<<64, 64>>>(array_device, p, len);
  addm_kernel(array_device, p, len);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void BigInt::retrieve(uint32_t *array_host) {
  uint32_t size = len * sizeof(uint32_t) * NWORDS_256BIT ;
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0) { cout << err << endl; assert(0); }
}

BigInt::~BigInt() {
  hipFree(array_device);
  hipFree(p);
}
