/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : z2_device.cu
//
// Date       : 23/03/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementatoin of Extended (2) Finite Field arithmetic
// 
// ------------------------------------------------------------------

*/

#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "utils_device.h"
#include "u256_device.h"
#include "z2_device.h"

__device__ Z2_t::Z2_t() {}
__device__ Z2_t::Z2_t(uint32_t *x) : el(x) {}

__device__ uint32_t * Z2_t::getu256()
{
  return el;
}

__device__ uint32_t * Z2_t::get2u256()
{
  return &el[NWORDS_256BIT];
}

__device__ uint32_t * Z2_t::getsingleu256(uint32_t offset)
{
  return &el[offset*NWORDS_256BIT];
}
__device__ uint32_t * Z2_t::getu256(uint32_t offset)
{
  return &el[offset*ECP2_JAC_N256W*NWORDS_256BIT];
}

__device__ uint32_t * Z2_t::get2u256(uint32_t offset)
{
  return &el[offset*(ECP2_JAC_N256W+1)*NWORDS_256BIT];
}

__device__ void Z2_t::setu256(uint32_t xoffset, Z2_t *y, uint32_t yoffset)
{ 
   //memcpy(&el[xoffset*ECP2_JAC_N256W*NWORDS_256BIT],
         //&y->el[yoffset*ECP2_JAC_N256W*NWORDS_256BIT],
         //3 * ECP2_JAC_N256W* NWORDS_256BIT * sizeof(uint32_t));
   movu256x6(&el[xoffset*NWORDS_256BIT],&y->el[yoffset*NWORDS_256BIT]);
   //movu256(&el[(xoffset+1)*NWORDS_256BIT],&y->el[(yoffset+1)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+2)*NWORDS_256BIT],&y->el[(yoffset+2)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+3)*NWORDS_256BIT],&y->el[(yoffset+3)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+4)*NWORDS_256BIT],&y->el[(yoffset+4)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+5)*NWORDS_256BIT],&y->el[(yoffset+5)*NWORDS_256BIT]);
}

__device__ void Z2_t::setu256(uint32_t xoffset, uint32_t *y, uint32_t yoffset)
{ 
    //memcpy(&el[xoffset*ECP2_JAC_N256W*NWORDS_256BIT],
           //&y[yoffset*ECP2_JAC_N256W*NWORDS_256BIT],
           //3* ECP2_JAC_N256W * NWORDS_256BIT * sizeof(uint32_t));
   movu256x6(&el[xoffset*NWORDS_256BIT],&y[yoffset*NWORDS_256BIT]);
   //movu256(&el[(xoffset+1)*NWORDS_256BIT],&y[(yoffset+1)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+2)*NWORDS_256BIT],&y[(yoffset+2)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+3)*NWORDS_256BIT],&y[(yoffset+3)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+4)*NWORDS_256BIT],&y[(yoffset+4)*NWORDS_256BIT]);
   //movu256(&el[(xoffset+5)*NWORDS_256BIT],&y[(yoffset+5)*NWORDS_256BIT]);
}

__device__ void Z2_t::setu256(uint32_t xoffset, Z2_t *y, uint32_t yoffset, uint32_t ysize)
{ 
       //memcpy(&el[xoffset*ECP2_JAC_N256W*NWORDS_256BIT],
              //&y->el[yoffset*ECP2_JAC_N256W*NWORDS_256BIT],
              //ysize * ECP2_JAC_N256W* NWORDS_256BIT * sizeof(uint32_t));
   movu256(&el[(xoffset)*2*NWORDS_256BIT],&y->el[(yoffset)*2*NWORDS_256BIT]);
   movu256(&el[(xoffset)*2*NWORDS_256BIT+NWORDS_256BIT],&y->el[(yoffset)*2*NWORDS_256BIT+NWORDS_256BIT]);
}

__device__ void Z2_t::setu256(uint32_t xoffset, uint32_t *y, uint32_t yoffset, uint32_t ysize)
{ 
    //memcpy(&el[xoffset*ECP2_JAC_N256W*NWORDS_256BIT],
           //&y[yoffset*ECP2_JAC_N256W*NWORDS_256BIT],
           //ysize * ECP2_JAC_N256W * NWORDS_256BIT * sizeof(uint32_t));
   movu256(&el[(xoffset)*2*NWORDS_256BIT],&y[(yoffset)*2*NWORDS_256BIT]);
   movu256(&el[(xoffset)*2*NWORDS_256BIT+NWORDS_256BIT],&y[(yoffset)*2*NWORDS_256BIT+NWORDS_256BIT]);
}
/*
__device__ void Z2_t::set2u256(uint32_t xoffset, uint32_t *y, uint32_t yoffset, uint32_t ysize)
{ 
    //memcpy(&el[xoffset*ECP2_JAC_N256W*NWORDS_256BIT],
           //&y[yoffset*ECP2_JAC_N256W*NWORDS_256BIT],
           //ysize * NWORDS_256BIT * sizeof(uint32_t));
   movu256(&el[(xoffset+1)*NWORDS_256BIT],&y[(yoffset+1)*NWORDS_256BIT]);
}
*/

__device__ void Z2_t::assign(uint32_t *y)
{ 
    el = y;
}

__device__  uint32_t Z2_t::getN()
{
    return ECP2_JAC_N256W;
}

////

__device__ uint32_t eq0z(Z2_t *x)
{ 
   return (eq0u256(x->getu256()) && eq0u256(x->get2u256()));
}

__device__ uint32_t eq1z(Z2_t *x)
{ 
   return (eq1u256(x->getu256()) && eq0u256(x->get2u256()));
}

__device__ uint32_t eqz(Z2_t *x, Z2_t *y)
{
  return (equ256(x->getu256(), y->getu256()) && equ256(x->get2u256(), y->get2u256()));
}
__device__ uint32_t eqz(Z2_t *x, uint32_t *y)
{
  return (equ256(x->getu256(), y) && equ256(x->get2u256(),  &y[NWORDS_256BIT]));
}

__device__  void squarez(Z2_t *z, Z2_t *x, mod_t midx)
{
  sqmontu256_2(z->getu256(), x->getu256(),         midx);  
}

__device__ void mulz(Z2_t *z,  Z2_t *x, Z2_t *y, mod_t midx)
{
  mulmontu256_2(z->getu256(), x->getu256(), y->getu256(), midx);  
}

__device__ void mul2z(Z2_t *z,  Z2_t *x, mod_t midx)
{
  addmu256(z->getu256(), x->getu256(), x->getu256(), midx);    
  addmu256(z->get2u256(), x->get2u256(), x->get2u256(), midx);    
}
__device__ void mul3z(Z2_t *z,  Z2_t *x, mod_t midx)
{
   addmu256(z->getu256(), x->getu256(), x->getu256(), midx);    
   addmu256(z->get2u256(), x->get2u256(), x->get2u256(), midx);    

   addmu256(z->getu256(), z->getu256(), x->getu256(), midx);    
   addmu256(z->get2u256(), z->get2u256(), x->get2u256(), midx);    
}
__device__ void mul4z(Z2_t *z,  Z2_t *x, mod_t midx)
{
  addmu256(z->getu256(), x->getu256(), x->getu256(), midx);    
  addmu256(z->get2u256(), x->get2u256(), x->get2u256(), midx);    

  addmu256(z->getu256(), z->getu256(), z->getu256(), midx);    
  addmu256(z->get2u256(), z->get2u256(), z->get2u256(), midx);    
}
__device__ void mul8z(Z2_t *z,  Z2_t *x, mod_t midx)
{
  addmu256(z->getu256(), x->getu256(), x->getu256(), midx);    
  addmu256(z->get2u256(), x->get2u256(), x->get2u256(), midx);    

  addmu256(z->getu256(), z->getu256(), z->getu256(), midx);    
  addmu256(z->get2u256(), z->get2u256(), z->get2u256(), midx);    
    
  addmu256(z->getu256(), z->getu256(), z->getu256(), midx);    
  addmu256(z->get2u256(), z->get2u256(), z->get2u256(), midx);    
}

__device__ void subz(Z2_t *z, Z2_t *x, Z2_t *y, mod_t midx)
{
  submu256(z->getu256(), x->getu256(), y->getu256(), midx);    
  submu256(z->get2u256(), x->get2u256(), y->get2u256(), midx);    
}
__device__ void addz(Z2_t *z, Z2_t *x, Z2_t *y, mod_t midx)
{
  addmu256(z->getu256(), x->getu256(), y->getu256(), midx);    
  addmu256(z->get2u256(), x->get2u256(), y->get2u256(), midx);    
}

__device__ void movz(Z2_t *x, uint32_t xoffset, uint32_t *y, uint32_t yoffset, uint32_t size)
{
  x->setu256(xoffset,y,yoffset,size);
}
__device__ void movz(uint32_t *y, uint32_t yoffset, Z2_t *x, uint32_t xoffset, uint32_t size)
{
  uint32_t i;
  
  #pragma unroll
  for (i=0; i< 2*size; i++){
    movu256(&y[yoffset],x->getu256(xoffset));
    xoffset += NWORDS_256BIT;
    yoffset += NWORDS_256BIT;
  }
}

__device__ void setkz(Z2_t *z, uint32_t offset, uint32_t *x)
{
  z->setu256(offset,x,0,1);
}
__device__ void xeccz(Z2_t *z, Z2_t *x)
{
  z->assign(x->getu256());
}
__device__ void yeccz(Z2_t *z, Z2_t *x)
{
  z->assign(x->getu256(1));
}
__device__ void zeccz(Z2_t *z, Z2_t *x)
{
  z->assign(x->getu256(2));
}

__device__ void infz(Z2_t *z, mod_t midx)
{
  z->assign(misc_const_ct[midx]._inf2);
}
