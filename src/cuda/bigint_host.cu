/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU
This class will get translated into python via swig
*/

#include <bigint_kernel.cu>
#include <bigint.hh>
#include <assert.h>
#include <iostream>
using namespace std;

BigInt::BigInt (uint32_t* array_host_, uint32_t length_) {
  array_host = array_host_;
  len = length_;
  uint32_t size = len * sizeof(uint32_t);
  hipError_t err = hipMalloc((void**) &array_device, size);
  assert(err == 0);
  err = hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
  assert(err == 0);
}

void BigInt::mod_add() {
  BigInt_ModAdd256<<<64, 64>>>(array_device, len);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void BigInt::retreive() {
  int size = length * sizeof(int);
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0) { cout << err << endl; assert(0); }
}

BigInt::~BigInt() {
  hipFree(array_device);
}
