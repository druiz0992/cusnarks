/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : z1_device.cu
//
// Date       : 20/03/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementatoin of Finite Field arithmetic
// 
// ------------------------------------------------------------------

*/

#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "utils_device.h"
#include "u256_device.h"
#include "z1_device.h"


__device__ uint32_t eq0z(Z1_t *x)
{ 
   return eq0u256(x->getu256());
}

__device__ uint32_t eqz(Z1_t *x, Z1_t *y)
{
  return equ256(x->getu256(), y->getu256());
}
__device__  void squarez(Z1_t *z, Z1_t *x, mod_t midx)
{
  sqmontu256(z->getu256(), x->getu256(),         midx);  
}

__device__ void mulz(Z1_t *z,  Z1_t *x, Z1_t *y, mod_t midx)
{
  mulmontu256(z->getu256(), x->getu256(), y->getu256(), midx);  
}

__device__ void subz(Z1_t *z, Z1_t *x, Z1_t *y, mod_t midx)
{
  submu256(z->getu256(), x->getu256(), y->getu256(), midx);    
}
__device__ void addz(Z1_t *z, Z1_t *x, Z1_t *y, mod_t midx)
{
  addmu256(z->getu256(), x->getu256(), y->getu256(), midx);    
}

__device__ void movz(Z1_t *x, uint32_t xoffset, uint32_t *y, uint32_t yoffset, uint32_t size)
{
  x->setu256(xoffset,y,yoffset,size);
}
__device__ void movz(uint32_t *y, uint32_t yoffset, Z1_t *x, uint32_t xoffset, uint32_t size)
{
  uint32_t i;
  
  #pragma unroll
  for (i=0; i< size; i++){
    movu256(&y[yoffset],x->getu256(xoffset));
    xoffset += NWORDS_256BIT;
    yoffset += NWORDS_256BIT;
  }
}

__device__ void xeccz(Z1_t *z, Z1_t *x)
{
  z->assign(x->getu256());
}
__device__ void yeccz(Z1_t *z, Z1_t *x)
{
  z->assign(x->getu256(1));
}
__device__ void zeccz(Z1_t *z, Z1_t *x)
{
  z->assign(x->getu256(2));
}

__device__ void infz(Z1_t *z, mod_t midx)
{
  z->assign(misc_const_ct[midx]._inf);
}
__device__ void _1z(Z1_t *z, mod_t midx)
{
  z->assign(misc_const_ct[midx]._1);
}
__device__ void _2z(Z1_t *z, mod_t midx)
{
  z->assign( misc_const_ct[midx]._2);
}
__device__ void _3z(Z1_t *z, mod_t midx)
{
  z->assign(misc_const_ct[midx]._3);
}
__device__ void _4z(Z1_t *z, mod_t midx)
{
  z->assign(misc_const_ct[midx]._4);
}
__device__ void _8z(Z1_t *z, mod_t midx)
{
  z->assign(misc_const_ct[midx]._8);
}
