/*
    Copyright 2018 0kims association.

    This file is part of cusnarks.

    cusnarks is a free software: you can redistribute it and/or
    modify it under the terms of the GNU General Public License as published by the
    Free Software Foundation, either version 3 of the License, or (at your option)
    any later version.

    cusnarks is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
    or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
    more details.

    You should have received a copy of the GNU General Public License along with
    cusnarks. If not, see <https://www.gnu.org/licenses/>.

// ------------------------------------------------------------------
// Author     : David Ruiz
//
// File name  : ec2bn128.cu
//
// Date       : 22/02/2019
//
// ------------------------------------------------------------------
//
// Description:
//  Implementation of CUDA EC Kernel processing for extended fields
//
// ------------------------------------------------------------------

*/

#include <assert.h>
#include <iostream>
#include <stdio.h>

#include "types.h"
#include "hip/hip_runtime.h"
#include "rng.h"
#include "cusnarks_kernel.h"
#include "ec2bn128.h"
#include "ecbn128_device.h"

using namespace std;

static kernel_cb ecbn128_2_kernel_callbacks[] = {addec2jacaff_kernel, addec2jac_kernel, doublec2jacaff_kernel, doublec2jac_kernel,
                                                 scmulec2jac_kernel, sc1mulec2jac_kernel, madec2jac_kernel, madec2jac_shfl_kernel};

EC2BN128::EC2BN128 (uint32_t len) : CUSnarks( len * ECP2_JAC_INDIMS, NWORDS_256BIT * sizeof(uint32_t) * len *  ECP2_JAC_INDIMS,
		                            len * ECP2_JAC_OUTDIMS,  NWORDS_256BIT * sizeof(uint32_t) * len * ECP2_JAC_OUTDIMS, 
                                            ecbn128_2_kernel_callbacks, 0)
{
}

EC2BN128::EC2BN128 (uint32_t len, const uint32_t seed) :  CUSnarks(len * ECP2_JAC_INDIMS, NWORDS_256BIT * sizeof(uint32_t) * len * ECP2_JAC_INDIMS,
				                                 len * ECP2_JAC_OUTDIMS, NWORDS_256BIT * sizeof(uint32_t) * len * ECP2_JAC_OUTDIMS,
						       ecbn128_2_kernel_callbacks, seed)
{
}

