#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void shl_xor_kernel(int *out_vector_d, int *in_vector_d, int lane, int n)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    //if (tid % 2 == 0){
       out_vector_d[tid] = __shfl_xor_sync(0xffffffff, in_vector_d[tid], lane, 1<<n);
    //} else {
       //out_vector_d[tid] = __shfl_xor_sync(0xffffffff, in_vector_d[tid], lane);
    //}

}

main()
{
  int *in_vector_d, *out_vector_d;
  int in_vector_h[32], out_vector_h[32];
  int i,j,k;

  for (i=0; i< 32; i++){
    in_vector_h[i] = i;
  }
 

  hipMalloc((void**) &in_vector_d, 32 * sizeof(int));
  hipMalloc((void**) &out_vector_d, 32 * sizeof(int));

  hipMemcpy(in_vector_d, in_vector_h, 32 * sizeof(int), hipMemcpyHostToDevice);


  k=5;
  //for (k=1;k<5;k++){
     for (j=0;j<k;j++){
        shl_xor_kernel<<<1,32>>>(out_vector_d, in_vector_d, 1<<j, k);
   
        hipDeviceSynchronize();
        hipMemcpy(out_vector_h, out_vector_d, 32 * sizeof(int), hipMemcpyDeviceToHost);
        printf("In:\t");
        for (i=0;i< 32; i++){
           printf("%d ",in_vector_h[i]);
        }
        printf("\n");
   
        printf("Out[%d]:\t", 1<<j);
        for (i=0;i< 32; i++){
           printf("%d ",out_vector_h[i]);
        }
        printf("\n");
     }
  //}

  hipFree(in_vector_d);
  hipFree(out_vector_d);
}

